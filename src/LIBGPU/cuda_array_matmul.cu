#include "hip/hip_runtime.h"
 
 #include <stdio.h>
 #include <stdlib.h>
 #include <hip/hip_runtime.h>
 #include <hipblas.h>
 #include <hip/hip_complex.h>

#define MAX_BLOCK 22

hipDoubleComplex *collect_array;


//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

texture<int2,1,hipReadModeElementType> tex;
inline void   bind_x(hipDoubleComplex *x) {    hipBindTexture(0,tex,x); };
inline void unbind_x()                   {  hipUnbindTexture(  tex  ); };
__inline__  __device__ hipDoubleComplex fetch_x(const int& i)
  {
         int  jj = 2*(i-1);
         int2 v  = tex1Dfetch(tex,jj);
      double rr  = __hiloint2double(v.y, v.x);
              v  = tex1Dfetch(tex,jj+1);
      double im  = __hiloint2double(v.y, v.x);
      return make_hipDoubleComplex(rr,im);
  }

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

__global__ void matmul_array_cuda( int nfrequ, int nnn, hipDoubleComplex *collect )


{

    hipDoubleComplex zero; zero=make_hipDoubleComplex(0.0,0.0); 
   
    __shared__ hipDoubleComplex shared_[256];
               hipDoubleComplex E[MAX_BLOCK*MAX_BLOCK];
    int ifrequ;

    for(int iorb=0;iorb<nnn*nnn;iorb++) shared_[iorb]=zero; ifrequ=threadIdx.x;

    if(ifrequ<nfrequ)
    { 
       for(int iorb=0;iorb<nnn*nnn;iorb++) E[iorb] = collect[iorb+nnn*nnn*ifrequ]; 
    };

}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

extern "C" void complex_array_of_matrices_matmul(int* nnn_, int* nfrequ_, hipDoubleComplex* collect_ )

{

  int nnn ; int nfrequ; nnn=*nnn_; nfrequ=*nfrequ_;

  if(nnn>MAX_BLOCK)   { printf( " sum of inverse complex cuda : matrices are too big!!!!! \n"); };
  if(nfrequ>512)      { printf( " too many blocks in inverse array cuda \n ");};

  hipMalloc (  (void**)&collect_array  , nfrequ*nnn*nnn*sizeof(hipDoubleComplex) ); 
  hipMemcpy ( collect_array ,collect_  , nnn*nnn*nfrequ*sizeof(hipDoubleComplex) , hipMemcpyHostToDevice);

  matmul_array_cuda <<<1,512>>> ( nfrequ,nnn,collect_array);

  hipEventSynchronize(0); hipDeviceSynchronize();
  hipMemcpy (collect_ ,collect_array  , nfrequ*nnn*nnn*sizeof(hipDoubleComplex) , hipMemcpyDeviceToHost);
  hipFree(collect_array);


}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
