#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_complex.h>

//********************************************
//********************************************
//********************************************

#define IBSET(a,b) ((a) |= (1<<(b)))
#define IBCLR(a,b) ((a) &= ~(1<<(b)))
#define BTEST(a,b) ((a) & (1<<(b)))

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

const int use_texture_lcf = 1 ;
      int use_texture_rank_c = 1 ;

#define WARP_SIZE 32
#define MAX_BLOCK 65500
#define max_rank 225000000

texture<int2,1,hipReadModeElementType> tex,texb,texc,texbc;
texture<int,1,hipReadModeElementType>  texr;

inline void   bind_x_r(int *x)         {hipBindTexture(0,texr,x);};
inline void unbind_x_r()               {hipUnbindTexture(texr);};

inline void   bind_x(hipDoubleComplex *x, int N) {hipBindTexture(0,tex,x,N*sizeof(hipDoubleComplex));};
inline void unbind_x()                          {hipUnbindTexture(tex);};

inline void   bind_x_(hipDoubleComplex *x, hipDoubleComplex *y, hipDoubleComplex *z ) 
                                       {hipBindTexture(0,texb,x); hipBindTexture(0,texc,y); hipBindTexture(0,texbc,z);};
inline void unbind_x_()                {hipUnbindTexture(texb);hipUnbindTexture(texc);hipUnbindTexture(texbc);};

__inline__  __device__ int    fetch_r(const int& i) {  int  v = tex1Dfetch(texr,i);  return v; }

__inline__  __device__ hipDoubleComplex fetch_x(const int& i)
  { int  jj = 2*(i); int2 v  = tex1Dfetch(tex,jj); double rr  = __hiloint2double(v.y, v.x); v  = tex1Dfetch(tex,jj+1);
    double im  = __hiloint2double(v.y, v.x); return make_hipDoubleComplex(rr,im); }
__inline__  __device__ hipDoubleComplex fetchb(const int& i)
  { int  jj = 2*(i); int2 v  = tex1Dfetch(texb,jj); double rr  = __hiloint2double(v.y, v.x); v  = tex1Dfetch(texb,jj+1);
    double im  = __hiloint2double(v.y, v.x); return make_hipDoubleComplex(rr,im); }
__inline__  __device__ hipDoubleComplex fetchc(const int& i)
  { int  jj = 2*(i); int2 v  = tex1Dfetch(texc,jj); double rr  = __hiloint2double(v.y, v.x); v  = tex1Dfetch(texc,jj+1);
    double im  = __hiloint2double(v.y, v.x); return make_hipDoubleComplex(rr,im); }
__inline__  __device__ hipDoubleComplex fetchbc(const int& i)
  { int  jj = 2*(i); int2 v  = tex1Dfetch(texbc,jj); double rr  = __hiloint2double(v.y, v.x); v  = tex1Dfetch(texbc,jj+1);
    double im  = __hiloint2double(v.y, v.x); return make_hipDoubleComplex(rr,im); }

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

inline void cublasZscal_no_device (int n, hipDoubleComplex alpha, hipDoubleComplex *y, int incy)
 { for (int i=0; i<n; i++) { y[i] = hipCmul( alpha, y[i] ); } }

//********************************************
//********************************************

inline void cublasZaxpy_no_device (int n, hipDoubleComplex alpha, hipDoubleComplex *x, int incx,hipDoubleComplex *y, int incy)
 { for (int i=0; i<n; i++) { y[i] = hipCadd( y[i],  hipCmul( alpha, x[i] ) ); } }

//********************************************
//********************************************

inline hipDoubleComplex cublasZdotu_no_device (int n, hipDoubleComplex *x, int incx, hipDoubleComplex *y, int incy)
 {
  hipDoubleComplex dot_; dot_=make_hipDoubleComplex(0.0,0.0) ;
  for (int i=0; i<n; i++) { dot_ = hipCadd(dot_,hipCmul( hipConj(x[i]), y[i] ) ); } return dot_;
 }

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

__global__ void norm_vec_ker_complex(int size , hipDoubleComplex *x, double *normv)

{
    int blocksize = blockDim.x ;
   __shared__ double  temp[512]; __shared__ double temp2[512]; int row; int  ii;

    temp[threadIdx.x]=0.;
    for(row=threadIdx.x; row<size; row+=blocksize)
   {
    temp2[threadIdx.x] = hipCabs(x[row]); temp2[threadIdx.x]*= temp2[threadIdx.x];
    temp[threadIdx.x] += temp2[threadIdx.x];
   };
   __syncthreads ();
    *normv=0.0; for(ii=0;ii<blocksize;ii++){ *normv+=temp[ii];}; *normv=sqrt(*normv);
}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

__global__ void real_scal_vec_ker_complex( int size , hipDoubleComplex *x, hipDoubleComplex *y, double *normv)

{
    int blocksize = blockDim.x ; __shared__ double temp[512]; int row , ii;

    temp[threadIdx.x]=0.0;
    for(row=threadIdx.x; row<size; row+=blocksize) { temp[threadIdx.x] += hipCreal( hipCmul(  hipConj( x[row] ) ,y[row]  ) ); };

  __syncthreads ();
   *normv=0.0; for(ii=0;ii<blocksize ;ii++){ *normv+=temp[ii];};
}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

__global__ void normalize_vec_ker_complex(int ngrid, int size ,hipDoubleComplex *x, double *normv)
{
    int blocksize = blockDim.x ;
    int row ;
    for(int iii=0;iii<=ngrid;iii++)
    {
    row= blocksize * (blockIdx.x + iii*MAX_BLOCK)+ threadIdx.x ;
    if(row<size) 
      { x[row]=hipCmul( make_hipDoubleComplex(1.0 / *normv ,0.0), x[row] ); };
    };
}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

  //************************************************//
  //               Kernel Hmult                     //
  //************************************************//

  __global__  void Hmult_ker_complex_fly(int ngrid, int blocksize, int psize, hipDoubleComplex *vec_out, const hipDoubleComplex *vec_in, 
    const double *quart, const hipDoubleComplex* Eb, const hipDoubleComplex* Ec, const hipDoubleComplex* Vbc, 
    const int* sector_states, const int* sector_rank, const int norbs, const int bathnorbs, const int impnorbs, 
    const int* imporbs_, const int* bathorbs_, const int* maskEb_, const int* maskEc_, const int* maskVbc_, const int use_texture_lcf,
    const int use_texture_rank_c)

{
   __shared__ hipDoubleComplex sdata[16][WARP_SIZE];
   __shared__ short int maskEc[32*32],maskEb[32*32],maskVbc[32*32];
   __shared__ short int imporbs[32],bathorbs[32];

    int istate ;  const int warp_lane   = threadIdx.y; const int thread_lane = threadIdx.x; 

    int jstate,kets_out,kets_in,jj,iorb,jorb,n1,n2; 
    hipDoubleComplex hoffdiag,vecin;
    short int fermion_sign;

    if(warp_lane==0) for(iorb=thread_lane;iorb<impnorbs*impnorbs;iorb+=WARP_SIZE)   maskEc[iorb]=maskEc_[iorb];
    if(warp_lane==1) for(iorb=thread_lane;iorb<bathnorbs*bathnorbs;iorb+=WARP_SIZE) maskEb[iorb]=maskEb_[iorb];
    if(warp_lane==2) for(iorb=thread_lane;iorb<bathnorbs*impnorbs;iorb+=WARP_SIZE)  maskVbc[iorb]=maskVbc_[iorb];
    if(warp_lane==3) for(iorb=thread_lane;iorb<bathnorbs;iorb+=WARP_SIZE)           bathorbs[iorb]=bathorbs_[iorb];
    if(warp_lane==4) for(iorb=thread_lane;iorb<impnorbs;iorb+=WARP_SIZE)            imporbs[iorb]=imporbs_[iorb];

    __syncthreads();


  for(int iii=0;iii<=ngrid;iii++){

    istate = blocksize * ( blockIdx.y + iii*MAX_BLOCK )  + threadIdx.y ;

    if(istate < psize)
  {

    if(use_texture_lcf==0){ vecin =  vec_in[istate];
                     }else{ vecin = fetch_x(istate); };

    kets_in = sector_states[istate];
    if(thread_lane==0){
      sdata[threadIdx.y][threadIdx.x] = hipCmul(make_hipDoubleComplex(quart[istate],0.0) , vecin);
    }else{
      sdata[threadIdx.y][threadIdx.x] = make_hipDoubleComplex(0.0,0.0) ;
    };

    //////////////////////////////////////////////////////////////////////////

   if(use_texture_lcf==0){
    for(iorb=thread_lane;iorb<impnorbs ;iorb+=WARP_SIZE)
      {if(BTEST(kets_in,imporbs[iorb]-1 )>0) sdata[threadIdx.y][threadIdx.x]=hipCadd(sdata[threadIdx.y][threadIdx.x],hipCmul(Ec[iorb* impnorbs+iorb],vecin));};
    for(iorb=thread_lane;iorb<bathnorbs;iorb+=WARP_SIZE)
      {if(BTEST(kets_in,bathorbs[iorb]-1)>0) sdata[threadIdx.y][threadIdx.x]=hipCadd(sdata[threadIdx.y][threadIdx.x],hipCmul(Eb[iorb*bathnorbs+iorb],vecin));};
   }else{
    for(iorb=thread_lane;iorb<impnorbs ;iorb+=WARP_SIZE)
      {if(BTEST(kets_in,imporbs[iorb]-1 )>0) sdata[threadIdx.y][threadIdx.x]=hipCadd(sdata[threadIdx.y][threadIdx.x],hipCmul(fetchc(iorb* impnorbs+iorb),vecin));};
    for(iorb=thread_lane;iorb<bathnorbs;iorb+=WARP_SIZE)
      {if(BTEST(kets_in,bathorbs[iorb]-1)>0) sdata[threadIdx.y][threadIdx.x]=hipCadd(sdata[threadIdx.y][threadIdx.x],hipCmul(fetchb(iorb*bathnorbs+iorb),vecin));};
   }

    //////////////////////////////////////////////////////////////////////////
     for(jorb=thread_lane;jorb<impnorbs;jorb+=WARP_SIZE){
      if(BTEST(kets_in,imporbs[jorb]-1)>0){

       for(iorb=0;iorb<impnorbs;iorb++){
       if(iorb!=jorb){
       if(maskEc[jorb*impnorbs+iorb]!=0){

        kets_out=kets_in; IBCLR(kets_out,imporbs[jorb]-1);
        if(BTEST(kets_out,imporbs[iorb]-1)==0)
        {
         IBSET(kets_out,imporbs[iorb]-1);
         n1=imporbs[iorb]-1;n2=imporbs[jorb]-1; fermion_sign=1;
         if(n1<n2){ for(jj=n1+1; jj<=n2-1; jj++) { if(BTEST(kets_out,jj)>0) fermion_sign=-fermion_sign; };};
         if(n2<n1){ for(jj=n2+1; jj<=n1-1; jj++) { if(BTEST(kets_out,jj)>0) fermion_sign=-fermion_sign; };};
           if(use_texture_rank_c==0){
             jstate                            = sector_rank[kets_out]-1;}else{
             jstate                            = fetch_r(kets_out)-1;};
           if(use_texture_lcf==0){
             hoffdiag                          = Ec[jorb*impnorbs+iorb];
             hoffdiag                          = make_hipDoubleComplex(hipCreal(hoffdiag)*fermion_sign,-hipCimag(hoffdiag)*fermion_sign );
             sdata[threadIdx.y][threadIdx.x]   =hipCadd(sdata[threadIdx.y][threadIdx.x], hipCmul(hoffdiag,vec_in[jstate]));
           }else{
             hoffdiag                          = fetchc(jorb*impnorbs+iorb);
             hoffdiag                          = make_hipDoubleComplex(hipCreal(hoffdiag)*fermion_sign,-hipCimag(hoffdiag)*fermion_sign );
             sdata[threadIdx.y][threadIdx.x]   =hipCadd(sdata[threadIdx.y][threadIdx.x], hipCmul(fetch_x(jstate),hoffdiag));
           }
        }; }; }; }; }; };
    //////////////////////////////////////////////////////////////////////////

    //////////////////////////////////////////////////////////////////////////
     for(jorb=thread_lane;jorb<bathnorbs;jorb+=WARP_SIZE){
      if(BTEST(kets_in,bathorbs[jorb]-1)>0){

       for(iorb=0;iorb<bathnorbs;iorb++){
       if(iorb!=jorb){
       if(maskEb[jorb*bathnorbs+iorb]!=0){

        kets_out=kets_in; IBCLR(kets_out,bathorbs[jorb]-1);
        if(BTEST(kets_out,bathorbs[iorb]-1)==0)
        {
         IBSET(kets_out,bathorbs[iorb]-1);
         n1=bathorbs[iorb]-1;n2=bathorbs[jorb]-1; fermion_sign=1;
         if(n1<n2){ for(int jj=n1+1; jj<=n2-1; jj++) { if(BTEST(kets_out,jj)>0) fermion_sign=-fermion_sign; };};
         if(n2<n1){ for(int jj=n2+1; jj<=n1-1; jj++) { if(BTEST(kets_out,jj)>0) fermion_sign=-fermion_sign; };};
         if(use_texture_rank_c==0){
            jstate                            = sector_rank[kets_out]-1;}else{
            jstate                            = fetch_r(kets_out)-1;};
            if(use_texture_lcf==0){
             hoffdiag                          = Eb[jorb*bathnorbs+iorb];
             hoffdiag                          = make_hipDoubleComplex(hipCreal(hoffdiag)*fermion_sign,-hipCimag(hoffdiag)*fermion_sign );
             sdata[threadIdx.y][threadIdx.x]   = hipCadd(sdata[threadIdx.y][threadIdx.x],hipCmul(hoffdiag,vec_in[jstate]));
           }else{
             hoffdiag                          = fetchb(jorb*bathnorbs+iorb);
             hoffdiag                          = make_hipDoubleComplex(hipCreal(hoffdiag)*fermion_sign,-hipCimag(hoffdiag)*fermion_sign );
             sdata[threadIdx.y][threadIdx.x]   =hipCadd(sdata[threadIdx.y][threadIdx.x], hipCmul(fetch_x(jstate),hoffdiag));
           };
        }; }; }; }; }; };
    //////////////////////////////////////////////////////////////////////////

    //////////////////////////////////////////////////////////////////////////
      for(jorb=thread_lane;jorb<impnorbs;jorb+=WARP_SIZE){ 
      if(BTEST(kets_in,imporbs[jorb]-1)>0){

      for(iorb=0;iorb<bathnorbs;iorb++){if(maskVbc[jorb*bathnorbs+iorb]!=0){
        kets_out=kets_in; IBCLR(kets_out,imporbs[jorb]-1);
        if(BTEST(kets_out,bathorbs[iorb]-1)==0)
        {
         IBSET(kets_out,bathorbs[iorb]-1);
         n1=imporbs[jorb]-1;n2=bathorbs[iorb]-1; fermion_sign=1;
         if(n1<n2){ for(int jj=n1+1; jj<=n2-1; jj++) { if(BTEST(kets_out,jj)>0) fermion_sign=-fermion_sign; };};
         if(n2<n1){ for(int jj=n2+1; jj<=n1-1; jj++) { if(BTEST(kets_out,jj)>0) fermion_sign=-fermion_sign; };};
          if(use_texture_rank_c==0){
            jstate                            = sector_rank[kets_out]-1;}else{
            jstate                            = fetch_r(kets_out)-1;};
            if(use_texture_lcf==0){
             hoffdiag                          = Vbc[jorb*bathnorbs+iorb];
             hoffdiag                          = make_hipDoubleComplex(hipCreal(hoffdiag)*fermion_sign,-hipCimag(hoffdiag)*fermion_sign );
             sdata[threadIdx.y][threadIdx.x]   =hipCadd(sdata[threadIdx.y][threadIdx.x], hipCmul(hoffdiag,vec_in[jstate]));
           }else{
             hoffdiag                          = fetchbc(jorb*bathnorbs+iorb);
             hoffdiag                          = make_hipDoubleComplex(hipCreal(hoffdiag)*fermion_sign,-hipCimag(hoffdiag)*fermion_sign );
             sdata[threadIdx.y][threadIdx.x]   =hipCadd(sdata[threadIdx.y][threadIdx.x], hipCmul(fetch_x(jstate),hoffdiag));
           }
        }; }; }; }; };
    //////////////////////////////////////////////////////////////////////////

    //////////////////////////////////////////////////////////////////////////
    for(iorb=thread_lane;iorb<bathnorbs;iorb+=WARP_SIZE){ 
      if(BTEST(kets_in,bathorbs[iorb]-1)>0){

        for(jorb=0;jorb<impnorbs;jorb++){ if(maskVbc[jorb*bathnorbs+iorb]!=0){
        kets_out=kets_in; IBCLR(kets_out,bathorbs[iorb]-1);
        if(BTEST(kets_out,imporbs[jorb]-1)==0)
        {
         IBSET(kets_out,imporbs[jorb]-1);
         n1=imporbs[jorb]-1;n2=bathorbs[iorb]-1; fermion_sign=1;
         if(n1<n2){ for(int jj=n1+1; jj<=n2-1; jj++) { if(BTEST(kets_out,jj)>0) fermion_sign=-fermion_sign; };};
         if(n2<n1){ for(int jj=n2+1; jj<=n1-1; jj++) { if(BTEST(kets_out,jj)>0) fermion_sign=-fermion_sign; };};
          if(use_texture_rank_c==0){
            jstate                            = sector_rank[kets_out]-1;}else{
            jstate                            = fetch_r(kets_out)-1;};
           if(use_texture_lcf==0){
             hoffdiag                          = Vbc[jorb*bathnorbs+iorb];
             hoffdiag                          = make_hipDoubleComplex(hipCreal(hoffdiag)*fermion_sign,-hipCimag(hoffdiag)*fermion_sign );
             sdata[threadIdx.y][threadIdx.x]   =hipCadd(sdata[threadIdx.y][threadIdx.x], hipCmul(hoffdiag,vec_in[jstate]));
           }else{
             hoffdiag                          = fetchbc(jorb*bathnorbs+iorb);
             hoffdiag                          = make_hipDoubleComplex(hipCreal(hoffdiag)*fermion_sign,-hipCimag(hoffdiag)*fermion_sign );
             sdata[threadIdx.y][threadIdx.x]   =hipCadd(sdata[threadIdx.y][threadIdx.x], hipCmul(fetch_x(jstate),hoffdiag));
           }
        }; }; }; }; };
    //////////////////////////////////////////////////////////////////////////

        if (thread_lane < 16) { sdata[threadIdx.y][threadIdx.x] = hipCadd(sdata[threadIdx.y][threadIdx.x], sdata[threadIdx.y][threadIdx.x + 16] ); };
        if (thread_lane <  8) { sdata[threadIdx.y][threadIdx.x] = hipCadd(sdata[threadIdx.y][threadIdx.x], sdata[threadIdx.y][threadIdx.x +  8] ); };
        if (thread_lane <  4) { sdata[threadIdx.y][threadIdx.x] = hipCadd(sdata[threadIdx.y][threadIdx.x], sdata[threadIdx.y][threadIdx.x +  4] ); };
        if (thread_lane <  2) { sdata[threadIdx.y][threadIdx.x] = hipCadd(sdata[threadIdx.y][threadIdx.x], sdata[threadIdx.y][threadIdx.x +  2] ); };
        if (thread_lane <  1) { sdata[threadIdx.y][threadIdx.x] = hipCadd(sdata[threadIdx.y][threadIdx.x], sdata[threadIdx.y][threadIdx.x +  1] ); };
        if (thread_lane == 0)   vec_out[istate] = sdata[threadIdx.y][threadIdx.x];

    //////////////////////////////////////////////////////////////////////////

  };
  };
}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

 void one_step_lanczos_complex_fly_cuda(int blocksize, int Niter, int size, int iter, hipDoubleComplex *vec_tmp_gpu, hipDoubleComplex *vec_in_gpu, 
     hipDoubleComplex *vec_out_gpu, hipDoubleComplex *vec_tmp_gpu_pointer, hipDoubleComplex *vec_in_gpu_pointer, hipDoubleComplex *vec_out_gpu_pointer,
     double *quart_gpu_pointer , double *diag, double *subdiag , hipDoubleComplex* Eb, hipDoubleComplex* Ec, hipDoubleComplex* Vbc,
     int* sector_states, int* sector_ranks,int norbs, int bathnorbs,int impnorbs,int* imporbs, int* bathorbs, int* maskEb, int* maskEc, int* maskVbc )

{
   int verbose=0; int psize=size; if(verbose==1) printf( " define blocksize \n ");
   double normv; double *normv_ker; hipMalloc((void**)&normv_ker,sizeof(double)); double *normv_loc; normv_loc = &normv; hipDoubleComplex coef;

   int nb=(size -size % blocksize)/blocksize + 1; int ngrid=nb/MAX_BLOCK; if(ngrid>0) nb=MAX_BLOCK;
   dim3 bl(1,nb),th(WARP_SIZE,blocksize);
   if(verbose==1) printf( " --------------- \n Nblock=%d Ngrid=%d \n ----------------- \n ",nb,ngrid);

   int nb2 =(size-size % 256 )/256 + 1; 
   int ngrid2=nb2/MAX_BLOCK; if(ngrid2>0) nb2=MAX_BLOCK;

   norm_vec_ker_complex<<<1,512>>>(size,vec_in_gpu_pointer,normv_ker); hipEventSynchronize(0); hipDeviceSynchronize();
   normalize_vec_ker_complex<<<nb2,256>>>(ngrid2, size, vec_in_gpu_pointer, normv_ker);
 
   if(verbose==1) printf( " one step Lanczos, norm vecin=%f, size of problem=%d \n " , normv, size); 
   if(verbose==1) printf( " use texture optimization = %d " , use_texture_lcf );

   Hmult_ker_complex_fly<<<bl,th>>>(ngrid,blocksize,psize,vec_out_gpu_pointer,vec_in_gpu_pointer,quart_gpu_pointer,Eb,Ec,Vbc,sector_states,
                                   sector_ranks,norbs,bathnorbs,impnorbs,imporbs,bathorbs,maskEb,maskEc,maskVbc,use_texture_lcf,
                                   use_texture_rank_c); 
   hipEventSynchronize(0); hipDeviceSynchronize();

  if(iter>0){ coef=make_hipDoubleComplex(-subdiag[iter],0.0); cublasZaxpy_no_device(size,coef,vec_tmp_gpu,1,vec_out_gpu,1); }; hipEventSynchronize(0); hipDeviceSynchronize();

  if(use_texture_lcf==0){
   hipMemcpy(vec_tmp_gpu,vec_in_gpu,size*sizeof(hipDoubleComplex),hipMemcpyHostToHost);hipEventSynchronize(0);hipDeviceSynchronize();
  }else{
   hipMemcpy(vec_tmp_gpu,vec_in_gpu_pointer,size*sizeof(hipDoubleComplex),hipMemcpyDeviceToHost);hipEventSynchronize(0);hipDeviceSynchronize();
  }

   real_scal_vec_ker_complex<<<1,512>>> (size,vec_out_gpu_pointer,vec_in_gpu_pointer,normv_ker);
   hipMemcpy(normv_loc,normv_ker,sizeof(double),hipMemcpyDeviceToHost);
   diag[iter]=*normv_loc ;

   hipEventSynchronize(0);hipDeviceSynchronize();
   coef=make_hipDoubleComplex(-diag[iter],0.0);
   cublasZaxpy_no_device(size,coef,vec_tmp_gpu,1,vec_out_gpu,1); hipEventSynchronize(0); hipDeviceSynchronize();

   normv = sqrt(hipCabs(cublasZdotu_no_device(size,vec_out_gpu,1,vec_out_gpu,1))); hipEventSynchronize(0); hipDeviceSynchronize();

   if(iter<Niter-1) subdiag[iter+1]=normv;

  if(use_texture_lcf==0){
   hipMemcpy( vec_in_gpu, vec_out_gpu, size*sizeof(hipDoubleComplex), hipMemcpyHostToHost); hipEventSynchronize(0); hipDeviceSynchronize();
  }else{
   hipMemcpy( vec_in_gpu_pointer, vec_out_gpu, size*sizeof(hipDoubleComplex), hipMemcpyHostToDevice); hipEventSynchronize(0); hipDeviceSynchronize();
  }

  hipFree(normv_ker);


}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

extern "C" void lanczos_complex_fly_cuda_(int *dimen_in, int *pblocksize, int* norbs_in, int *Niter_lanczos_, int* ntot, double* quart, 
      double* diag, double* subdiag, hipDoubleComplex* Eb_in, hipDoubleComplex* Ec_in, hipDoubleComplex* Vbc_in, int* sector_states_in, int* sector_ranks_in 
     ,int* bathnorbs_in,int* impnorbs_in,int* imporbs_in, int* bathorbs_in, int* maskEb_in, int* maskEc_in, int* maskVbc_in)
{

  if(*dimen_in<max_rank){ use_texture_rank_c = 1;}else{use_texture_rank_c = 0;};

 //---------------------------------------------------------------------------------------//
     int verbose=0;
 //---------------------------------------------------------------------------------------//

     int Niter_lanczos=*Niter_lanczos_; int norbs=*norbs_in; int dimen=*dimen_in;
     
     if(verbose==1) printf(" start Lanczos Real on GPU \n" );

     int size      = *ntot; int blocksize = *pblocksize;
     int bathnorbs = *bathnorbs_in; int impnorbs=*impnorbs_in;

     hipDoubleComplex  *vec_in_gpu,*vec_out_gpu,*vec_tmp_gpu;
     double           *quart_gpu;
     hipDoubleComplex  *vec_in_gpu_pointer,*vec_out_gpu_pointer,*vec_tmp_gpu_pointer;
     double           *quart_gpu_pointer;
     hipDoubleComplex  *Eb,*Ec,*Vbc,*Eb_p,*Ec_p,*Vbc_p;

     int *sector_states_p,*sector_ranks_p,*imporbs_p,*bathorbs_p,*maskEb_p,*maskEc_p,*maskVbc_p;
     int *sector_states,*sector_ranks,*imporbs,*bathorbs,*maskEb,*maskEc,*maskVbc;

     if(verbose==1) printf(" GPU get eigenvalues, size of Lanczos vectors = %d ", size);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

       hipSetDevice(0); hipSetDeviceFlags( hipDeviceMapHost ); hipEventSynchronize(0);

       if(use_texture_lcf==1){
         hipMalloc((void**)&vec_in_gpu_pointer,sizeof(hipDoubleComplex)*size);
         hipMalloc((void**)&Eb_p  , bathnorbs*bathnorbs*sizeof(hipDoubleComplex) );
         hipMalloc((void**)&Ec_p  ,  impnorbs*impnorbs*sizeof(hipDoubleComplex) );
         hipMalloc((void**)&Vbc_p , bathnorbs*impnorbs*sizeof(hipDoubleComplex) );
         hipMemcpy(Eb_p  ,  Eb_in,  bathnorbs*bathnorbs*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
         hipMemcpy(Ec_p  ,  Ec_in,   impnorbs*impnorbs*sizeof(hipDoubleComplex),  hipMemcpyHostToDevice);
         hipMemcpy(Vbc_p , Vbc_in,  bathnorbs*impnorbs*sizeof(hipDoubleComplex),  hipMemcpyHostToDevice);
         bind_x(vec_in_gpu_pointer,size);
         bind_x_(Eb_p,Ec_p,Vbc_p);
       }else{
         hipHostAlloc((void**)&vec_in_gpu , size*sizeof(hipDoubleComplex)  , hipHostMallocMapped | hipHostMallocPortable );
         hipHostGetDevicePointer((void**)   &vec_in_gpu_pointer  , vec_in_gpu , 0 );
         hipHostAlloc((void**)&Eb         , bathnorbs*bathnorbs*sizeof(hipDoubleComplex), hipHostMallocMapped | hipHostMallocPortable );
         hipHostAlloc((void**)&Ec         , impnorbs*impnorbs*sizeof(hipDoubleComplex)  , hipHostMallocMapped | hipHostMallocPortable );
         hipHostAlloc((void**)&Vbc        , bathnorbs*impnorbs*sizeof(hipDoubleComplex) , hipHostMallocMapped | hipHostMallocPortable );
         hipMemcpy(Eb,  Eb_in,  bathnorbs*bathnorbs*sizeof(hipDoubleComplex), hipMemcpyHostToHost);
         hipMemcpy(Ec,  Ec_in,  impnorbs*impnorbs*sizeof(hipDoubleComplex),   hipMemcpyHostToHost);
         hipMemcpy(Vbc, Vbc_in, bathnorbs*impnorbs*sizeof(hipDoubleComplex),  hipMemcpyHostToHost);
         hipHostGetDevicePointer((void**)   &Eb_p  , Eb  , 0 );
         hipHostGetDevicePointer((void**)   &Ec_p  , Ec  , 0 );
         hipHostGetDevicePointer((void**)   &Vbc_p , Vbc , 0 );
       }

      if(verbose==1) printf( " bind sector_ranks to texture if necessary \n ");

      if(use_texture_rank_c==0){
       hipHostAlloc((void**)&sector_ranks , dimen*sizeof(int), hipHostMallocMapped | hipHostMallocPortable );
       hipMemcpy(sector_ranks , sector_ranks_in , dimen*sizeof(int), hipMemcpyHostToHost);
       hipHostGetDevicePointer((void**)  &sector_ranks_p      , sector_ranks   , 0 );
      }else{
       hipMalloc((void**)&sector_ranks_p , dimen*sizeof(int) );
       hipMemcpy(         sector_ranks_p , sector_ranks_in , dimen*sizeof(int), hipMemcpyHostToDevice);
       bind_x_r(sector_ranks_p);
      };

       hipHostAlloc((void**)&vec_tmp_gpu   , size*sizeof(hipDoubleComplex),       hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&vec_out_gpu   , size*sizeof(hipDoubleComplex),       hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&quart_gpu     , size*sizeof(double),                hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&sector_states , size*sizeof(int),                   hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&imporbs       , impnorbs*sizeof(int),               hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&bathorbs      , bathnorbs*sizeof(int),              hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&maskEb        , bathnorbs*bathnorbs*sizeof(int),    hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&maskEc        , impnorbs*impnorbs*sizeof(int),      hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&maskVbc       , bathnorbs*impnorbs*sizeof(int),     hipHostMallocMapped | hipHostMallocPortable );

       hipEventSynchronize(0);
       hipMemcpy(quart_gpu, quart , size*sizeof(double), hipMemcpyHostToHost);
       hipMemcpy(sector_states, sector_states_in, size*sizeof(int), hipMemcpyHostToHost);
       hipMemcpy(imporbs, imporbs_in , impnorbs*sizeof(int),  hipMemcpyHostToHost);
       hipMemcpy(bathorbs,bathorbs_in, bathnorbs*sizeof(int), hipMemcpyHostToHost);
       hipMemcpy(maskEb  ,maskEb_in , bathnorbs *bathnorbs  *sizeof(int), hipMemcpyHostToHost);
       hipMemcpy(maskEc  ,maskEc_in , impnorbs  *impnorbs   *sizeof(int), hipMemcpyHostToHost);
       hipMemcpy(maskVbc ,maskVbc_in, impnorbs  *bathnorbs  *sizeof(int), hipMemcpyHostToHost);

       hipEventSynchronize(0);
       hipHostGetDevicePointer((void**)   &vec_tmp_gpu_pointer , vec_tmp_gpu    , 0 );
       hipHostGetDevicePointer((void**)   &vec_out_gpu_pointer , vec_out_gpu    , 0 );
       hipHostGetDevicePointer((void**)   &quart_gpu_pointer   , quart_gpu      , 0 );
       hipHostGetDevicePointer((void**)   &sector_states_p     , sector_states  , 0 );
       hipHostGetDevicePointer((void**)   &imporbs_p           , imporbs        , 0 );
       hipHostGetDevicePointer((void**)   &bathorbs_p          , bathorbs       , 0 );
       hipHostGetDevicePointer((void**)   &maskEb_p            , maskEb         , 0 );
       hipHostGetDevicePointer((void**)   &maskEc_p            , maskEc         , 0 );
       hipHostGetDevicePointer((void**)   &maskVbc_p           , maskVbc        , 0 );
       hipEventSynchronize(0);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

  if(verbose==1) printf( " initialized, now run actual Lanczos \n " );

 if(use_texture_lcf==0){
  for(int i=0;i<size;i++) vec_in_gpu[i]=make_hipDoubleComplex(1.0,0.0); hipDeviceSynchronize();
 }else{
   for(int i=0;i<size;i++) vec_out_gpu[i]=make_hipDoubleComplex(1.0,0.0); hipDeviceSynchronize();
   hipMemcpy(vec_in_gpu_pointer,vec_out_gpu,size*sizeof(hipDoubleComplex),hipMemcpyHostToDevice);
 }

  for(int iter=0;iter<Niter_lanczos;iter++){

    if(verbose==1) printf( " Lanczos iteration %d / %d \n", iter,Niter_lanczos);

    one_step_lanczos_complex_fly_cuda(blocksize,Niter_lanczos,size,iter,vec_tmp_gpu,vec_in_gpu,
                    vec_out_gpu,vec_tmp_gpu_pointer,vec_in_gpu_pointer,vec_out_gpu_pointer,
                    quart_gpu_pointer,diag,subdiag,Eb_p,Ec_p,Vbc_p,sector_states_p,sector_ranks_p,norbs,
                    bathnorbs,impnorbs,imporbs_p,bathorbs_p,maskEb_p,maskEc_p,maskVbc_p);
  };

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

   hipEventSynchronize(0); 
   if(use_texture_lcf==0)
   {
    hipHostFree(vec_in_gpu);hipHostFree(Eb);hipHostFree(Ec);hipHostFree(Vbc);
   }else{
    unbind_x(); unbind_x_(); hipFree(vec_in_gpu_pointer); hipFree(Eb_p);hipFree(Ec_p);hipFree(Vbc_p);
   }

   if(use_texture_rank_c==0){ hipHostFree(sector_ranks);}else{unbind_x_r(); hipFree(sector_ranks_p);};

   hipHostFree(vec_tmp_gpu);    hipHostFree(vec_out_gpu); 
   hipHostFree(sector_states);  hipHostFree(quart_gpu);
   hipHostFree(bathorbs);       hipHostFree(imporbs);        hipHostFree(maskVbc);
   hipHostFree(maskEb);         hipHostFree(maskEc);
 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

extern "C" void lanczos_complex_fly_dynamic_cuda_(int *dimen_in, int *pblocksize, int* norbs_in, int *Niter_lanczos_, int* ntot, double* quart, 
        double* diag, double* subdiag, hipDoubleComplex* Eb_in, hipDoubleComplex* Ec_in, hipDoubleComplex* Vbc_in, int* sector_states_in, int* sector_ranks_in 
       ,int* bathnorbs_in,int* impnorbs_in,int* imporbs_in, int* bathorbs_in, int* maskEb_in, int* maskEc_in, int* maskVbc_in, hipDoubleComplex* vecinit)
{

  if(*dimen_in<max_rank){ use_texture_rank_c = 1;}else{use_texture_rank_c = 0;};

 //---------------------------------------------------------------------------------------//
     int verbose=0;
 //---------------------------------------------------------------------------------------//

     int Niter_lanczos=*Niter_lanczos_; int norbs=*norbs_in; int dimen=*dimen_in;
     
     if(verbose==1) printf(" start Lanczos Real on GPU \n" );

     int size = *ntot; int blocksize = *pblocksize; int bathnorbs=*bathnorbs_in; int impnorbs=*impnorbs_in;

     hipDoubleComplex *vec_in_gpu,*vec_out_gpu,*vec_tmp_gpu;
     double          *quart_gpu;
     hipDoubleComplex *vec_in_gpu_pointer,*vec_out_gpu_pointer,*vec_tmp_gpu_pointer;
     double          *quart_gpu_pointer;
     hipDoubleComplex *Eb,*Ec,*Vbc,*Eb_p,*Ec_p,*Vbc_p;

     int *sector_states_p,*sector_ranks_p,*imporbs_p,*bathorbs_p,*maskEb_p,*maskEc_p,*maskVbc_p;
     int *sector_states,*sector_ranks,*imporbs,*bathorbs,*maskEb,*maskEc,*maskVbc;

     if(verbose==1) printf(" GPU get eigenvalues, size of Lanczos vectors = %d ", size);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

      hipSetDevice(0); hipSetDeviceFlags( hipDeviceMapHost ); hipEventSynchronize(0);

      if(use_texture_lcf==1){
         hipMalloc((void**)&vec_in_gpu_pointer,sizeof(hipDoubleComplex)*size);
         hipMalloc((void**)&Eb_p  , bathnorbs*bathnorbs*sizeof(hipDoubleComplex) );
         hipMalloc((void**)&Ec_p  , impnorbs*impnorbs*sizeof(hipDoubleComplex) );
         hipMalloc((void**)&Vbc_p , bathnorbs*impnorbs*sizeof(hipDoubleComplex) );
         hipMemcpy(Eb_p,  Eb_in,  bathnorbs*bathnorbs*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
         hipMemcpy(Ec_p,  Ec_in,  impnorbs*impnorbs*sizeof(hipDoubleComplex),   hipMemcpyHostToDevice);
         hipMemcpy(Vbc_p, Vbc_in, bathnorbs*impnorbs*sizeof(hipDoubleComplex),  hipMemcpyHostToDevice);
         bind_x(vec_in_gpu_pointer,size); bind_x_(Eb_p,Ec_p,Vbc_p);
       }else{
         hipHostAlloc((void**)&vec_in_gpu    , size*sizeof(hipDoubleComplex),        hipHostMallocMapped | hipHostMallocPortable );
         hipHostGetDevicePointer((void**)      &vec_in_gpu_pointer  ,  vec_in_gpu    , 0 );
         hipHostAlloc((void**)&Eb            , bathnorbs*bathnorbs*sizeof(hipDoubleComplex), hipHostMallocMapped | hipHostMallocPortable );
         hipHostAlloc((void**)&Ec            , impnorbs*impnorbs*sizeof(hipDoubleComplex),   hipHostMallocMapped | hipHostMallocPortable );
         hipHostAlloc((void**)&Vbc           , bathnorbs*impnorbs*sizeof(hipDoubleComplex),  hipHostMallocMapped | hipHostMallocPortable );
         hipMemcpy(Eb,  Eb_in,  bathnorbs*bathnorbs*sizeof(hipDoubleComplex), hipMemcpyHostToHost);
         hipMemcpy(Ec,  Ec_in,  impnorbs*impnorbs*sizeof(hipDoubleComplex),   hipMemcpyHostToHost);
         hipMemcpy(Vbc, Vbc_in, bathnorbs*impnorbs*sizeof(hipDoubleComplex),  hipMemcpyHostToHost);
         hipHostGetDevicePointer((void**) &Eb_p  , Eb  , 0 );
         hipHostGetDevicePointer((void**) &Ec_p  , Ec  , 0 );
         hipHostGetDevicePointer((void**) &Vbc_p , Vbc , 0 );
      }

      if(use_texture_rank_c==0){
       hipHostAlloc((void**)&sector_ranks  , dimen*sizeof(int), hipHostMallocMapped | hipHostMallocPortable );
       hipMemcpy(sector_ranks , sector_ranks_in , dimen*sizeof(int), hipMemcpyHostToHost);
       hipHostGetDevicePointer((void**)   &sector_ranks_p      , sector_ranks   , 0 );
      }else{
       hipMalloc((void**)&sector_ranks_p  , dimen*sizeof(int) );
       hipMemcpy(         sector_ranks_p , sector_ranks_in , dimen*sizeof(int), hipMemcpyHostToDevice);
       bind_x_r(sector_ranks_p);
      };

       hipHostAlloc((void**)&vec_tmp_gpu   , size*sizeof(hipDoubleComplex),       hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&vec_out_gpu   , size*sizeof(hipDoubleComplex),       hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&quart_gpu     , size*sizeof(double),                hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&sector_states , size*sizeof(int),                   hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&imporbs       , impnorbs*sizeof(int),               hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&bathorbs      , bathnorbs*sizeof(int),              hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&maskEb        , bathnorbs*bathnorbs*sizeof(int),   hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&maskEc        , impnorbs*impnorbs*sizeof(int),     hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&maskVbc       , bathnorbs*impnorbs*sizeof(int),    hipHostMallocMapped | hipHostMallocPortable );

       hipEventSynchronize(0);
       hipMemcpy(quart_gpu,   quart        , size*sizeof(double), hipMemcpyHostToHost);
       hipMemcpy(sector_states, sector_states_in, size*sizeof(int), hipMemcpyHostToHost);
       hipMemcpy(imporbs, imporbs_in , impnorbs*sizeof(int),  hipMemcpyHostToHost);
       hipMemcpy(bathorbs,bathorbs_in, bathnorbs*sizeof(int), hipMemcpyHostToHost);
       hipMemcpy(maskEb  ,maskEb_in , bathnorbs *bathnorbs  *sizeof(int), hipMemcpyHostToHost);
       hipMemcpy(maskEc  ,maskEc_in , impnorbs  *impnorbs   *sizeof(int), hipMemcpyHostToHost);
       hipMemcpy(maskVbc ,maskVbc_in, impnorbs  *bathnorbs  *sizeof(int), hipMemcpyHostToHost);
       hipEventSynchronize(0);

       hipHostGetDevicePointer((void**)   &vec_tmp_gpu_pointer , vec_tmp_gpu    , 0 );
       hipHostGetDevicePointer((void**)   &vec_out_gpu_pointer , vec_out_gpu    , 0 );
       hipHostGetDevicePointer((void**)   &quart_gpu_pointer   , quart_gpu      , 0 );
       hipHostGetDevicePointer((void**)   &sector_states_p     , sector_states  , 0 );
       hipHostGetDevicePointer((void**)   &imporbs_p           , imporbs        , 0 );
       hipHostGetDevicePointer((void**)   &bathorbs_p          , bathorbs       , 0 );
       hipHostGetDevicePointer((void**)   &maskEb_p            , maskEb         , 0 );
       hipHostGetDevicePointer((void**)   &maskEc_p            , maskEc         , 0 );
       hipHostGetDevicePointer((void**)   &maskVbc_p           , maskVbc        , 0 );
       hipEventSynchronize(0);

       hipDeviceSynchronize(); hipEventSynchronize(0); 
       if(use_texture_lcf==0){
        hipMemcpy(vec_in_gpu,vecinit,size*sizeof(hipDoubleComplex),hipMemcpyHostToHost);
       }else{
        hipMemcpy(vec_out_gpu,vecinit,size*sizeof(hipDoubleComplex),hipMemcpyHostToHost);
        hipMemcpy(vec_in_gpu_pointer,vec_out_gpu,size*sizeof(hipDoubleComplex),hipMemcpyHostToDevice);
       }
       hipDeviceSynchronize(); hipEventSynchronize(0);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

  if(verbose==1) printf( " initialized, now run actual Lanczos \n " );

  for(int iter=0;iter<Niter_lanczos;iter++){

    if(verbose==1) printf( " Lanczos iteration %d / %d \n", iter,Niter_lanczos);

    one_step_lanczos_complex_fly_cuda(blocksize,Niter_lanczos,size,iter,vec_tmp_gpu,vec_in_gpu,
                    vec_out_gpu,vec_tmp_gpu_pointer,vec_in_gpu_pointer,vec_out_gpu_pointer,
                    quart_gpu_pointer,diag,subdiag,Eb_p,Ec_p,Vbc_p,sector_states_p,sector_ranks_p,norbs,
                    bathnorbs,impnorbs,imporbs_p,bathorbs_p,maskEb_p,maskEc_p,maskVbc_p);
  };

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//
   hipEventSynchronize(0); 
   if(use_texture_lcf==0)
   {
     hipHostFree(vec_in_gpu);hipHostFree(Eb);hipHostFree(Ec);hipHostFree(Vbc);
   }else{
     unbind_x(); unbind_x_();
     hipFree(vec_in_gpu_pointer); hipFree(Eb_p);hipFree(Ec_p);hipFree(Vbc_p);
   }
   if(use_texture_rank_c==0){ hipHostFree(sector_ranks);}else{unbind_x_r(); hipFree(sector_ranks_p);};

   hipHostFree(vec_tmp_gpu);   hipHostFree(vec_out_gpu);   hipHostFree(quart_gpu);
   hipHostFree(sector_states); hipHostFree(bathorbs);      hipHostFree(imporbs);
   hipHostFree(maskEb);        hipHostFree(maskEc);        hipHostFree(maskVbc);
 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

extern "C" void lanczos_complex_fly_gs_cuda_(int *dimen_in, int *pblocksize, int* norbs_in, int *Niter_lanczos_, int* ntot, double* quart, 
  hipDoubleComplex* Eb_in, hipDoubleComplex* Ec_in, hipDoubleComplex* Vbc_in, int* sector_states_in, int* sector_ranks_in 
 ,int* bathnorbs_in,int* impnorbs_in,int* imporbs_in, int* bathorbs_in, int* maskEb_in, int* maskEc_in, int* maskVbc_in,double *vecp,hipDoubleComplex *GS)
{

  if(*dimen_in<max_rank){ use_texture_rank_c = 1;}else{use_texture_rank_c = 0;};

 //---------------------------------------------------------------------------------------//
     int verbose=0;
 //---------------------------------------------------------------------------------------//

     int Niter_lanczos=*Niter_lanczos_; int norbs=*norbs_in; int dimen=*dimen_in;
     
     if(verbose==1) printf(" start Lanczos Real on GPU \n" );

     int size = *ntot; int blocksize = *pblocksize;
     int bathnorbs=*bathnorbs_in; int impnorbs=*impnorbs_in;

     hipDoubleComplex    *vec_in_gpu,*vec_out_gpu,*vec_tmp_gpu;
     double             *quart_gpu;
     hipDoubleComplex    *vec_in_gpu_pointer,*vec_out_gpu_pointer,*vec_tmp_gpu_pointer;
     double             *quart_gpu_pointer;
     hipDoubleComplex    *Eb,*Ec,*Vbc,*Eb_p,*Ec_p,*Vbc_p;
     int                *sector_states_p,*sector_ranks_p,*imporbs_p,*bathorbs_p;
     int                *maskEb_p,*maskEc_p,*maskVbc_p,*sector_states,*sector_ranks,*imporbs,*bathorbs;
     int                *maskEb,*maskEc,*maskVbc;
     hipDoubleComplex    *GS_gpu,*GS_gpu_pointer;

     double diag[Niter_lanczos], subdiag[Niter_lanczos];

     if(verbose==1) printf(" GPU get eigenvalues, size of Lanczos vectors = %d ", size);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

       hipSetDevice(0); hipSetDeviceFlags( hipDeviceMapHost ); hipEventSynchronize(0);
       if(use_texture_lcf==1){
         hipMalloc((void**)&vec_in_gpu_pointer,sizeof(hipDoubleComplex)*size);
         hipMalloc((void**)&Eb_p  , bathnorbs*bathnorbs*sizeof(hipDoubleComplex) );
         hipMalloc((void**)&Ec_p  , impnorbs*impnorbs*sizeof(hipDoubleComplex) );
         hipMalloc((void**)&Vbc_p , bathnorbs*impnorbs*sizeof(hipDoubleComplex) );
         hipMemcpy(Eb_p,  Eb_in,  bathnorbs*bathnorbs*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
         hipMemcpy(Ec_p,  Ec_in,  impnorbs*impnorbs*sizeof(hipDoubleComplex),   hipMemcpyHostToDevice);
         hipMemcpy(Vbc_p, Vbc_in, bathnorbs*impnorbs*sizeof(hipDoubleComplex),  hipMemcpyHostToDevice);
         bind_x(vec_in_gpu_pointer,size);
         bind_x_(Eb_p,Ec_p,Vbc_p);
       }else{
         hipHostAlloc((void**)&vec_in_gpu    , size*sizeof(hipDoubleComplex),        hipHostMallocMapped | hipHostMallocPortable );
         hipHostGetDevicePointer((void**)      &vec_in_gpu_pointer  ,  vec_in_gpu    , 0 );
         hipHostAlloc((void**)&Eb            , bathnorbs*bathnorbs*sizeof(hipDoubleComplex), hipHostMallocMapped | hipHostMallocPortable );
         hipHostAlloc((void**)&Ec            , impnorbs*impnorbs*sizeof(hipDoubleComplex),   hipHostMallocMapped | hipHostMallocPortable );
         hipHostAlloc((void**)&Vbc           , bathnorbs*impnorbs*sizeof(hipDoubleComplex),  hipHostMallocMapped | hipHostMallocPortable );
         hipMemcpy(Eb,  Eb_in,  bathnorbs*bathnorbs*sizeof(hipDoubleComplex), hipMemcpyHostToHost);
         hipMemcpy(Ec,  Ec_in,  impnorbs*impnorbs*sizeof(hipDoubleComplex),   hipMemcpyHostToHost);
         hipMemcpy(Vbc, Vbc_in, bathnorbs*impnorbs*sizeof(hipDoubleComplex),  hipMemcpyHostToHost);
         hipHostGetDevicePointer((void**)  &Eb_p   , Eb  , 0 );
         hipHostGetDevicePointer((void**)  &Ec_p   , Ec  , 0 );
         hipHostGetDevicePointer((void**)  &Vbc_p  , Vbc , 0 );
       }

       if(use_texture_rank_c==0){
         hipHostAlloc((void**)&sector_ranks  , dimen*sizeof(int), hipHostMallocMapped | hipHostMallocPortable );
         hipMemcpy(sector_ranks , sector_ranks_in , dimen*sizeof(int), hipMemcpyHostToHost);
         hipHostGetDevicePointer((void**)   &sector_ranks_p      , sector_ranks   , 0 );
       }else{
         hipMalloc((void**)&sector_ranks_p  , dimen*sizeof(int) );
         hipMemcpy(         sector_ranks_p , sector_ranks_in , dimen*sizeof(int), hipMemcpyHostToDevice);
         bind_x_r(sector_ranks_p);
       };

       hipHostAlloc((void**)&vec_tmp_gpu   , size*sizeof(hipDoubleComplex),     hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&vec_out_gpu   , size*sizeof(hipDoubleComplex),     hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&quart_gpu     , size*sizeof(double),              hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&sector_states , size*sizeof(int),                 hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&imporbs       , impnorbs*sizeof(int),             hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&bathorbs      , bathnorbs*sizeof(int),            hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&maskEb        , bathnorbs*bathnorbs*sizeof(int),  hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&maskEc        , impnorbs*impnorbs*sizeof(int),    hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&maskVbc       , bathnorbs*impnorbs*sizeof(int),   hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&GS_gpu        , size*sizeof(hipDoubleComplex),     hipHostMallocMapped | hipHostMallocPortable );

       hipEventSynchronize(0);
       hipMemcpy(quart_gpu,   quart        , size*sizeof(double), hipMemcpyHostToHost);
       hipMemcpy(sector_states, sector_states_in, size*sizeof(int), hipMemcpyHostToHost);
       hipMemcpy(imporbs, imporbs_in , impnorbs*sizeof(int),  hipMemcpyHostToHost);
       hipMemcpy(bathorbs,bathorbs_in, bathnorbs*sizeof(int), hipMemcpyHostToHost);
       hipMemcpy(maskEb  ,maskEb_in , bathnorbs *bathnorbs  *sizeof(int), hipMemcpyHostToHost);
       hipMemcpy(maskEc  ,maskEc_in , impnorbs  *impnorbs   *sizeof(int), hipMemcpyHostToHost);
       hipMemcpy(maskVbc ,maskVbc_in, impnorbs  *bathnorbs  *sizeof(int), hipMemcpyHostToHost);

       hipEventSynchronize(0);
       hipHostGetDevicePointer((void**)   &GS_gpu_pointer ,      GS_gpu        , 0 );
       hipHostGetDevicePointer((void**)   &vec_tmp_gpu_pointer , vec_tmp_gpu   , 0 );
       hipHostGetDevicePointer((void**)   &vec_out_gpu_pointer , vec_out_gpu   , 0 );
       hipHostGetDevicePointer((void**)   &quart_gpu_pointer   , quart_gpu     , 0 );
       hipHostGetDevicePointer((void**)   &sector_states_p     , sector_states , 0 );
       hipHostGetDevicePointer((void**)   &imporbs_p           , imporbs       , 0 );
       hipHostGetDevicePointer((void**)   &bathorbs_p          , bathorbs      , 0 );
       hipHostGetDevicePointer((void**)   &maskEb_p            , maskEb        , 0 );
       hipHostGetDevicePointer((void**)   &maskEc_p            , maskEc        , 0 );
       hipHostGetDevicePointer((void**)   &maskVbc_p           , maskVbc       , 0 );
       hipEventSynchronize(0);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

   memset ((void **)GS_gpu, 0, size*sizeof(hipDoubleComplex));

    if(use_texture_lcf==0){
     for(int i=0;i<size;i++) vec_in_gpu[i]=make_hipDoubleComplex(1.0,0.0); hipDeviceSynchronize();
    }else{
     for(int i=0;i<size;i++) vec_out_gpu[i]=make_hipDoubleComplex(1.0,0.0); hipDeviceSynchronize();
     hipMemcpy(vec_in_gpu_pointer,vec_out_gpu,size*sizeof(hipDoubleComplex),hipMemcpyHostToDevice);
    }

    double *normv_ker; double normv; double *normv_loc; hipMalloc((void**)&normv_ker,sizeof(double)); hipDoubleComplex coef;

 if(use_texture_lcf==0){
   coef = make_hipDoubleComplex(1.0/sqrt(hipCabs(cublasZdotu_no_device(size,vec_in_gpu,1,vec_in_gpu,1)))*vecp[0],0); hipDeviceSynchronize();  hipEventSynchronize(0);
   cublasZaxpy_no_device(size,coef,vec_in_gpu,1,GS_gpu,1); hipDeviceSynchronize(); hipEventSynchronize(0);hipDeviceSynchronize();
 }else{
   norm_vec_ker_complex<<<1,512>>>(size,vec_in_gpu_pointer,normv_ker); hipEventSynchronize(0); hipDeviceSynchronize();
   normv_loc=&normv; hipMemcpy(normv_loc,normv_ker,sizeof(double),hipMemcpyDeviceToHost);
   hipMemcpy(vec_out_gpu,vec_in_gpu,size*sizeof(hipDoubleComplex),hipMemcpyDeviceToHost);
   coef=make_hipDoubleComplex(vecp[0]/normv,0.0);
   cublasZaxpy_no_device(size,coef,vec_out_gpu,1,GS_gpu,1); hipDeviceSynchronize(); hipEventSynchronize(0);hipDeviceSynchronize();
 };

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

  if(verbose==1) printf( " first step ... \n ");

    one_step_lanczos_complex_fly_cuda(blocksize,Niter_lanczos,size,0,vec_tmp_gpu,vec_in_gpu,
                    vec_out_gpu,vec_tmp_gpu_pointer,vec_in_gpu_pointer,vec_out_gpu_pointer,
                    quart_gpu_pointer,diag,subdiag,Eb_p,Ec_p,Vbc_p,sector_states_p,sector_ranks_p,norbs,
                    bathnorbs,impnorbs,imporbs_p,bathorbs_p,maskEb_p,maskEc_p,maskVbc_p);

  for(int iter=1;iter<Niter_lanczos-1;iter++){

   coef = make_hipDoubleComplex(1.0/sqrt(hipCabs(cublasZdotu_no_device(size,vec_out_gpu,1,vec_out_gpu,1))),0.0);
   cublasZscal_no_device (size,coef, vec_out_gpu,1);  hipEventSynchronize(0);

   coef = make_hipDoubleComplex(vecp[iter],0);
   cublasZaxpy_no_device(size,coef,vec_out_gpu,1,GS_gpu,1); hipDeviceSynchronize(); hipEventSynchronize(0);

   one_step_lanczos_complex_fly_cuda(blocksize,Niter_lanczos,size,iter,vec_tmp_gpu,vec_in_gpu,
                    vec_out_gpu,vec_tmp_gpu_pointer,vec_in_gpu_pointer,vec_out_gpu_pointer,
                    quart_gpu_pointer,diag,subdiag,Eb_p,Ec_p,Vbc_p,sector_states_p,sector_ranks_p,norbs,
                    bathnorbs,impnorbs,imporbs_p,bathorbs_p,maskEb_p,maskEc_p,maskVbc_p);

  };

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

   if(verbose==1) printf("done...\n");

   coef=make_hipDoubleComplex(1.0/sqrt(hipCabs(cublasZdotu_no_device(size,vec_out_gpu,1,vec_out_gpu,1))),0.0);
   cublasZscal_no_device(size,coef,vec_out_gpu,1);hipEventSynchronize(0);

   coef=make_hipDoubleComplex(vecp[Niter_lanczos-1],0.0);
   cublasZaxpy_no_device(size,coef,vec_out_gpu,1,GS_gpu,1); hipDeviceSynchronize(); hipEventSynchronize(0);

   coef=make_hipDoubleComplex(1.0/sqrt(hipCabs(cublasZdotu_no_device(size,GS_gpu,1,GS_gpu,1))),0.0);  hipDeviceSynchronize();  hipEventSynchronize(0);
   cublasZscal_no_device(size,coef,GS_gpu,1); hipEventSynchronize(0); hipDeviceSynchronize();

   hipMemcpy(GS,GS_gpu,size*sizeof(hipDoubleComplex),hipMemcpyHostToHost); hipEventSynchronize(0); hipDeviceSynchronize();

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

   hipFree(normv_ker);
 
   hipEventSynchronize(0); 
   if(use_texture_lcf==0)
   {
    hipHostFree(vec_in_gpu);hipHostFree(Eb);hipHostFree(Ec);hipHostFree(Vbc);
   }else{
    unbind_x(); unbind_x_();
    hipFree(vec_in_gpu_pointer); hipFree(Eb_p);hipFree(Ec_p);hipFree(Vbc_p);
   }
   if(use_texture_rank_c==0){ hipHostFree(sector_ranks);}else{unbind_x_r(); hipFree(sector_ranks_p);};

   hipHostFree(GS_gpu);        hipHostFree(vec_tmp_gpu);  hipHostFree(vec_out_gpu);   hipHostFree(quart_gpu);
   hipHostFree(sector_states); 
   hipHostFree(bathorbs);      hipHostFree(imporbs);
   hipHostFree(maskEb);        hipHostFree(maskEc);       hipHostFree(maskVbc);
 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//
}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
