#include "hip/hip_runtime.h"
#define MAX_SIZE 16

#include <stdio.h>
#include <stdlib.h>
#include <cutil.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>


//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

__global__ void Muld_gpu_fortran_c(int BLOCKSIZE, hipDoubleComplex* A, hipDoubleComplex* B, hipDoubleComplex* C, int ha, int wA, int wB)
{

// A: ha x wA
// B: wA x wB
// C: ha x wB

int bx = blockIdx.x; int by = blockIdx.y; int tx = threadIdx.x; int ty = threadIdx.y;


int aBegin  =   BLOCKSIZE * by                 ;
int aEnd    =   BLOCKSIZE * by  + (wA - 1)*ha  ;
int bBegin  =   BLOCKSIZE * bx *   wA          ;

int aStep   =   BLOCKSIZE * ha                 ;
int bStep   =   BLOCKSIZE                      ;

int c       =   BLOCKSIZE * by + BLOCKSIZE * bx * ha  ;

hipDoubleComplex Csub = make_hipDoubleComplex(0.0,0.0);

/*******************************************************************************/
for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) 
{
__shared__ hipDoubleComplex As[MAX_SIZE][MAX_SIZE]; __shared__ hipDoubleComplex Bs[MAX_SIZE][MAX_SIZE];
   As[ty][tx] = A[a    +      ty + tx*ha];   Bs[ty][tx] = B[b   +      ty + tx*wA ];
__syncthreads();
  for (int k = 0; k < BLOCKSIZE; ++k) Csub = hipCadd(hipCmul(As[ty][k], Bs[k][tx]),Csub );
__syncthreads();
}
   C[c  +      ty + tx*ha ] = Csub ;
/*******************************************************************************/

}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

__global__ void Muld_gpu_cxx_c(int BLOCKSIZE, hipDoubleComplex* A, hipDoubleComplex* B, hipDoubleComplex* C, int ha, int wA, int wB)
{

// A: ha x wA
// B: wA x wB
// C: ha x wB

int bx = blockIdx.x; int by = blockIdx.y; int tx = threadIdx.x; int ty = threadIdx.y;


int aBegin  =  wA * BLOCKSIZE * by           ;
int aEnd    =  wA * BLOCKSIZE * by  + wA - 1 ;
int bBegin  =       BLOCKSIZE * bx           ;

int aStep   =       BLOCKSIZE                ;
int bStep   =       BLOCKSIZE * wB           ;

int c       =  wB * BLOCKSIZE * by + BLOCKSIZE * bx ;

hipDoubleComplex Csub = make_hipDoubleComplex(0.0,0.0);

/*******************************************************************************/
for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) 
{
__shared__ hipDoubleComplex As[MAX_SIZE][MAX_SIZE]; __shared__ hipDoubleComplex Bs[MAX_SIZE][MAX_SIZE];
 As[ty][tx] = A[a    + wA * ty + tx   ];   Bs[ty][tx] = B[b   + wB * ty + tx    ];
__syncthreads();
for (int k = 0; k < BLOCKSIZE; ++k) Csub = hipCadd(hipCmul(As[ty][k], Bs[k][tx]),Csub );  
__syncthreads();
}
 C[c  + wB * ty + tx    ] = Csub ;
/*******************************************************************************/

}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

extern "C" void matmul_gpu_fortran_c_(int* pBLOCKSIZE, hipDoubleComplex** A, hipDoubleComplex** B, hipDoubleComplex** C,  int* hA, int* wA, int* wB)
{

// A: ha x wA
// B: wA x wB
// C: ha x wB

int verbose   =  0 ;
int pinned    =  1 ;
if(*hA<2600 && *wA<2600 && *wB<2600) pinned=0;
int BLOCKSIZE = *pBLOCKSIZE;

int sizea; int sizeb; int sizec;
hipDoubleComplex *Ad  ; hipDoubleComplex *Bd  ; hipDoubleComplex *Cd  ;
hipDoubleComplex *Add ; hipDoubleComplex *Bdd ; hipDoubleComplex *Cdd ;
sizea = *hA * *wA * sizeof(hipDoubleComplex);
sizeb = *wA * *wB * sizeof(hipDoubleComplex);
sizec = *hA * *wB * sizeof(hipDoubleComplex);

if(verbose){printf(" allocate \n ");};
if(pinned==0){
 if(verbose) printf(" allocate memory on device \n");
  hipMalloc((void**) &Ad, sizea);
  hipMalloc((void**) &Bd, sizeb);
  hipMalloc((void**) &Cd, sizec);
}else{
  if(verbose) printf( " allocate pinned memory \n ");
  hipSetDevice(0); hipSetDeviceFlags( hipDeviceMapHost );
  hipHostAlloc((void**)&Ad, sizea, hipHostMallocMapped | hipHostMallocPortable );
  hipHostAlloc((void**)&Bd, sizeb, hipHostMallocMapped | hipHostMallocPortable );
  hipHostAlloc((void**)&Cd, sizec, hipHostMallocMapped | hipHostMallocPortable );
  hipHostGetDevicePointer((void**)&Add, Ad, 0 );
  hipHostGetDevicePointer((void**)&Bdd, Bd, 0 );
  hipHostGetDevicePointer((void**)&Cdd, Cd, 0 );
}
 if(verbose){printf(" copy inputs \n ");};
 if(pinned==0){
   hipMemcpy(Bd, B, sizeb, hipMemcpyHostToDevice);
   hipMemcpy(Ad, A, sizea, hipMemcpyHostToDevice);
 }else{
   hipMemcpy(Bd, B, sizeb, hipMemcpyHostToHost);
   hipMemcpy(Ad, A, sizea, hipMemcpyHostToHost);
 };

 dim3 dimBlock(BLOCKSIZE, BLOCKSIZE);
 dim3 dimGrid(*wB / dimBlock.x, *hA / dimBlock.y);

 if(pinned==0){
 Muld_gpu_fortran_c<<<dimGrid, dimBlock>>>(BLOCKSIZE, Ad, Bd, Cd, *hA, *wA, *wB);
}else{
 Muld_gpu_fortran_c<<<dimGrid, dimBlock>>>(BLOCKSIZE, Add , Bdd , Cdd ,*hA, *wA, *wB );
}

 if(verbose){printf(" copy results \n ");};
 if(pinned==0){
 hipMemcpy(C,Cd,sizec,hipMemcpyDeviceToHost);
 }else{
 hipEventSynchronize(0);
 hipMemcpy(C,Cd,sizec,hipMemcpyHostToHost);
 };
 if(verbose){printf(" free memory \n ");};
 if(pinned==1){
  hipHostFree(Ad); hipHostFree(Bd); hipHostFree(Cd);
 }else{
  hipFree(Ad); hipFree(Bd); hipFree(Cd);
 };


}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

extern "C" void matmul_gpu_cxx_c(int* pBLOCKSIZE, hipDoubleComplex** A, hipDoubleComplex** B, hipDoubleComplex** C,  int* hA, int* wA, int* wB)
{

// A: ha x wA
// B: wA x wB
// C: ha x wB

int verbose   =  0 ;
int pinned    =  1 ;
if(*hA<2600 && *wA<2600 && *wB<2600) pinned=0;
int BLOCKSIZE = *pBLOCKSIZE;

int sizea; int sizeb; int sizec;
hipDoubleComplex *Ad  ; hipDoubleComplex *Bd  ; hipDoubleComplex *Cd  ;
hipDoubleComplex *Add ; hipDoubleComplex *Bdd ; hipDoubleComplex *Cdd ;
sizea = *hA * *wA * sizeof(hipDoubleComplex);
sizeb = *wA * *wB * sizeof(hipDoubleComplex);
sizec = *hA * *wB * sizeof(hipDoubleComplex);

if(verbose){printf(" allocate \n ");};
if(pinned==0){
 if(verbose) printf(" allocate memory on device \n");
  hipMalloc((void**) &Ad, sizea);
  hipMalloc((void**) &Bd, sizeb);
  hipMalloc((void**) &Cd, sizec);
}else{
  if(verbose) printf( " allocate pinned memory \n ");
  hipSetDevice(0); hipSetDeviceFlags( hipDeviceMapHost );
  hipHostAlloc((void**)&Ad, sizea, hipHostMallocMapped | hipHostMallocPortable );
  hipHostAlloc((void**)&Bd, sizeb, hipHostMallocMapped | hipHostMallocPortable );
  hipHostAlloc((void**)&Cd, sizec, hipHostMallocMapped | hipHostMallocPortable );
  hipHostGetDevicePointer((void**)&Add, Ad, 0 );
  hipHostGetDevicePointer((void**)&Bdd, Bd, 0 );
  hipHostGetDevicePointer((void**)&Cdd, Cd, 0 );
}
 if(verbose){printf(" copy inputs \n ");};
 if(pinned==0){
   hipMemcpy(Bd, B, sizeb, hipMemcpyHostToDevice);
   hipMemcpy(Ad, A, sizea, hipMemcpyHostToDevice);
 }else{
   hipMemcpy(Bd, B, sizeb, hipMemcpyHostToHost);
   hipMemcpy(Ad, A, sizea, hipMemcpyHostToHost);
 };

 dim3 dimBlock(BLOCKSIZE, BLOCKSIZE);
 dim3 dimGrid(*wB / dimBlock.x, *hA / dimBlock.y);

if(pinned==0){
 Muld_gpu_cxx_c<<<dimGrid, dimBlock>>>(BLOCKSIZE, Ad, Bd, Cd, *hA, *wA, *wB);
}else{
 Muld_gpu_cxx_c<<<dimGrid, dimBlock>>>(BLOCKSIZE, Add , Bdd , Cdd ,*hA, *wA, *wB );
}

 if(verbose){printf(" copy results \n ");};
 if(pinned==0){
 hipMemcpy(C,Cd,sizec,hipMemcpyDeviceToHost);
 }else{
 hipEventSynchronize(0);
 hipMemcpy(C,Cd,sizec,hipMemcpyHostToHost);
 };
 if(verbose){printf(" free memory \n ");};
 if(pinned==1){
  hipHostFree(Ad); hipHostFree(Bd); hipHostFree(Cd);
 }else{
  hipFree(Ad); hipFree(Bd); hipFree(Cd);
 };

}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

