#include "hip/hip_runtime.h"
 
 #include <stdio.h>
 #include <stdlib.h>
 #include <hip/hip_runtime.h>
 #include <hipblas.h>
 #include <hip/hip_complex.h>


#define MAX_BLOCK 22

hipDoubleComplex *Eb , *frequ , *collect_array;


//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

texture<int2,1,hipReadModeElementType> tex;
inline void   bind_x(hipDoubleComplex *x) {    hipBindTexture(0,tex,x); };
inline void unbind_x()                   {  hipUnbindTexture(  tex  ); };
__inline__  __device__ hipDoubleComplex fetch_x(const int& i)
  {
         int  jj = 2*(i-1);
         int2 v  = tex1Dfetch(tex,jj);
      double rr  = __hiloint2double(v.y, v.x);
              v  = tex1Dfetch(tex,jj+1);
      double im  = __hiloint2double(v.y, v.x);
      return make_hipDoubleComplex(rr,im);
  }

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

  __global__ void build_array_inverse_array (int nthreads, int nfrequ, int nnn, double *collect )

{

    double sum,x;
    double Eb_[32*32];

    /////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////

    for(int ifrequ=threadIdx.x;ifrequ<nfrequ;ifrequ+=nthreads) {

    for(int iorb=0;iorb<nnn*nnn;iorb++)   Eb_[iorb]  = collect[iorb+nnn*nnn*ifrequ];

    for (int i=1; i < nnn; i++) Eb_[i] /= Eb_[0]; 

    for (int i=1; i < nnn; i++)  
      {
      for (int j=i; j < nnn; j++)  { 
        sum = 0.0;
        for (int k = 0; k < i; k++)
            sum += Eb_[j*nnn+k] * Eb_[k*nnn+i];
        Eb_[j*nnn+i] -= sum;
        }
      if (i == nnn-1) continue;

    for (int j=i+1; j < nnn; j++)  
       {  
        sum = 0.0;
        for (int k = 0; k < i; k++) sum += Eb_[i*nnn+k]*Eb_[k*nnn+j];
        Eb_[i*nnn+j] = (Eb_[i*nnn+j]-sum) / Eb_[i*nnn+i];
        }
      }

    for ( int i = 0; i < nnn; i++ )  
      for ( int j = i; j < nnn; j++ )  {
        x = 1.0;
        if ( i != j ) {
          x = 0.0;
          for ( int k = i; k < j; k++ )
              x -= Eb_[j*nnn+k]*Eb_[k*nnn+i];
          }
        Eb_[j*nnn+i] = x / Eb_[j*nnn+j];
        }

    for ( int i = 0; i < nnn; i++ )  
      for ( int j = i; j < nnn; j++ )  {
        if ( i == j ) continue;
        sum = 0.0;
        for ( int k = i; k < j; k++ )
            sum += Eb_[k*nnn+j]*( (i==k) ? 1.0 : Eb_[i*nnn+k] );
        Eb_[i*nnn+j] = -sum;
        }

    for ( int i = 0; i < nnn; i++ )   
      for ( int j = 0; j < nnn; j++ )  {
        sum = 0.0;
        for ( int k = ((i>j)?i:j); k < nnn; k++ )
            sum += ((j==k)?1.0:Eb_[j*nnn+k])*Eb_[k*nnn+i];
        Eb_[j*nnn+i] = sum;
        }

    for(int iorb=0;iorb<nnn*nnn;iorb++) collect[iorb+ifrequ*nnn*nnn]=Eb_[iorb];

   };

    /////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////

}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

  __global__ void build_array_inverse_collect (int nthreads, int nfrequ, int nnn, double* Eb,double *collect, double *vec )

{

    double sum,x;

    __shared__ double shared_[32*32];
               double Eb_[32*32];
    
    for(int iorb=0;iorb<nnn*nnn;iorb++) shared_[iorb]=Eb[iorb];

    /////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////

    for(int ifrequ=threadIdx.x;ifrequ<nfrequ;ifrequ+=nthreads) {

    for(int iorb=0;iorb<nnn*nnn;iorb++)   Eb_[iorb         ]  = -shared_[iorb];
    for(int iorb=0;iorb<nnn    ;iorb++)   Eb_[iorb*nnn+iorb] +=  vec[ifrequ];

    for (int i=1; i < nnn; i++) Eb_[i] /= Eb_[0]; 

    for (int i=1; i < nnn; i++)  
      {
      for (int j=i; j < nnn; j++)  { 
        sum = 0.0;
        for (int k = 0; k < i; k++)
            sum += Eb_[j*nnn+k] * Eb_[k*nnn+i];
        Eb_[j*nnn+i] -= sum;
        }
      if (i == nnn-1) continue;

    for (int j=i+1; j < nnn; j++)  
       {  
        sum = 0.0;
        for (int k = 0; k < i; k++) sum += Eb_[i*nnn+k]*Eb_[k*nnn+j];
        Eb_[i*nnn+j] = (Eb_[i*nnn+j]-sum) / Eb_[i*nnn+i];
        }
      }

    for ( int i = 0; i < nnn; i++ )  
      for ( int j = i; j < nnn; j++ )  {
        x = 1.0;
        if ( i != j ) {
          x = 0.0;
          for ( int k = i; k < j; k++ )
              x -= Eb_[j*nnn+k]*Eb_[k*nnn+i];
          }
        Eb_[j*nnn+i] = x / Eb_[j*nnn+j];
        }

    for ( int i = 0; i < nnn; i++ )  
      for ( int j = i; j < nnn; j++ )  {
        if ( i == j ) continue;
        sum = 0.0;
        for ( int k = i; k < j; k++ )
            sum += Eb_[k*nnn+j]*( (i==k) ? 1.0 : Eb_[i*nnn+k] );
        Eb_[i*nnn+j] = -sum;
        }

    for ( int i = 0; i < nnn; i++ )   
      for ( int j = 0; j < nnn; j++ )  {
        sum = 0.0;
        for ( int k = ((i>j)?i:j); k < nnn; k++ )
            sum += ((j==k)?1.0:Eb_[j*nnn+k])*Eb_[k*nnn+i];
        Eb_[j*nnn+i] = sum;
        }

    for(int iorb=0;iorb<nnn*nnn;iorb++) collect[iorb+ifrequ*nnn*nnn]=Eb_[iorb];

   };

    /////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////

}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

  __global__ void build_array_inverse (int nthreads, int nfrequ, int nnn, double* Eb, double *vec )

{

    double sum,x;

    __shared__ double shared_[32*32];
               double Eb_[32*32],tot[32*32];
    
    for(int iorb=0;iorb<nnn*nnn;iorb++) shared_[iorb]=Eb[iorb];

    /////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////

    for(int iorb=0;iorb<nnn*nnn;iorb++) tot[iorb]=0.0;

    for(int ifrequ=threadIdx.x;ifrequ<nfrequ;ifrequ+=nthreads) {

    for(int iorb=0;iorb<nnn*nnn;iorb++)   Eb_[iorb         ]  = -shared_[iorb];
    for(int iorb=0;iorb<nnn    ;iorb++)   Eb_[iorb*nnn+iorb] +=  vec[ifrequ];

    for (int i=1; i < nnn; i++) Eb_[i] /= Eb_[0]; 

    for (int i=1; i < nnn; i++)  
      {
      for (int j=i; j < nnn; j++)  { 
        sum = 0.0;
        for (int k = 0; k < i; k++)
            sum += Eb_[j*nnn+k] * Eb_[k*nnn+i];
        Eb_[j*nnn+i] -= sum;
        }
      if (i == nnn-1) continue;

    for (int j=i+1; j < nnn; j++)  
       {  
        sum = 0.0;
        for (int k = 0; k < i; k++) sum += Eb_[i*nnn+k]*Eb_[k*nnn+j];
        Eb_[i*nnn+j] = (Eb_[i*nnn+j]-sum) / Eb_[i*nnn+i];
        }
      }

    for ( int i = 0; i < nnn; i++ )  
      for ( int j = i; j < nnn; j++ )  {
        x = 1.0;
        if ( i != j ) {
          x = 0.0;
          for ( int k = i; k < j; k++ )
              x -= Eb_[j*nnn+k]*Eb_[k*nnn+i];
          }
        Eb_[j*nnn+i] = x / Eb_[j*nnn+j];
        }

    for ( int i = 0; i < nnn; i++ )  
      for ( int j = i; j < nnn; j++ )  {
        if ( i == j ) continue;
        sum = 0.0;
        for ( int k = i; k < j; k++ )
            sum += Eb_[k*nnn+j]*( (i==k) ? 1.0 : Eb_[i*nnn+k] );
        Eb_[i*nnn+j] = -sum;
        }

    for ( int i = 0; i < nnn; i++ )   
      for ( int j = 0; j < nnn; j++ )  {
        sum = 0.0;
        for ( int k = ((i>j)?i:j); k < nnn; k++ )
            sum += ((j==k)?1.0:Eb_[j*nnn+k])*Eb_[k*nnn+i];
        Eb_[j*nnn+i] = sum;
        }

    for(int iorb=0;iorb<nnn*nnn;iorb++) tot[iorb]+=Eb_[iorb];

   };

    /////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////

   for(int iorb=0;iorb<nnn*nnn;iorb++) Eb_[iorb]=0.0;
   for(int ii=0;ii<nthreads;ii++) 
   {
    if(threadIdx.x==ii){ for(int iorb=0;iorb<nnn*nnn;iorb++) shared_[iorb] = tot[iorb];};
                         __syncthreads();
                         for(int iorb=0;iorb<nnn*nnn;iorb++) Eb_[iorb]+= shared_[iorb];  
   };
                         __syncthreads();
   for(int iorb=0;iorb<nnn*nnn;iorb++) Eb[iorb] = Eb_[iorb];

}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

extern "C" void sum_of_inverse_frequ_(int* nnn_, int* nfrequ_,  double* Eb_, double* totsum_ , double *frequ_)

{

  int nnn ; int nfrequ; nnn=*nnn_; 
  nfrequ=*nfrequ_;

  double *Eb , *frequ ;

  hipMalloc (  (void**)&Eb     , nnn*nnn   *sizeof(double) );
  hipMalloc (  (void**)&frequ  , nfrequ *sizeof(double) );
  hipMemcpy ( Eb      ,Eb_     , nnn*nnn*sizeof(double)   , hipMemcpyHostToDevice);
  hipMemcpy ( frequ   ,frequ_  , nfrequ*sizeof(double) , hipMemcpyHostToDevice);

  if(nnn>32) { printf( " sum of inverse real cuda (1) : matrices are too big!!!!! \n");};

  build_array_inverse <<<1,512>>> ( 512, nfrequ, nnn, Eb, frequ );

  hipEventSynchronize(0); hipDeviceSynchronize();

  hipMemcpy(totsum_ , Eb , nnn*nnn*sizeof(double) , hipMemcpyDeviceToHost);
  hipFree(frequ); hipFree(Eb);

}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

extern "C" void sum_of_inverse_frequ_collect_(int* nnn_, int* nfrequ_,  double* Eb_, double* totsum_ , double *frequ_)

{

  int nnn ; int nfrequ; nnn=*nnn_; 
  nfrequ=*nfrequ_;

  double *Eb , *frequ , *collect;

  hipMalloc (  (void**)&Eb     , nnn*nnn*sizeof(double) );
  hipMalloc (  (void**)&frequ  , nfrequ *sizeof(double) );
  hipMalloc (  (void**)&collect, nnn*nnn*nfrequ *sizeof(double) );

  hipMemcpy ( Eb      ,Eb_     , nnn*nnn*sizeof(double)   , hipMemcpyHostToDevice);
  hipMemcpy ( frequ   ,frequ_  , nfrequ*sizeof(double) , hipMemcpyHostToDevice);

  if(nnn>32) { printf( " sum of inverse real cuda (2) : matrices are too big!!!!! \n" );};

  build_array_inverse_collect <<<1,512>>> ( 512, nfrequ, nnn, Eb, collect, frequ );

  hipEventSynchronize(0); hipDeviceSynchronize();

  hipMemcpy(totsum_ , collect , nnn*nnn*nfrequ*sizeof(double) , hipMemcpyDeviceToHost);
  hipFree(frequ); hipFree(Eb);

}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

extern "C" void sum_of_inverse_frequ_array_(int* nnn_, int* nfrequ_, double* totsum_)

{

  int nnn ; int nfrequ; nnn=*nnn_; 
   nfrequ=*nfrequ_;

  double *collect;

  hipMalloc (  (void**)&collect, nnn*nnn*nfrequ *sizeof(double) );
  hipMemcpy ( collect      ,totsum_     , nnn*nnn*nfrequ*sizeof(double)   , hipMemcpyHostToDevice);

  if(nnn>32) { printf( " sum of inverse real cuda (3) : matrices are too big!!!!! \n" );};

  build_array_inverse_array <<<1,512>>> ( 512, nfrequ, nnn, collect );

  hipEventSynchronize(0); hipDeviceSynchronize();

  hipMemcpy(totsum_ , collect , nnn*nnn*nfrequ*sizeof(double) , hipMemcpyDeviceToHost);
  hipFree(collect); 

}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

  __global__ void build_array_inverse_complex (int blocksize, int nfrequ, int nnn, hipDoubleComplex* Eb, hipDoubleComplex *vec )

{
    hipDoubleComplex sum,x,moins1,zero,uno;
    zero=make_hipDoubleComplex(0.0,0.0); moins1=make_hipDoubleComplex(-1.0,0.0); uno=make_hipDoubleComplex(1.0,0.0);

    __shared__ hipDoubleComplex sum_[512];
    __shared__ hipDoubleComplex shared_[MAX_BLOCK*MAX_BLOCK]; 
               hipDoubleComplex Eb_[MAX_BLOCK*MAX_BLOCK]; 
               hipDoubleComplex tot_[MAX_BLOCK*MAX_BLOCK];

    for(int iorb=0;iorb<nnn*nnn;iorb++) shared_[iorb]=hipCmul(Eb[iorb],moins1);
    for(int iorb=0;iorb<nnn*nnn;iorb++) tot_[iorb]=zero;

    for(int ifrequ=threadIdx.x;ifrequ<nfrequ;ifrequ+=blocksize) {

    /////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////

    for(int iorb=0;iorb<nnn*nnn;iorb++)   Eb_[iorb         ]      = shared_[iorb];
    for(int iorb=0;iorb<nnn    ;iorb++)   Eb_[iorb*nnn+iorb]      = hipCadd(Eb_[iorb*nnn+iorb],vec[ifrequ]);

    for (int i=1; i < nnn; i++) Eb_[i] = hipCdiv(Eb_[i],Eb_[0]); 

    for (int i=1; i < nnn; i++)  
      {
      for (int j=i; j < nnn; j++)  { 
        sum = zero;
        for (int k = 0; k < i; k++)
            sum = hipCadd(sum, hipCmul(Eb_[j*nnn+k] , Eb_[k*nnn+i]));
        Eb_[j*nnn+i] = hipCsub(Eb_[j*nnn+i] ,sum);
      }
      if (i == nnn-1) continue;

    for (int j=i+1; j < nnn; j++)  
       {  
        sum = zero;
        for (int k = 0; k < i; k++) sum = hipCadd(sum,hipCmul(Eb_[i*nnn+k],Eb_[k*nnn+j]));
        Eb_[i*nnn+j] = hipCdiv( hipCsub( Eb_[i*nnn+j],sum),Eb_[i*nnn+i]);
        }
      }

    for ( int i = 0; i < nnn; i++ )  
      for ( int j = i; j < nnn; j++ )  {
        x = uno ;
        if ( i != j ) {
          x = zero;
          for ( int k = i; k < j; k++ )
              x = hipCsub(x, hipCmul(Eb_[j*nnn+k],Eb_[k*nnn+i]));
          }
        Eb_[j*nnn+i] = hipCdiv(x , Eb_[j*nnn+j]);
        }

    for ( int i = 0; i < nnn; i++ )  
      for ( int j = i; j < nnn; j++ )  {
        if ( i == j ) continue;
        sum = zero;
        for ( int k = i; k < j; k++ )
           if(i==k){
            sum = hipCadd(sum,Eb_[k*nnn+j]);
           }else{
            sum = hipCadd(sum,hipCmul(Eb_[k*nnn+j],Eb_[i*nnn+k] ));
           };
        Eb_[i*nnn+j] = hipCmul(sum,moins1);
        }

    for ( int i = 0; i < nnn; i++ )   
      for ( int j = 0; j < nnn; j++ )  {
        sum = zero;
        for ( int k = ((i>j)?i:j); k < nnn; k++ )
          if(j==k){
            sum = hipCadd(sum,Eb_[k*nnn+i]);
          }else{
            sum = hipCadd(sum,hipCmul(Eb_[j*nnn+k],Eb_[k*nnn+i]));
          }
        Eb_[j*nnn+i] = sum;
        }

      for(int iorb=0;iorb<nnn*nnn;iorb++) 
      { tot_[iorb]=hipCadd(tot_[iorb],Eb_[iorb]); };

   };

    /////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////
  
   // parallel reduction
   for(int iorb=0;iorb<nnn*nnn;iorb++) 
    {
     int i =threadIdx.x;
     sum_[i]=tot_[iorb]; 
     __syncthreads();
     for(int bit=blocksize/2; bit>0; bit/=2)
      {
      hipDoubleComplex t = hipCadd(sum_[i],sum_[i^bit]); __syncthreads();
      sum_[i]=t ;__syncthreads();
      }
      Eb[iorb]=sum_[i];
   };
}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

extern "C" void sum_of_inverse_frequ_complex_(int* nnn_, int* nfrequ_,  hipDoubleComplex* Eb_, hipDoubleComplex* totsum_ , 
                                             hipDoubleComplex *frequ_, int* firstlast)

{

  int nnn ; int nfrequ; nnn=*nnn_; 
  nfrequ=*nfrequ_;

  const int nthreads = 512 ;

  if(nnn>MAX_BLOCK)  { printf( " sum of inverse complex cuda (1) : matrices are too big!!!!! \n"); };

if(*firstlast==1){
  hipMalloc (  (void**)&Eb     , nnn*nnn   *sizeof(hipDoubleComplex) );
  hipMalloc (  (void**)&frequ  , nfrequ *sizeof(hipDoubleComplex) );
};

  hipMemcpy ( Eb      ,Eb_     , nnn*nnn*sizeof(hipDoubleComplex)   , hipMemcpyHostToDevice);
if(*firstlast==1){
  hipMemcpy ( frequ   ,frequ_  , nfrequ*sizeof(hipDoubleComplex) , hipMemcpyHostToDevice);
}

  build_array_inverse_complex <<<1,nthreads>>> ( nthreads, nfrequ, nnn, Eb, frequ);
  hipEventSynchronize(0); hipDeviceSynchronize();

  hipMemcpy(totsum_ , Eb , nnn*nnn*sizeof(hipDoubleComplex) , hipMemcpyDeviceToHost);
  if(*firstlast==2){ hipFree(frequ); hipFree(Eb);  };

}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

  __global__ void build_array_inverse_complex_collect (int blocksize, int nfrequ, int nnn, hipDoubleComplex* Eb, 
                                 hipDoubleComplex *tot,hipDoubleComplex *vec )

{
    hipDoubleComplex sum,x,moins1,zero,uno;
    zero=make_hipDoubleComplex(0.0,0.0); moins1=make_hipDoubleComplex(-1.0,0.0); uno=make_hipDoubleComplex(1.0,0.0);

    __shared__ hipDoubleComplex shared_[MAX_BLOCK*MAX_BLOCK]; 
               hipDoubleComplex Eb_[MAX_BLOCK*MAX_BLOCK]; 

    for(int iorb=0;iorb<nnn*nnn;iorb++) shared_[iorb]=hipCmul(Eb[iorb],moins1);

    for(int ifrequ=threadIdx.x;ifrequ<nfrequ;ifrequ+=blocksize) {

    /////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////

    for(int iorb=0;iorb<nnn*nnn;iorb++)   Eb_[iorb         ]      = shared_[iorb];
    for(int iorb=0;iorb<nnn    ;iorb++)   Eb_[iorb*nnn+iorb]      = hipCadd(Eb_[iorb*nnn+iorb],vec[ifrequ]);

    for (int i=1; i < nnn; i++) Eb_[i] = hipCdiv(Eb_[i],Eb_[0]); 

    for (int i=1; i < nnn; i++)  
      {
      for (int j=i; j < nnn; j++)  { 
        sum = zero;
        for (int k = 0; k < i; k++)
            sum = hipCadd(sum, hipCmul(Eb_[j*nnn+k] , Eb_[k*nnn+i]));
        Eb_[j*nnn+i] = hipCsub(Eb_[j*nnn+i] ,sum);
      }
      if (i == nnn-1) continue;

    for (int j=i+1; j < nnn; j++)  
       {  
        sum = zero;
        for (int k = 0; k < i; k++) sum = hipCadd(sum,hipCmul(Eb_[i*nnn+k],Eb_[k*nnn+j]));
        Eb_[i*nnn+j] = hipCdiv( hipCsub( Eb_[i*nnn+j],sum),Eb_[i*nnn+i]);
        }
      }

    for ( int i = 0; i < nnn; i++ )  
      for ( int j = i; j < nnn; j++ )  {
        x = uno ;
        if ( i != j ) {
          x = zero;
          for ( int k = i; k < j; k++ )
              x = hipCsub(x, hipCmul(Eb_[j*nnn+k],Eb_[k*nnn+i]));
          }
        Eb_[j*nnn+i] = hipCdiv(x , Eb_[j*nnn+j]);
        }

    for ( int i = 0; i < nnn; i++ )  
      for ( int j = i; j < nnn; j++ )  {
        if ( i == j ) continue;
        sum = zero;
        for ( int k = i; k < j; k++ )
           if(i==k){
            sum = hipCadd(sum,Eb_[k*nnn+j]);
           }else{
            sum = hipCadd(sum,hipCmul(Eb_[k*nnn+j],Eb_[i*nnn+k] ));
           };
        Eb_[i*nnn+j] = hipCmul(sum,moins1);
        }

    for ( int i = 0; i < nnn; i++ )   
      for ( int j = 0; j < nnn; j++ )  {
        sum = zero;
        for ( int k = ((i>j)?i:j); k < nnn; k++ )
          if(j==k){
            sum = hipCadd(sum,Eb_[k*nnn+i]);
          }else{
            sum = hipCadd(sum,hipCmul(Eb_[j*nnn+k],Eb_[k*nnn+i]));
          }
        Eb_[j*nnn+i] = sum;
        }

      for(int iorb=0;iorb<nnn*nnn;iorb++) { tot[iorb+ifrequ*nnn*nnn]=Eb_[iorb];};

   };

    /////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////
  
}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

  __global__ void build_array_inverse_complex_array (int blocksize, int nfrequ, int nnn, hipDoubleComplex *tot )

{
    hipDoubleComplex sum,x,moins1,zero,uno;
    zero=make_hipDoubleComplex(0.0,0.0); moins1=make_hipDoubleComplex(-1.0,0.0); uno=make_hipDoubleComplex(1.0,0.0);
    hipDoubleComplex Eb_[MAX_BLOCK*MAX_BLOCK]; 
    int ifrequ=blocksize*blockIdx.x+threadIdx.x;

    if(ifrequ<nfrequ) {

    /////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////

    for(int iorb=0;iorb<nnn*nnn;iorb++) { Eb_[iorb] = tot[iorb+nnn*nnn*ifrequ];}

    for (int i=1; i < nnn; i++) Eb_[i] = hipCdiv(Eb_[i],Eb_[0]); 

    for (int i=1; i < nnn; i++)  
      {
      for (int j=i; j < nnn; j++)  { 
        sum = zero;
        for (int k = 0; k < i; k++)
            sum = hipCadd(sum, hipCmul(Eb_[j*nnn+k] , Eb_[k*nnn+i]));
        Eb_[j*nnn+i] = hipCsub(Eb_[j*nnn+i] ,sum);
      }
      if (i == nnn-1) continue;

    for (int j=i+1; j < nnn; j++)  
       {  
        sum = zero;
        for (int k = 0; k < i; k++) sum = hipCadd(sum,hipCmul(Eb_[i*nnn+k],Eb_[k*nnn+j]));
        Eb_[i*nnn+j] = hipCdiv( hipCsub( Eb_[i*nnn+j],sum),Eb_[i*nnn+i]);
        }
      }

    for ( int i = 0; i < nnn; i++ )  
      for ( int j = i; j < nnn; j++ )  {
        x = uno ;
        if ( i != j ) {
          x = zero;
          for ( int k = i; k < j; k++ )
              x = hipCsub(x, hipCmul(Eb_[j*nnn+k],Eb_[k*nnn+i]));
          }
        Eb_[j*nnn+i] = hipCdiv(x , Eb_[j*nnn+j]);
        }

    for ( int i = 0; i < nnn; i++ )  
      for ( int j = i; j < nnn; j++ )  {
        if ( i == j ) continue;
        sum = zero;
        for ( int k = i; k < j; k++ )
           if(i==k){
            sum = hipCadd(sum,Eb_[k*nnn+j]);
           }else{
            sum = hipCadd(sum,hipCmul(Eb_[k*nnn+j],Eb_[i*nnn+k] ));
           };
        Eb_[i*nnn+j] = hipCmul(sum,moins1);
        }

    for ( int i = 0; i < nnn; i++ )   
      for ( int j = 0; j < nnn; j++ )  {
        sum = zero;
        for ( int k = ((i>j)?i:j); k < nnn; k++ )
          if(j==k){
            sum = hipCadd(sum,Eb_[k*nnn+i]);
          }else{
            sum = hipCadd(sum,hipCmul(Eb_[j*nnn+k],Eb_[k*nnn+i]));
          }
        Eb_[j*nnn+i] = sum;
        }

      for(int iorb=0;iorb<nnn*nnn;iorb++) { tot[iorb+ifrequ*nnn*nnn]=Eb_[iorb];};

   };

    /////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////
  
}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

extern "C" void sum_of_inverse_frequ_complex_collect_(int* nnn_, int* nfrequ_,  hipDoubleComplex* Eb_, 
                                             hipDoubleComplex* collect_ , 
                                             hipDoubleComplex *frequ_, int* firstlast)

{

  int nnn ; int nfrequ; nnn=*nnn_; 
  nfrequ=*nfrequ_;

  const int nthreads = 512 ;

  if(nnn>MAX_BLOCK)           { printf( " sum of inverse complex cuda (2) : matrices are too big!!!!! \n"); };

if(*firstlast==1){
  hipMalloc (  (void**)&Eb     , nnn*nnn   *sizeof(hipDoubleComplex) );
  hipMalloc (  (void**)&frequ  , nfrequ *sizeof(hipDoubleComplex) );
  hipMalloc (  (void**)&collect_array  , nfrequ*nnn*nnn*sizeof(hipDoubleComplex) );
};

  hipMemcpy ( Eb      ,Eb_     , nnn*nnn*sizeof(hipDoubleComplex)   , hipMemcpyHostToDevice);
if(*firstlast==1){
  hipMemcpy ( frequ   ,frequ_  , nfrequ*sizeof(hipDoubleComplex) , hipMemcpyHostToDevice);
}

  build_array_inverse_complex_collect <<<1,nthreads>>> ( nthreads,nfrequ,nnn,Eb,collect_array,frequ);
  hipEventSynchronize(0); hipDeviceSynchronize();
  hipMemcpy (collect_ ,collect_array  , nfrequ*nnn*nnn*sizeof(hipDoubleComplex) , hipMemcpyDeviceToHost);
  hipEventSynchronize(0); hipDeviceSynchronize();

  if(*firstlast==2){ hipFree(frequ); hipFree(Eb); hipFree(collect_array);};

}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

extern "C" void sum_of_inverse_frequ_complex_array_(int* nnn_, int* nfrequ_, hipDoubleComplex* collect_ , int* firstlast)

{

  int nnn ; int nfrequ; nnn=*nnn_; nfrequ=*nfrequ_;
  const int nthreads = 256 ;

  if(nnn>MAX_BLOCK)           { printf( " sum of inverse complex cuda (3) : matrices are too big!!!!! \n"); };
  if(nfrequ/nthreads+1>65500) { printf( " too many blocks in inverse array cuda \n ");};

if(*firstlast==1){
  hipMalloc (  (void**)&collect_array  , nfrequ*nnn*nnn*sizeof(hipDoubleComplex) );
};
  hipMemcpy ( collect_array ,collect_  , nnn*nnn*nfrequ*sizeof(hipDoubleComplex) , hipMemcpyHostToDevice);

  build_array_inverse_complex_array <<<nfrequ/nthreads+1,nthreads>>> ( nthreads,nfrequ,nnn,collect_array);
  hipEventSynchronize(0); hipDeviceSynchronize();
  
  hipMemcpy (collect_ ,collect_array  , nfrequ*nnn*nnn*sizeof(hipDoubleComplex) , hipMemcpyDeviceToHost);

  if(*firstlast==2){hipFree(collect_array);};

}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
