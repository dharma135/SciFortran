#include "hip/hip_runtime.h"

 #include <stdio.h>
 #include <stdlib.h>
 #include <hip/hip_runtime.h>
 #include <hipblas.h>

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

const int use_texture_lr = 1 ;

#define WARP_SIZE 32
#define MAX_BLOCK 65500

texture<int2,1,hipReadModeElementType> tex;

inline void   bind_x(double *x, int N) {    hipBindTexture(0,tex,x,N*sizeof(double)); };
inline void unbind_x()                 {  hipUnbindTexture(  tex  ); };

__inline__    __device__ double fetch_x(const int& i)
  {   int2 v = tex1Dfetch(tex,i-1); return __hiloint2double(v.y, v.x); }

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

  //************************************************//
  //               Kernel Hmult                     //
  //************************************************//

     //---------------------------------------//
     //---------------------------------------//
     //---------------------------------------//
     //---------------------------------------//

  __global__  void Hmult_ker(int ngrid,int BLOCK_SIZE, int num_rows, double *y, const double *x, const double *QUART, const double *diagsz, 
                             const int *noffsz, const int *Aj, const double *Ax, const int *offdia, const int use_texture_lr)
{
   __shared__ double sdata[16][WARP_SIZE]; 
   __shared__ int     ptrs[32][2];
   __shared__ double  temp[32][3];

          int row ;  
    const int warp_lane   = threadIdx.y; 
    const int thread_lane = threadIdx.x;

    int row_start; int row_end; int jj;

   for(int iii=0; iii<=ngrid; iii++){

   row = BLOCK_SIZE * (blockIdx.y+iii*MAX_BLOCK) + threadIdx.y ;

    if(row<num_rows)
   {
        if(thread_lane==0) ptrs[warp_lane][0]=offdia[row];
        if(thread_lane==1) ptrs[warp_lane][1]=noffsz[row];
        if(thread_lane==2) temp[warp_lane][0]=QUART[row];
        if(thread_lane==3) temp[warp_lane][1]=diagsz[row];
        
         if(use_texture_lr==0)
         {
           y[row]=(temp[warp_lane][0]+temp[warp_lane][1]) * x[row];
         }else{
           y[row]=(temp[warp_lane][0]+temp[warp_lane][1]) * fetch_x(row+1);
         };

        row_start = ptrs[warp_lane][0] ; row_end = ptrs[warp_lane][1]+row_start ; 

        sdata[threadIdx.y][threadIdx.x]=0.0;
        
       if(use_texture_lr==1)
       { 
        for(jj=row_start+thread_lane;jj<row_end;jj+=WARP_SIZE) sdata[threadIdx.y][threadIdx.x]+=Ax[jj] * fetch_x(Aj[jj]); 
       }else{
        for(jj=row_start+thread_lane;jj<row_end;jj+=WARP_SIZE) sdata[threadIdx.y][threadIdx.x]+=Ax[jj] *     x[Aj[jj]-1]; 
       }

        if (thread_lane < 16) { sdata[threadIdx.y][threadIdx.x] += sdata[threadIdx.y][threadIdx.x + 16]; };
        if (thread_lane <  8) { sdata[threadIdx.y][threadIdx.x] += sdata[threadIdx.y][threadIdx.x +  8]; };
        if (thread_lane <  4) { sdata[threadIdx.y][threadIdx.x] += sdata[threadIdx.y][threadIdx.x +  4]; };
        if (thread_lane <  2) { sdata[threadIdx.y][threadIdx.x] += sdata[threadIdx.y][threadIdx.x +  2]; };
        if (thread_lane <  1) { sdata[threadIdx.y][threadIdx.x] += sdata[threadIdx.y][threadIdx.x +  1]; };

        if (thread_lane == 0) y[row] += sdata[threadIdx.y][threadIdx.x];
   };
   };
}
     //---------------------------------------//
     //---------------------------------------//
     //---------------------------------------//
     //---------------------------------------//
     //---------------------------------------//
     //---------------------------------------//
     //---------------------------------------//

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

extern "C" void hmult_sz_real_cuda_rout_(int *pblocksize, int *offdiasize, int *roff, 
int* ntot, double* QUART, double* diagsz, double* vec_in, double* vec_out, int* noffsz, 
int* rankoffsz, double* offdiagsz )
{

 //---------------------------------------------------------------------------------------//
     int verbose=0;
 //---------------------------------------------------------------------------------------//

     if(verbose==1) printf(" start Hmult GPU \n" );

     int blocksize= *pblocksize; int size = *ntot;
 
    int nb=(size-size % blocksize)/blocksize+1; int ngrid=nb/MAX_BLOCK; if(ngrid>0) nb=MAX_BLOCK;
    dim3 bl(1,nb),th(WARP_SIZE,blocksize);

    if(verbose==1) printf( " --------------- \n  Nblock=%d Ngrid=%d \n ----------------- \n ",nb,ngrid);

     double  *vec_in_gpu,*vec_out_gpu,*QUART_gpu,*diagsz_gpu,*offdiagsz_gpu;
     int     *noffsz_gpu,*rankoffsz_gpu;
     double  *vec_in_gpu_pointer,*vec_out_gpu_pointer;
     double  *QUART_gpu_pointer,*diagsz_gpu_pointer,*offdiagsz_gpu_pointer;
     int     *noffsz_gpu_pointer,*rankoffsz_gpu_pointer;
     int     *offdia_gpu, *offdia_gpu_pointer;

     if(verbose==1) printf(" GPU , size of Lanczos vector = %d ", size);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

       hipSetDevice(0); hipSetDeviceFlags( hipDeviceMapHost ); hipEventSynchronize(0);

       if(use_texture_lr==1){
         hipMalloc((void**)&vec_in_gpu_pointer,sizeof(double)*size);
         bind_x(vec_in_gpu_pointer,size);
       }else{
         hipHostAlloc((void**)&vec_in_gpu    , size*sizeof(double),        hipHostMallocMapped | hipHostMallocPortable );
         hipHostGetDevicePointer((void**)  &vec_in_gpu_pointer  ,  vec_in_gpu    , 0 );
       }

       hipHostAlloc((void**)&vec_out_gpu   , size*sizeof(double),        hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&QUART_gpu     , size*sizeof(double),        hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&diagsz_gpu    , size*sizeof(double),        hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&offdiagsz_gpu , *offdiasize*sizeof(double), hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&noffsz_gpu    , size*sizeof(int)   ,        hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&rankoffsz_gpu , *roff*sizeof(int)  ,        hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&offdia_gpu    , size*sizeof(int)   ,        hipHostMallocMapped | hipHostMallocPortable );
       hipEventSynchronize(0);

       hipHostGetDevicePointer((void**)   &vec_out_gpu_pointer ,  vec_out_gpu    , 0 );
       hipHostGetDevicePointer((void**)   &QUART_gpu_pointer ,    QUART_gpu      , 0 );
       hipHostGetDevicePointer((void**)   &diagsz_gpu_pointer ,   diagsz_gpu     , 0 );
       hipHostGetDevicePointer((void**)   &offdiagsz_gpu_pointer, offdiagsz_gpu  , 0 );
       hipHostGetDevicePointer((void**)   &noffsz_gpu_pointer,    noffsz_gpu     , 0 );
       hipHostGetDevicePointer((void**)   &rankoffsz_gpu_pointer, rankoffsz_gpu  , 0 );
       hipHostGetDevicePointer((void**)   &offdia_gpu_pointer,    offdia_gpu     , 0 );
       hipEventSynchronize(0);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//
       if(use_texture_lr==0) {
        hipMemcpy(vec_in_gpu,    vec_in,      size*sizeof(double),        hipMemcpyHostToHost);
       }else{
        hipMemcpy(vec_in_gpu_pointer,    vec_in,      size*sizeof(double),        hipMemcpyHostToDevice);
       }
       hipMemcpy(vec_out_gpu,   vec_out,     size*sizeof(double),        hipMemcpyHostToHost);
       hipMemcpy(QUART_gpu,     QUART,       size*sizeof(double),        hipMemcpyHostToHost);
       hipMemcpy(diagsz_gpu,    diagsz,      size*sizeof(double),        hipMemcpyHostToHost);
       hipMemcpy(offdiagsz_gpu, offdiagsz,   *offdiasize*sizeof(double), hipMemcpyHostToHost);
       hipMemcpy(noffsz_gpu,    noffsz ,     size*sizeof(int),           hipMemcpyHostToHost);
       hipMemcpy(rankoffsz_gpu, rankoffsz,   *roff*sizeof(int),          hipMemcpyHostToHost);
       hipEventSynchronize(0);
       offdia_gpu[0]=0; for(int istate=1; istate<size; istate++) { offdia_gpu[istate]=offdia_gpu[istate-1]+noffsz[istate-1]; };
       hipEventSynchronize(0);
 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

  if(verbose==1) printf(" call kernel  \n ");
  Hmult_ker<<<bl,th>>>(ngrid,blocksize,size,vec_out_gpu_pointer,vec_in_gpu_pointer,QUART_gpu_pointer,
  diagsz_gpu_pointer,noffsz_gpu_pointer,rankoffsz_gpu_pointer,offdiagsz_gpu_pointer,offdia_gpu_pointer,use_texture_lr); 
  hipEventSynchronize(0); hipDeviceSynchronize();
  if(verbose==1) printf(" .....done.....  \n ");

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

     hipMemcpy(vec_out,vec_out_gpu,size*sizeof(double),hipMemcpyHostToHost);
     hipEventSynchronize(0);
     hipHostFree(vec_out_gpu);
     if(use_texture_lr==0){
      hipHostFree(vec_in_gpu);}
     else{
      unbind_x();
      hipFree(vec_in_gpu_pointer);
     }
     hipHostFree(offdia_gpu);
     hipHostFree(QUART_gpu);
     hipHostFree(diagsz_gpu);
     hipHostFree(rankoffsz_gpu);
     hipHostFree(offdiagsz_gpu);
     hipHostFree(noffsz_gpu);
     hipEventSynchronize(0);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

 void one_step_lanczos_cuda(int blocksize, int Niter, int size, int iter, double *diag, 
 double *subdiag, double *vec_tmp_gpu, double *vec_in_gpu, double *vec_out_gpu, double *vec_tmp_gpu_pointer, 
 double *vec_in_gpu_pointer, double *vec_out_gpu_pointer, double *QUART_gpu_pointer, double *diagsz_gpu_pointer, 
 int *noffsz_gpu_pointer, int *rankoffsz_gpu_pointer, double *offdiagsz_gpu_pointer, int *offdia_gpu_pointer, 
 double *QUART_gpu, double *diagsz_gpu, int *noffsz_gpu, int *rankoffsz_gpu, double *offdiagsz_gpu, int *offdia_gpu)

{
    int verbose=0 ; int psize=size; 

    int nb=(size-size % blocksize)/blocksize+1 ; int ngrid=nb/MAX_BLOCK; if(ngrid>0)nb=MAX_BLOCK; dim3 bl(1,nb),th(WARP_SIZE,blocksize);

    if(verbose==1) printf( " \n --------------- \n  Nblock=%d \n Ngrid=%d \n blocksize=%d \n ----------------- \n ",nb,ngrid,blocksize);

    if(verbose==1) printf ( " Sdot, vec in norm \n ");
    double normv = sqrt(hipblasDdot(size,vec_in_gpu_pointer,1,vec_in_gpu_pointer,1)); hipEventSynchronize(0); hipDeviceSynchronize();
    if(verbose==1) printf( " norm=%f \n ", normv);

    hipblasDscal(size,1.0/normv,vec_in_gpu_pointer,1); hipEventSynchronize(0); hipDeviceSynchronize(); 

    if(verbose==1) printf( " call kernel ... \n ");
    Hmult_ker<<<bl,th>>>(ngrid,blocksize,psize,vec_out_gpu_pointer,vec_in_gpu_pointer,QUART_gpu_pointer,diagsz_gpu_pointer,
                         noffsz_gpu_pointer,rankoffsz_gpu_pointer,offdiagsz_gpu_pointer,offdia_gpu_pointer,use_texture_lr); 

   if(verbose==1) printf( " done.... \n " );
   hipEventSynchronize(0); hipDeviceSynchronize();

   if(iter>0){hipblasDaxpy(size,-subdiag[iter],vec_tmp_gpu_pointer,1,vec_out_gpu_pointer,1);}; hipEventSynchronize(0); hipDeviceSynchronize();

  if(use_texture_lr==0){
   hipMemcpy(vec_tmp_gpu,vec_in_gpu,size*sizeof(double),hipMemcpyHostToHost);hipEventSynchronize(0);hipDeviceSynchronize();
  }else{
   hipMemcpy(vec_tmp_gpu,vec_in_gpu_pointer,size*sizeof(double),hipMemcpyDeviceToHost);hipEventSynchronize(0);hipDeviceSynchronize();
  }

   diag[iter]=hipblasDdot(size, vec_out_gpu_pointer,1,vec_in_gpu_pointer,1);hipEventSynchronize(0);hipDeviceSynchronize();

   hipblasDaxpy(size,-diag[iter],vec_tmp_gpu_pointer,1,vec_out_gpu_pointer,1); hipEventSynchronize(0); hipDeviceSynchronize();
   normv = sqrt(hipblasDdot(size, vec_out_gpu_pointer,1,vec_out_gpu_pointer,1)); hipEventSynchronize(0); hipDeviceSynchronize();

   if(iter<Niter-1) subdiag[iter+1]=normv;

  if(use_texture_lr==0){
   hipMemcpy( vec_in_gpu, vec_out_gpu, size*sizeof(double), hipMemcpyHostToHost); hipEventSynchronize(0); hipDeviceSynchronize();
  }else{
   hipMemcpy( vec_in_gpu_pointer, vec_out_gpu, size*sizeof(double), hipMemcpyHostToDevice); hipEventSynchronize(0); hipDeviceSynchronize();
  }
 
}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

extern "C" void lanczos_real_dynamic_cuda_(int *pblocksize, int *Niter_lanczos_,int *offdiasize, int *roff, int *ntot, 
double *QUART, double *diagsz, int *noffsz, int *rankoffsz, double *offdiagsz, double *diag, double *subdiag , 
double *vecinit)
{

 //---------------------------------------------------------------------------------------//
     int verbose=0;
 //---------------------------------------------------------------------------------------//

     int Niter_lanczos= *Niter_lanczos_; 

     if(verbose==1) printf(" start Lanczos Real on GPU \n" );

     int size = *ntot; int blocksize= *pblocksize; 

     double   *vec_in_gpu,*vec_out_gpu,*QUART_gpu,*diagsz_gpu,*offdiagsz_gpu,*vec_tmp_gpu;
     int      *noffsz_gpu,*rankoffsz_gpu;
     double   *vec_in_gpu_pointer,*vec_out_gpu_pointer,*vec_tmp_gpu_pointer;
     double   *QUART_gpu_pointer,*diagsz_gpu_pointer,*offdiagsz_gpu_pointer;
     int      *noffsz_gpu_pointer,*rankoffsz_gpu_pointer;
     int      *offdia_gpu, *offdia_gpu_pointer;

     if(verbose==1) printf(" GPU get eigenvalues, size of Lanczos vectors = %d ", size);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

       hipSetDevice(0); hipSetDeviceFlags( hipDeviceMapHost ); hipEventSynchronize(0);

       hipHostAlloc((void**)&vec_tmp_gpu   , size*sizeof(double),        hipHostMallocMapped | hipHostMallocPortable );
  
       if(use_texture_lr==1){
         hipMalloc((void**)&vec_in_gpu_pointer,sizeof(double)*size);
         bind_x(vec_in_gpu_pointer,size);
       }else{
         hipHostAlloc((void**)&vec_in_gpu    , size*sizeof(double),        hipHostMallocMapped | hipHostMallocPortable );
         hipHostGetDevicePointer((void**)  &vec_in_gpu_pointer  ,  vec_in_gpu    , 0 );
       }

       hipHostAlloc((void**)&vec_out_gpu   , size*sizeof(double),        hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&QUART_gpu     , size*sizeof(double),        hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&diagsz_gpu    , size*sizeof(double),        hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&offdiagsz_gpu , *offdiasize*sizeof(double), hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&noffsz_gpu    , size*sizeof(int)   ,        hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&rankoffsz_gpu , *roff*sizeof(int)  ,        hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&offdia_gpu    , size*sizeof(int)   ,        hipHostMallocMapped | hipHostMallocPortable );
       hipEventSynchronize(0);

       hipHostGetDevicePointer((void**)  &vec_tmp_gpu_pointer ,  vec_tmp_gpu   , 0 );
       hipHostGetDevicePointer((void**)  &vec_out_gpu_pointer ,  vec_out_gpu   , 0 );
       hipHostGetDevicePointer((void**)  &QUART_gpu_pointer ,    QUART_gpu     , 0 );
       hipHostGetDevicePointer((void**)  &diagsz_gpu_pointer ,   diagsz_gpu    , 0 );
       hipHostGetDevicePointer((void**)  &offdiagsz_gpu_pointer, offdiagsz_gpu , 0 );
       hipHostGetDevicePointer((void**)  &noffsz_gpu_pointer,    noffsz_gpu    , 0 );
       hipHostGetDevicePointer((void**)  &rankoffsz_gpu_pointer, rankoffsz_gpu , 0 );
       hipHostGetDevicePointer((void**)  &offdia_gpu_pointer,    offdia_gpu    , 0 );
       hipEventSynchronize(0);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//
       hipMemcpy(QUART_gpu,     QUART,       size*sizeof(double),        hipMemcpyHostToHost);
       hipMemcpy(diagsz_gpu,    diagsz,      size*sizeof(double),        hipMemcpyHostToHost);
       hipMemcpy(offdiagsz_gpu, offdiagsz,   *offdiasize*sizeof(double), hipMemcpyHostToHost);
       hipMemcpy(noffsz_gpu,    noffsz ,     size*sizeof(int),           hipMemcpyHostToHost);
       hipMemcpy(rankoffsz_gpu, rankoffsz,   *roff*sizeof(int),          hipMemcpyHostToHost);
       hipEventSynchronize(0);
       offdia_gpu[0]=0; for(int istate=1; istate<size; istate++) { offdia_gpu[istate]=offdia_gpu[istate-1]+noffsz[istate-1]; };
       hipEventSynchronize(0);
 //---------------------------------------------------------------------------------------//

  if(verbose==1) printf( " initialized, now run actual Lanczos \n " );

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

  hipDeviceSynchronize(); hipEventSynchronize(0); 

 if(use_texture_lr==0){
   hipMemcpy(vec_in_gpu,vecinit,size*sizeof(double),hipMemcpyHostToHost);
 }else{
   hipMemcpy(vec_out_gpu,vecinit,size*sizeof(double),hipMemcpyHostToHost);
   hipMemcpy(vec_in_gpu_pointer,vec_out_gpu,size*sizeof(double),hipMemcpyHostToDevice);
 }

  hipDeviceSynchronize(); hipEventSynchronize(0);

  for(int iter=0;iter<Niter_lanczos;iter++){
   if(verbose==1) printf( " Lanczos iteration %d / %d \n", iter,Niter_lanczos);
   one_step_lanczos_cuda(blocksize,Niter_lanczos,size,iter,diag,subdiag,vec_tmp_gpu,vec_in_gpu,vec_out_gpu,vec_tmp_gpu_pointer,vec_in_gpu_pointer,vec_out_gpu_pointer,QUART_gpu_pointer,diagsz_gpu_pointer,noffsz_gpu_pointer,rankoffsz_gpu_pointer,offdiagsz_gpu_pointer,offdia_gpu_pointer,QUART_gpu,diagsz_gpu,noffsz_gpu,rankoffsz_gpu, offdiagsz_gpu, offdia_gpu);

  };

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

     hipEventSynchronize(0);
     hipHostFree(vec_tmp_gpu);
     if(use_texture_lr==0){
      hipHostFree(vec_in_gpu);}
     else{
      unbind_x();
      hipFree(vec_in_gpu_pointer);
     }
     hipHostFree(vec_out_gpu);
     hipHostFree(offdia_gpu);
     hipHostFree(QUART_gpu);
     hipHostFree(diagsz_gpu);
     hipHostFree(rankoffsz_gpu);
     hipHostFree(offdiagsz_gpu);
     hipHostFree(noffsz_gpu);
     hipEventSynchronize(0);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

extern "C" void lanczos_real_cuda_(int *pblocksize,  int *Niter_lanczos_,int *offdiasize, int *roff, int *ntot, 
double *QUART, double *diagsz, int *noffsz, int *rankoffsz, double *offdiagsz, double *diag, double *subdiag )
{

 //---------------------------------------------------------------------------------------//
     int verbose=0;
 //---------------------------------------------------------------------------------------//

     int Niter_lanczos= *Niter_lanczos_; 
     if(verbose==1) printf(" start Lanczos Real on GPU \n" );
     int size = *ntot; int blocksize = *pblocksize;

     double   *vec_in_gpu,*vec_out_gpu,*QUART_gpu,*diagsz_gpu,*offdiagsz_gpu,*vec_tmp_gpu;
     int      *noffsz_gpu,*rankoffsz_gpu;
     double   *vec_in_gpu_pointer,*vec_out_gpu_pointer,*vec_tmp_gpu_pointer;
     double   *QUART_gpu_pointer,*diagsz_gpu_pointer,*offdiagsz_gpu_pointer;
     int      *noffsz_gpu_pointer,*rankoffsz_gpu_pointer;
     int      *offdia_gpu, *offdia_gpu_pointer;


     if(verbose==1) printf(" GPU get eigenvalues, size of Lanczos vectors = %d \n", size);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

       hipSetDevice(0); hipSetDeviceFlags( hipDeviceMapHost ); hipEventSynchronize(0);

       if(use_texture_lr==1){
         if(verbose==1) printf(" GPU go for texture \n");
         hipMalloc((void**)&vec_in_gpu_pointer,sizeof(double)*size);
         bind_x(vec_in_gpu_pointer,size);
       }else{
         if(verbose==1) printf(" GPU HostAlloc ");
         hipHostAlloc((void**)&vec_in_gpu    , size*sizeof(double),        hipHostMallocMapped | hipHostMallocPortable );
         hipHostGetDevicePointer((void**) &vec_in_gpu_pointer ,vec_in_gpu , 0 );
       }

       if(verbose==1) printf(" GPU Host Alloc arrays \n");
       hipHostAlloc((void**)&vec_tmp_gpu   , size*sizeof(double),        hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&vec_out_gpu   , size*sizeof(double),        hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&QUART_gpu     , size*sizeof(double),        hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&diagsz_gpu    , size*sizeof(double),        hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&offdiagsz_gpu , *offdiasize*sizeof(double), hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&noffsz_gpu    , size*sizeof(int)   ,        hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&rankoffsz_gpu , *roff*sizeof(int)  ,        hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&offdia_gpu    , size*sizeof(int)   ,        hipHostMallocMapped | hipHostMallocPortable );
       hipEventSynchronize(0);

       if(verbose==1) printf(" GPU allocate Device Pointer \n");
       hipHostGetDevicePointer((void**)   &vec_tmp_gpu_pointer ,  vec_tmp_gpu    , 0 );
       hipHostGetDevicePointer((void**)   &vec_out_gpu_pointer ,  vec_out_gpu    , 0 );
       hipHostGetDevicePointer((void**)   &QUART_gpu_pointer ,    QUART_gpu      , 0 );
       hipHostGetDevicePointer((void**)   &diagsz_gpu_pointer ,   diagsz_gpu     , 0 );
       hipHostGetDevicePointer((void**)   &offdiagsz_gpu_pointer, offdiagsz_gpu  , 0 );
       hipHostGetDevicePointer((void**)   &noffsz_gpu_pointer,    noffsz_gpu     , 0 );
       hipHostGetDevicePointer((void**)   &rankoffsz_gpu_pointer, rankoffsz_gpu  , 0 );
       hipHostGetDevicePointer((void**)   &offdia_gpu_pointer,    offdia_gpu     , 0 );
       hipEventSynchronize(0);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//
       if(verbose==1) printf(" GPU MemCpy arrays \n");
       hipMemcpy(QUART_gpu,     QUART,       size*sizeof(double),        hipMemcpyHostToHost);
       if(verbose==1) printf(" GPU MemCpy arrays diagsz \n");
       hipMemcpy(diagsz_gpu,    diagsz,      size*sizeof(double),        hipMemcpyHostToHost);
       if(verbose==1) printf(" GPU MemCpy arrays offdiagsz \n");
       hipMemcpy(offdiagsz_gpu, offdiagsz,   *offdiasize*sizeof(double), hipMemcpyHostToHost);
       if(verbose==1) printf(" GPU MemCpy arrays noffsz \n");
       hipMemcpy(noffsz_gpu,    noffsz ,     size*sizeof(int),           hipMemcpyHostToHost);
       if(verbose==1) printf(" GPU MemCpy arrays rankoffsz \n");
       hipMemcpy(rankoffsz_gpu, rankoffsz,   *roff*sizeof(int),          hipMemcpyHostToHost);
       if(verbose==1) printf(" Build up offdiag GPU \n");
       hipEventSynchronize(0);
       offdia_gpu[0]=0; for(int istate=1; istate<size; istate++) { offdia_gpu[istate]=offdia_gpu[istate-1]+noffsz[istate-1]; };
       hipEventSynchronize(0);
 //---------------------------------------------------------------------------------------//

  if(verbose==1) printf( " initialized, now run actual Lanczos \n " );

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

 if(use_texture_lr==0){
  for(int i=0;i<size;i++) vec_in_gpu[i]=1.0; hipDeviceSynchronize();
 }else{
   for(int i=0;i<size;i++) vec_out_gpu[i]=1.0; hipDeviceSynchronize();
   hipMemcpy(vec_in_gpu_pointer,vec_out_gpu,size*sizeof(double),hipMemcpyHostToDevice);
 }

  for(int iter=0;iter<Niter_lanczos;iter++){
    if(verbose==1) printf( " Lanczos iteration %d / %d \n", iter,Niter_lanczos);
    one_step_lanczos_cuda(blocksize,Niter_lanczos,size,iter,diag,subdiag,vec_tmp_gpu,vec_in_gpu,vec_out_gpu,vec_tmp_gpu_pointer,vec_in_gpu_pointer,vec_out_gpu_pointer,QUART_gpu_pointer,diagsz_gpu_pointer,noffsz_gpu_pointer,rankoffsz_gpu_pointer,offdiagsz_gpu_pointer,offdia_gpu_pointer,QUART_gpu,diagsz_gpu,noffsz_gpu,rankoffsz_gpu, offdiagsz_gpu, offdia_gpu);
  };

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

     hipEventSynchronize(0);
     hipHostFree(vec_tmp_gpu);
     if(use_texture_lr==0){
      hipHostFree(vec_in_gpu);}
     else{
      unbind_x();
      hipFree(vec_in_gpu_pointer);
     }
     hipHostFree(vec_out_gpu);
     hipHostFree(offdia_gpu);
     hipHostFree(QUART_gpu);
     hipHostFree(diagsz_gpu);
     hipHostFree(rankoffsz_gpu);
     hipHostFree(offdiagsz_gpu);
     hipHostFree(noffsz_gpu);
     hipEventSynchronize(0);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

extern "C" void lanczos_real_get_gs_cuda_(int *pblocksize, int *Niter_lanczos_,int *offdiasize, int *roff, 
int *ntot, double *QUART, double *diagsz, int *noffsz, int *rankoffsz, double *offdiagsz, double *vecp, double *GS)
{

 //---------------------------------------------------------------------------------------//
     int verbose=0;
 //---------------------------------------------------------------------------------------//

     if(verbose==1) printf(" start Lanczos Real on GPU \n" );
     int Niter_lanczos= *Niter_lanczos_;
     double diag[Niter_lanczos], subdiag[Niter_lanczos];
     int size = *ntot; int blocksize = *pblocksize;
     double *vec_in_gpu,*vec_out_gpu,*QUART_gpu,*diagsz_gpu,*offdiagsz_gpu,*vec_tmp_gpu;
     int *noffsz_gpu,*rankoffsz_gpu;
     double *vec_in_gpu_pointer,*vec_out_gpu_pointer,*vec_tmp_gpu_pointer;
     double *QUART_gpu_pointer,*diagsz_gpu_pointer,*offdiagsz_gpu_pointer;
     int *noffsz_gpu_pointer,*rankoffsz_gpu_pointer;
     int *offdia_gpu, *offdia_gpu_pointer;

     double *GS_gpu, *GS_gpu_pointer;

     if(verbose==1) printf(" GPU get Ground State, size of Lanczos vectors = %d ", size);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

       hipSetDevice(0); hipSetDeviceFlags( hipDeviceMapHost ); hipEventSynchronize(0);

       hipHostAlloc((void**)&GS_gpu        , size*sizeof(double),        hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&vec_tmp_gpu   , size*sizeof(double),        hipHostMallocMapped | hipHostMallocPortable );

       if(use_texture_lr==1){
        hipMalloc((void**)&vec_in_gpu_pointer,sizeof(double)*size);
        bind_x(vec_in_gpu_pointer,size);
      }else{
        hipHostAlloc((void**)&vec_in_gpu    , size*sizeof(double),        hipHostMallocMapped | hipHostMallocPortable );
        hipHostGetDevicePointer((void**)   &vec_in_gpu_pointer  ,  vec_in_gpu     , 0 );
      }

       hipHostAlloc((void**)&vec_out_gpu   , size*sizeof(double),        hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&QUART_gpu     , size*sizeof(double),        hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&diagsz_gpu    , size*sizeof(double),        hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&offdiagsz_gpu , *offdiasize*sizeof(double), hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&noffsz_gpu    , size*sizeof(int)   ,        hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&rankoffsz_gpu , *roff*sizeof(int)  ,        hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&offdia_gpu    , size*sizeof(int)   ,        hipHostMallocMapped | hipHostMallocPortable );
       hipEventSynchronize(0);

       hipHostGetDevicePointer((void**)   &GS_gpu_pointer ,       GS_gpu         , 0 );
       hipHostGetDevicePointer((void**)   &vec_tmp_gpu_pointer ,  vec_tmp_gpu    , 0 );
       hipHostGetDevicePointer((void**)   &vec_out_gpu_pointer ,  vec_out_gpu    , 0 );
       hipHostGetDevicePointer((void**)   &QUART_gpu_pointer ,    QUART_gpu      , 0 );
       hipHostGetDevicePointer((void**)   &diagsz_gpu_pointer ,   diagsz_gpu     , 0 );
       hipHostGetDevicePointer((void**)   &offdiagsz_gpu_pointer, offdiagsz_gpu  , 0 );
       hipHostGetDevicePointer((void**)   &noffsz_gpu_pointer,    noffsz_gpu     , 0 );
       hipHostGetDevicePointer((void**)   &rankoffsz_gpu_pointer, rankoffsz_gpu  , 0 );
       hipHostGetDevicePointer((void**)   &offdia_gpu_pointer,    offdia_gpu     , 0 );
       hipEventSynchronize(0);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//
       hipMemcpy(QUART_gpu,     QUART,       size*sizeof(double),        hipMemcpyHostToHost);
       hipMemcpy(diagsz_gpu,    diagsz,      size*sizeof(double),        hipMemcpyHostToHost);
       hipMemcpy(offdiagsz_gpu, offdiagsz,   *offdiasize*sizeof(double), hipMemcpyHostToHost);
       hipMemcpy(noffsz_gpu,    noffsz ,     size*sizeof(int),           hipMemcpyHostToHost);
       hipMemcpy(rankoffsz_gpu, rankoffsz,   *roff*sizeof(int),          hipMemcpyHostToHost);
       hipEventSynchronize(0);
       offdia_gpu[0]=0; for(int istate=1; istate<size; istate++) { offdia_gpu[istate]=offdia_gpu[istate-1]+noffsz[istate-1]; };
       hipEventSynchronize(0);
 //---------------------------------------------------------------------------------------//

  if(verbose==1) printf( " initialized, now run actual Lanczos \n " );

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

  memset ((void **)GS_gpu, 0, size*sizeof(double));

 if(use_texture_lr==0){
  for(int i=0;i<size;i++) vec_in_gpu[i]=1.0; hipDeviceSynchronize();
 }else{
   for(int i=0;i<size;i++) vec_out_gpu[i]=1.0; hipDeviceSynchronize();
   hipMemcpy(vec_in_gpu_pointer,vec_out_gpu,size*sizeof(double),hipMemcpyHostToDevice);
 }

  double coef= 1.0/sqrt(hipblasDdot(size,vec_in_gpu_pointer,1,vec_in_gpu_pointer,1))*vecp[0]; hipDeviceSynchronize();  hipEventSynchronize(0);
  hipblasDaxpy(size,coef,vec_in_gpu_pointer,1,GS_gpu_pointer,1); hipDeviceSynchronize(); hipEventSynchronize(0);hipDeviceSynchronize();

   one_step_lanczos_cuda(blocksize, Niter_lanczos,size,0,diag,subdiag,vec_tmp_gpu,vec_in_gpu,vec_out_gpu,vec_tmp_gpu_pointer,vec_in_gpu_pointer,vec_out_gpu_pointer,QUART_gpu_pointer,diagsz_gpu_pointer,noffsz_gpu_pointer,rankoffsz_gpu_pointer,offdiagsz_gpu_pointer,offdia_gpu_pointer,QUART_gpu,diagsz_gpu,noffsz_gpu,rankoffsz_gpu, offdiagsz_gpu, offdia_gpu);

  for(int iter=1;iter<Niter_lanczos-1;iter++){

   coef=1.0/sqrt(hipblasDdot(size,vec_out_gpu_pointer,1,vec_out_gpu_pointer,1)); hipblasDscal (size,coef,vec_out_gpu_pointer,1);  hipEventSynchronize(0);
   hipblasDaxpy(size,vecp[iter],vec_out_gpu_pointer,1,GS_gpu_pointer,1); hipDeviceSynchronize(); hipEventSynchronize(0);

   one_step_lanczos_cuda(blocksize,Niter_lanczos,size,iter,diag,subdiag,vec_tmp_gpu,vec_in_gpu,vec_out_gpu,vec_tmp_gpu_pointer,vec_in_gpu_pointer,vec_out_gpu_pointer,QUART_gpu_pointer,diagsz_gpu_pointer,noffsz_gpu_pointer,rankoffsz_gpu_pointer,offdiagsz_gpu_pointer,offdia_gpu_pointer,QUART_gpu,diagsz_gpu,noffsz_gpu,rankoffsz_gpu, offdiagsz_gpu, offdia_gpu);  hipEventSynchronize(0);hipDeviceSynchronize();
  };

   coef=1.0/sqrt(hipblasDdot(size,vec_out_gpu_pointer,1,vec_out_gpu_pointer,1)); hipblasDscal(size,coef,vec_out_gpu_pointer,1);hipEventSynchronize(0);
   hipblasDaxpy(size,vecp[Niter_lanczos-1],vec_out_gpu_pointer,1,GS_gpu_pointer,1); hipDeviceSynchronize(); hipEventSynchronize(0); 

   coef=1.0/sqrt(hipblasDdot(size,GS_gpu_pointer,1,GS_gpu_pointer,1)); hipDeviceSynchronize();  hipEventSynchronize(0);
   hipblasDscal(size,coef,GS_gpu_pointer,1); hipEventSynchronize(0); hipDeviceSynchronize();

   hipMemcpy(GS,GS_gpu,size*sizeof(double),hipMemcpyHostToHost);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

     hipEventSynchronize(0);
     hipHostFree(vec_tmp_gpu);
     if(use_texture_lr==0){
      hipHostFree(vec_in_gpu);}
     else{
      unbind_x();
      hipFree(vec_in_gpu_pointer);
     }
     hipHostFree(vec_out_gpu);
     hipHostFree(offdia_gpu);
     hipHostFree(QUART_gpu);
     hipHostFree(diagsz_gpu);
     hipHostFree(rankoffsz_gpu);
     hipHostFree(offdiagsz_gpu);
     hipHostFree(noffsz_gpu);
     hipEventSynchronize(0);

     hipHostFree(GS_gpu); hipEventSynchronize(0);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
