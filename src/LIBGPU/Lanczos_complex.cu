#include "hip/hip_runtime.h"

 #include <stdio.h>
 #include <stdlib.h>
 #include <hip/hip_runtime.h>
 #include <hipblas.h>
 #include <hip/hip_complex.h>

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

const int use_texture_lc = 1 ;

#define WARP_SIZE 32
#define MAX_BLOCK 65500

texture<int2,1,hipReadModeElementType> tex;

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

inline void cublasZscal_no_device (int n, hipDoubleComplex alpha, hipDoubleComplex *y, int incy)
 { for (int i=0; i<n; i++) { y[i] = hipCmul( alpha, y[i] ); } }

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

inline void cublasZaxpy_no_device (int n, hipDoubleComplex alpha, hipDoubleComplex *x, int incx,hipDoubleComplex *y, int incy)
 { for (int i=0; i<n; i++) { y[i] = hipCadd( y[i],  hipCmul( alpha, x[i] ) ); } }

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

inline hipDoubleComplex cublasZdotu_no_device (int n, hipDoubleComplex *x, int incx, hipDoubleComplex *y, int incy)
 { 
  hipDoubleComplex dot_; dot_=make_hipDoubleComplex(0.0,0.0) ;
  for (int i=0; i<n; i++) { dot_ = hipCadd(dot_,hipCmul( hipConj(x[i]), y[i] ) ); } 
  return dot_;
 }

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

inline void   bind_x(hipDoubleComplex *x, int N) {    hipBindTexture(0,tex,x,N*sizeof(hipDoubleComplex)); };
inline void unbind_x()                          {  hipUnbindTexture(  tex  ); };

__inline__  __device__ hipDoubleComplex fetch_x(const int& i)
  {  
         int  jj = 2*(i-1); 
         int2 v  = tex1Dfetch(tex,jj); 
      double rr  = __hiloint2double(v.y, v.x); 
              v  = tex1Dfetch(tex,jj+1);
      double im  = __hiloint2double(v.y, v.x);  
      return make_hipDoubleComplex(rr,im);
  }

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

  //************************************************//
  //               Kernel Hmult                     //
  //************************************************//

  __global__  void Hmult_ker_complex_(int ngrid,int blocksize, int psize, hipDoubleComplex *vec_out, 
                const hipDoubleComplex *vec_in,  const double *QUART, const double *diagsz, const int *noffsz, const int *rankoffsz, 
		const hipDoubleComplex *offdiagsz, const int *offdia, const int use_texture_lc)
{
   int nhoffdiag,jstate,noff; 
   hipDoubleComplex hoffdiag,tmp; 
   int istate;

   for(int iii=0;iii<=ngrid;iii++){

    istate = (blockIdx.x+iii*MAX_BLOCK)*blocksize + threadIdx.x ;

    if(istate < psize ){
    tmp                =  make_hipDoubleComplex(QUART[istate]+diagsz[istate],0.0);
    vec_out[istate]    =  hipCmul(tmp,vec_in[istate]); 
            noff       =  noffsz[istate] ;
            nhoffdiag  =  offdia[istate] ;
 
   for(int irank=0;irank<noff;irank++)
    { jstate           =  rankoffsz[nhoffdiag+irank]  ;
      hoffdiag         =  hipConj(offdiagsz[nhoffdiag+irank]);
      vec_out[istate]  =  hipCadd(  vec_out[istate],  hipCmul(hoffdiag,vec_in[jstate-1]) ) ;
    };
  };
  };
}

  //************************************************//
  //               Kernel Hmult                     //
  //************************************************//

     //---------------------------------------//
     //---------------------------------------//
     //---------------------------------------//
     //---------------------------------------//

  __global__  void Hmult_ker_complex(int ngrid,int BLOCK_SIZE, int num_rows, hipDoubleComplex *y,
         const hipDoubleComplex *x, const double *QUART, const double *diagsz, const int *noffsz, const int *Aj,
         const hipDoubleComplex *Ax, const int *offdia, const int use_texture_lc)
{
   __shared__ hipDoubleComplex   sdata[16][WARP_SIZE];
   __shared__ int                ptrs[32][2];
   __shared__ double            temp2[32][2];

   const int warp_lane   = threadIdx.y ; const int thread_lane = threadIdx.x ; 

   int row_start; int row_end; int jj, row;

   for(int iii=0;iii<=ngrid;iii++){

    row = BLOCK_SIZE * (blockIdx.y + iii*MAX_BLOCK) + threadIdx.y;

    if(row<num_rows)
   {
        if(thread_lane==0)   ptrs[warp_lane][0] = offdia[row];
        if(thread_lane==1)   ptrs[warp_lane][1] = noffsz[row];
        if(thread_lane==2)  temp2[warp_lane][0] = QUART[row];
        if(thread_lane==3)  temp2[warp_lane][1] = diagsz[row];

        row_start = ptrs[warp_lane][0] ; row_end = ptrs[warp_lane][1]+row_start ;

        if(use_texture_lc==0)  {
          y[row] = hipCmul( make_hipDoubleComplex( temp2[warp_lane][0]+temp2[warp_lane][1],0.0 )  , x[row]);
        }else{
          y[row] = hipCmul( make_hipDoubleComplex( temp2[warp_lane][0]+temp2[warp_lane][1],0.0 )  , fetch_x(row+1) );
        };

        sdata[threadIdx.y][threadIdx.x]=make_hipDoubleComplex(0.0,0.0);

        if(use_texture_lc==1)
        {
        for(jj=row_start+thread_lane;jj<row_end;jj+=WARP_SIZE) 
           sdata[threadIdx.y][threadIdx.x]=hipCadd(sdata[threadIdx.y][threadIdx.x], hipCmul(hipConj(Ax[jj]),fetch_x(Aj[jj])));
        }else{
        for(jj=row_start+thread_lane;jj<row_end;jj+=WARP_SIZE) 
           sdata[threadIdx.y][threadIdx.x]=hipCadd(sdata[threadIdx.y][threadIdx.x], hipCmul(hipConj(Ax[jj]),x[Aj[jj]-1]));
        };

        if (thread_lane < 16) { sdata[threadIdx.y][threadIdx.x] = hipCadd(sdata[threadIdx.y][threadIdx.x], sdata[threadIdx.y][threadIdx.x + 16] ); };
        if (thread_lane <  8) { sdata[threadIdx.y][threadIdx.x] = hipCadd(sdata[threadIdx.y][threadIdx.x], sdata[threadIdx.y][threadIdx.x +  8] ); };
        if (thread_lane <  4) { sdata[threadIdx.y][threadIdx.x] = hipCadd(sdata[threadIdx.y][threadIdx.x], sdata[threadIdx.y][threadIdx.x +  4] ); };
        if (thread_lane <  2) { sdata[threadIdx.y][threadIdx.x] = hipCadd(sdata[threadIdx.y][threadIdx.x], sdata[threadIdx.y][threadIdx.x +  2] ); };
        if (thread_lane <  1) { sdata[threadIdx.y][threadIdx.x] = hipCadd(sdata[threadIdx.y][threadIdx.x], sdata[threadIdx.y][threadIdx.x +  1] ); };
        if (thread_lane == 0)   y[row] = hipCadd(y[row],sdata[threadIdx.y][threadIdx.x]) ;
   }; }; }

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
__global__ void norm_vec_ker(int size , hipDoubleComplex *x, double *normv)
{
    int blocksize = blockDim.x ;
   __shared__ double  temp[512]; __shared__ double temp2[512]; int row; int  ii;

    temp[threadIdx.x]=0.;
    for(row=threadIdx.x; row<size; row+=blocksize)
   {
    temp2[threadIdx.x] = hipCabs(x[row]); temp2[threadIdx.x]*= temp2[threadIdx.x];
    temp[threadIdx.x] += temp2[threadIdx.x];
   };
   __syncthreads ();
    *normv=0.0; for(ii=0;ii<blocksize;ii++){ *normv+=temp[ii];}; *normv=sqrt(*normv); 
}
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
__global__ void real_scal_vec_ker( int size , hipDoubleComplex *x, hipDoubleComplex *y, double *normv)
{
   int blocksize = blockDim.x ; __shared__ double temp[512]; int row , ii; 

   temp[threadIdx.x]=0.0;
   for(row=threadIdx.x; row<size; row+=blocksize) { temp[threadIdx.x] += hipCreal( hipCmul(  hipConj( x[row] ) ,y[row]  ) ); };

  __syncthreads ();

   *normv=0.0; for(ii=0;ii<blocksize ;ii++){ *normv+=temp[ii];};
}
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

__global__ void normalize_vec_ker(int ngrid,  int size ,hipDoubleComplex *x, double *normv)
{
    int blocksize = blockDim.x ;
    int row ;
    for(int iii=0;iii<=ngrid;iii++){
    row = blocksize * (blockIdx.x+iii*MAX_BLOCK) + threadIdx.x ;
    if(row<size) { x[row]=hipCmul( make_hipDoubleComplex(1.0 / *normv ,0.0), x[row] ); };};
}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

extern "C" void hmult_sz_complex_cuda_rout_(int *pblocksize, int *offdiasize, int *roff, int* ntot, 
double* QUART, double* diagsz, hipDoubleComplex* vec_in, hipDoubleComplex* vec_out, int* noffsz, int* rankoffsz, 
hipDoubleComplex* offdiagsz )
{

 //---------------------------------------------------------------------------------------//
     int verbose=0;
 //---------------------------------------------------------------------------------------//

     if(verbose==1) printf(" start Hmult GPU \n" );

     int blocksize=*pblocksize; int size = *ntot; 

    int nb=(size-size % blocksize)/blocksize+1; int ngrid=nb/MAX_BLOCK; if(ngrid>0) nb=MAX_BLOCK;
    dim3 bl(1,nb),th(WARP_SIZE,blocksize);

    if(verbose==1) printf( " --------------- \n  Nblock=%d Ngrid=%d \n ----------------- \n ",nb,ngrid);

     hipDoubleComplex  *vec_in_gpu,*vec_out_gpu;
     double           *QUART_gpu,*diagsz_gpu;
     hipDoubleComplex  *offdiagsz_gpu;
     int              *noffsz_gpu,*rankoffsz_gpu;
     hipDoubleComplex  *vec_in_gpu_pointer,*vec_out_gpu_pointer;
     double           *QUART_gpu_pointer,*diagsz_gpu_pointer;
     hipDoubleComplex  *offdiagsz_gpu_pointer;
     int              *noffsz_gpu_pointer,*rankoffsz_gpu_pointer;
     int              *offdia_gpu, *offdia_gpu_pointer;

     if(verbose==1) printf(" GPU , size of Lanczos vector = %d ", size);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

       hipSetDevice(0); hipSetDeviceFlags( hipDeviceMapHost ); hipEventSynchronize(0);

       if(use_texture_lc==1){
         hipMalloc((void**)&vec_in_gpu_pointer,sizeof(hipDoubleComplex)*size);
         bind_x(vec_in_gpu_pointer,size);
       }else{
         hipHostAlloc((void**)&vec_in_gpu    , size*sizeof(hipDoubleComplex),  hipHostMallocMapped | hipHostMallocPortable );
         hipHostGetDevicePointer((void**)  &vec_in_gpu_pointer  ,  vec_in_gpu    , 0 );
       }

       hipHostAlloc((void**)&vec_out_gpu   , size*sizeof(hipDoubleComplex),        hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&QUART_gpu     , size*sizeof(double),                 hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&diagsz_gpu    , size*sizeof(double),                 hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&offdiagsz_gpu , *offdiasize*sizeof(hipDoubleComplex), hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&noffsz_gpu    , size*sizeof(int)   ,                 hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&rankoffsz_gpu , *roff*sizeof(int)  ,                 hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&offdia_gpu    , size*sizeof(int)   ,                 hipHostMallocMapped | hipHostMallocPortable );
       hipEventSynchronize(0);

       hipHostGetDevicePointer((void**)   &vec_out_gpu_pointer ,  vec_out_gpu    , 0 );
       hipHostGetDevicePointer((void**)   &QUART_gpu_pointer ,    QUART_gpu      , 0 );
       hipHostGetDevicePointer((void**)   &diagsz_gpu_pointer ,   diagsz_gpu     , 0 );
       hipHostGetDevicePointer((void**)   &offdiagsz_gpu_pointer, offdiagsz_gpu  , 0 );
       hipHostGetDevicePointer((void**)   &noffsz_gpu_pointer,    noffsz_gpu     , 0 );
       hipHostGetDevicePointer((void**)   &rankoffsz_gpu_pointer, rankoffsz_gpu  , 0 );
       hipHostGetDevicePointer((void**)   &offdia_gpu_pointer,    offdia_gpu     , 0 );
       hipEventSynchronize(0);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

      if(use_texture_lc==0){
       hipMemcpy(vec_in_gpu, vec_in, size*sizeof(hipDoubleComplex), hipMemcpyHostToHost);
      }else{
        hipMemcpy(vec_in_gpu_pointer, vec_in, size*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
      }
       hipMemcpy(vec_out_gpu,   vec_out,     size*sizeof(hipDoubleComplex),        hipMemcpyHostToHost);
       hipMemcpy(QUART_gpu,     QUART,       size*sizeof(double),        hipMemcpyHostToHost);
       hipMemcpy(diagsz_gpu,    diagsz,      size*sizeof(double),        hipMemcpyHostToHost);
       hipMemcpy(offdiagsz_gpu, offdiagsz,   *offdiasize*sizeof(hipDoubleComplex), hipMemcpyHostToHost);
       hipMemcpy(noffsz_gpu,    noffsz ,     size*sizeof(int),           hipMemcpyHostToHost);
       hipMemcpy(rankoffsz_gpu, rankoffsz,   *roff*sizeof(int),          hipMemcpyHostToHost);
       hipEventSynchronize(0);
       offdia_gpu[0]=0; for(int istate=1; istate<size; istate++) { offdia_gpu[istate]=offdia_gpu[istate-1]+noffsz[istate-1]; };
       hipEventSynchronize(0);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

  if(verbose==1) printf(" call kernel  \n ");
  Hmult_ker_complex<<<bl,th>>>(ngrid,blocksize,size,vec_out_gpu_pointer,vec_in_gpu_pointer,QUART_gpu_pointer,diagsz_gpu_pointer,
                              noffsz_gpu_pointer,rankoffsz_gpu_pointer,offdiagsz_gpu_pointer,offdia_gpu_pointer,use_texture_lc); 
                              hipEventSynchronize(0); hipDeviceSynchronize();
  if(verbose==1) printf(" .....done.....  \n ");

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

     hipMemcpy(vec_out,vec_out_gpu,size*sizeof(hipDoubleComplex),hipMemcpyHostToHost);
     hipEventSynchronize(0);
     hipHostFree(vec_out_gpu);
    if(use_texture_lc==0){
     hipHostFree(vec_in_gpu);
    }else{
     unbind_x();
     hipFree(vec_in_gpu_pointer);
    }
     hipHostFree(offdia_gpu);
     hipHostFree(QUART_gpu);
     hipHostFree(diagsz_gpu);
     hipHostFree(rankoffsz_gpu);
     hipHostFree(offdiagsz_gpu);
     hipHostFree(noffsz_gpu);
     hipEventSynchronize(0);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

 void one_step_lanczos_cuda_complex(int blocksize, int Niter, int size, int iter, double *diag, double *subdiag, 
 hipDoubleComplex *vec_tmp_gpu, hipDoubleComplex *vec_in_gpu, hipDoubleComplex *vec_out_gpu, 
 hipDoubleComplex *vec_tmp_gpu_pointer, hipDoubleComplex *vec_in_gpu_pointer, hipDoubleComplex *vec_out_gpu_pointer, 
 double *QUART_gpu_pointer, double *diagsz_gpu_pointer, int *noffsz_gpu_pointer, int *rankoffsz_gpu_pointer, 
 hipDoubleComplex *offdiagsz_gpu_pointer, int *offdia_gpu_pointer, double *QUART_gpu, double *diagsz_gpu, int *noffsz_gpu, 
 int *rankoffsz_gpu, hipDoubleComplex *offdiagsz_gpu, int *offdia_gpu)
{

   int psize=size; int verbose=0;

   double normv; double *normv_ker; hipMalloc((void**)&normv_ker,sizeof(double)); double *normv_loc; normv_loc = &normv;
   hipDoubleComplex coef;

   int nb  =(size -size % blocksize)/blocksize + 1; int ngrid=nb/MAX_BLOCK; if(ngrid>0) nb=MAX_BLOCK;
   dim3 bl(1,nb),th(WARP_SIZE,blocksize);
   if(verbose==1) printf( " --------------- \n  Nblock=%d Ngrid=%d \n ----------------- \n ",nb,ngrid);

   int nb2 =(size-size % 256)/256 + 1; 
   int ngrid2=nb2/MAX_BLOCK; if(ngrid2>0) nb2=MAX_BLOCK;

   norm_vec_ker<<<1,512>>>(size,vec_in_gpu_pointer,normv_ker); hipEventSynchronize(0); hipDeviceSynchronize();
   normalize_vec_ker<<<nb2,256>>>(ngrid2,size,vec_in_gpu_pointer,normv_ker);

   Hmult_ker_complex<<<bl,th>>>(ngrid,blocksize,psize,vec_out_gpu_pointer,vec_in_gpu_pointer,QUART_gpu_pointer,diagsz_gpu_pointer,noffsz_gpu_pointer,rankoffsz_gpu_pointer,offdiagsz_gpu_pointer,offdia_gpu_pointer,use_texture_lc); hipEventSynchronize(0); hipDeviceSynchronize();

  if(iter>0){ coef=make_hipDoubleComplex(-subdiag[iter],0.0); cublasZaxpy_no_device(size,coef,vec_tmp_gpu,1,vec_out_gpu,1); }; hipEventSynchronize(0); hipDeviceSynchronize();

  if(use_texture_lc==0){
   hipMemcpy(vec_tmp_gpu,vec_in_gpu,size*sizeof(hipDoubleComplex),hipMemcpyHostToHost);hipEventSynchronize(0);hipDeviceSynchronize();
  }else{
   hipMemcpy(vec_tmp_gpu,vec_in_gpu_pointer,size*sizeof(hipDoubleComplex),hipMemcpyDeviceToHost);hipEventSynchronize(0);hipDeviceSynchronize();
  }

   real_scal_vec_ker<<<1,512>>> (size,vec_out_gpu_pointer,vec_in_gpu_pointer,normv_ker);
   hipMemcpy(normv_loc,normv_ker,sizeof(double),hipMemcpyDeviceToHost);
   diag[iter]=*normv_loc ; 

   hipEventSynchronize(0);hipDeviceSynchronize();
   coef=make_hipDoubleComplex(-diag[iter],0.0);
   cublasZaxpy_no_device(size,coef,vec_tmp_gpu,1,vec_out_gpu,1); hipEventSynchronize(0); hipDeviceSynchronize();

   normv = sqrt(hipCabs(cublasZdotu_no_device(size,vec_out_gpu,1,vec_out_gpu,1))); hipEventSynchronize(0); hipDeviceSynchronize();

   if(iter<Niter-1) subdiag[iter+1]=normv;

  if(use_texture_lc==0){
   hipMemcpy( vec_in_gpu, vec_out_gpu, size*sizeof(hipDoubleComplex), hipMemcpyHostToHost); hipEventSynchronize(0); hipDeviceSynchronize();
  }else{
   hipMemcpy( vec_in_gpu_pointer, vec_out_gpu, size*sizeof(hipDoubleComplex), hipMemcpyHostToDevice); hipEventSynchronize(0); hipDeviceSynchronize();
  }

  hipFree(normv_ker);
}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

extern "C" void lanczos_dynamic_cuda_complex_(int *pblocksize, int *Niter_lanczos_,int *offdiasize, int *roff, 
int *ntot, double *QUART, double *diagsz, int *noffsz, int *rankoffsz, hipDoubleComplex *offdiagsz, double *diag, 
double *subdiag , hipDoubleComplex *vecinit)
{

 //---------------------------------------------------------------------------------------//
     int verbose=0;
 //---------------------------------------------------------------------------------------//

     int Niter_lanczos=*Niter_lanczos_; 

     if(verbose==1) printf(" start Lanczos Real on GPU \n" );

     int size = *ntot; int blocksize=*pblocksize; 

     hipDoubleComplex   *vec_in_gpu,*vec_out_gpu;
     double            *QUART_gpu,*diagsz_gpu;
     hipDoubleComplex   *offdiagsz_gpu,*vec_tmp_gpu;

     int               *noffsz_gpu,*rankoffsz_gpu;
     hipDoubleComplex   *vec_in_gpu_pointer,*vec_out_gpu_pointer,*vec_tmp_gpu_pointer;
     double            *QUART_gpu_pointer,*diagsz_gpu_pointer;
     hipDoubleComplex   *offdiagsz_gpu_pointer;
     int               *noffsz_gpu_pointer,*rankoffsz_gpu_pointer;
     int               *offdia_gpu, *offdia_gpu_pointer;

     if(verbose==1) printf(" GPU get eigenvalues, size of Lanczos vectors = %d ", size);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

       hipSetDevice(0); hipSetDeviceFlags( hipDeviceMapHost ); hipEventSynchronize(0);

       if(use_texture_lc==1){
         hipMalloc((void**)&vec_in_gpu_pointer,sizeof(hipDoubleComplex)*size);
         bind_x(vec_in_gpu_pointer,size);
       }else{
         hipHostAlloc((void**)&vec_in_gpu    , size*sizeof(hipDoubleComplex),        hipHostMallocMapped | hipHostMallocPortable );
         hipHostGetDevicePointer((void**)  &vec_in_gpu_pointer  ,  vec_in_gpu    , 0 );
       }

       hipHostAlloc((void**)&vec_tmp_gpu   , size*sizeof(hipDoubleComplex),        hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&vec_out_gpu   , size*sizeof(hipDoubleComplex),        hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&QUART_gpu     , size*sizeof(double),                 hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&diagsz_gpu    , size*sizeof(double),                 hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&offdiagsz_gpu , *offdiasize*sizeof(hipDoubleComplex), hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&noffsz_gpu    , size*sizeof(int)   ,                 hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&rankoffsz_gpu , *roff*sizeof(int)  ,                 hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&offdia_gpu    , size*sizeof(int)   ,                 hipHostMallocMapped | hipHostMallocPortable );
       hipEventSynchronize(0);

       hipHostGetDevicePointer((void**)  &vec_tmp_gpu_pointer ,  vec_tmp_gpu   , 0 );
       hipHostGetDevicePointer((void**)  &vec_out_gpu_pointer ,  vec_out_gpu   , 0 );
       hipHostGetDevicePointer((void**)  &QUART_gpu_pointer ,    QUART_gpu     , 0 );
       hipHostGetDevicePointer((void**)  &diagsz_gpu_pointer ,   diagsz_gpu    , 0 );
       hipHostGetDevicePointer((void**)  &offdiagsz_gpu_pointer, offdiagsz_gpu , 0 );
       hipHostGetDevicePointer((void**)  &noffsz_gpu_pointer,    noffsz_gpu    , 0 );
       hipHostGetDevicePointer((void**)  &rankoffsz_gpu_pointer, rankoffsz_gpu , 0 );
       hipHostGetDevicePointer((void**)  &offdia_gpu_pointer,    offdia_gpu    , 0 );
       hipEventSynchronize(0);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

       hipMemcpy(QUART_gpu,     QUART,       size*sizeof(double),        hipMemcpyHostToHost);
       hipMemcpy(diagsz_gpu,    diagsz,      size*sizeof(double),        hipMemcpyHostToHost);
       hipMemcpy(offdiagsz_gpu, offdiagsz,   *offdiasize*sizeof(hipDoubleComplex), hipMemcpyHostToHost);
       hipMemcpy(noffsz_gpu,    noffsz ,     size*sizeof(int),           hipMemcpyHostToHost);
       hipMemcpy(rankoffsz_gpu, rankoffsz,   *roff*sizeof(int),          hipMemcpyHostToHost);
       hipEventSynchronize(0);
       offdia_gpu[0]=0; for(int istate=1; istate<size; istate++) { offdia_gpu[istate]=offdia_gpu[istate-1]+noffsz[istate-1]; };
       hipEventSynchronize(0);

  if(verbose==1) printf( " initialized, now run actual Lanczos \n " );

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

  hipDeviceSynchronize(); hipEventSynchronize(0); 

 if(use_texture_lc==0){
   hipMemcpy(vec_in_gpu,vecinit,size*sizeof(hipDoubleComplex),hipMemcpyHostToHost);
 }else{
   hipMemcpy(vec_out_gpu,vecinit,size*sizeof(hipDoubleComplex),hipMemcpyHostToHost);
   hipMemcpy(vec_in_gpu_pointer,vec_out_gpu,size*sizeof(hipDoubleComplex),hipMemcpyHostToDevice);
 }

  for(int iter=0;iter<Niter_lanczos;iter++){
   if(verbose==1) printf( " Lanczos iteration %d / %d \n", iter,Niter_lanczos);
   one_step_lanczos_cuda_complex(blocksize,Niter_lanczos,size,iter,diag,subdiag,vec_tmp_gpu,vec_in_gpu,vec_out_gpu,vec_tmp_gpu_pointer,vec_in_gpu_pointer,vec_out_gpu_pointer,QUART_gpu_pointer,diagsz_gpu_pointer,noffsz_gpu_pointer,rankoffsz_gpu_pointer,offdiagsz_gpu_pointer,offdia_gpu_pointer,QUART_gpu,diagsz_gpu,noffsz_gpu,rankoffsz_gpu, offdiagsz_gpu, offdia_gpu);

  };

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

     hipEventSynchronize(0);
     hipHostFree(vec_tmp_gpu);
    if(use_texture_lc==0){
     hipHostFree(vec_in_gpu);
    }else{
     unbind_x();
     hipFree(vec_in_gpu_pointer);
    }
     hipHostFree(vec_out_gpu);
     hipHostFree(offdia_gpu);
     hipHostFree(QUART_gpu);
     hipHostFree(diagsz_gpu);
     hipHostFree(rankoffsz_gpu);
     hipHostFree(offdiagsz_gpu);
     hipHostFree(noffsz_gpu);
     hipEventSynchronize(0);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

extern "C" void lanczos_cuda_complex_(int *pblocksize,  int *Niter_lanczos_,int *offdiasize, int *roff, int *ntot, 
double *QUART, double *diagsz, int *noffsz, int *rankoffsz, hipDoubleComplex *offdiagsz, double *diag, double *subdiag )
{

 //---------------------------------------------------------------------------------------//
     int verbose=0;
 //---------------------------------------------------------------------------------------//

     int Niter_lanczos=*Niter_lanczos_; 
     if(verbose==1) printf(" start Lanczos Real on GPU \n" );
     int size = *ntot; int blocksize = *pblocksize;

     hipDoubleComplex  *vec_in_gpu,*vec_out_gpu;
     double           *QUART_gpu,*diagsz_gpu;
     hipDoubleComplex  *offdiagsz_gpu,*vec_tmp_gpu;
     int *noffsz_gpu, *rankoffsz_gpu;
     hipDoubleComplex  *vec_in_gpu_pointer,*vec_out_gpu_pointer,*vec_tmp_gpu_pointer;
     double           *QUART_gpu_pointer,*diagsz_gpu_pointer;
     hipDoubleComplex  *offdiagsz_gpu_pointer;
     int              *noffsz_gpu_pointer,*rankoffsz_gpu_pointer;
     int              *offdia_gpu, *offdia_gpu_pointer;

     if(verbose==1) printf(" GPU get eigenvalues, size of Lanczos vectors = %d ", size);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

       hipSetDevice(0); hipSetDeviceFlags( hipDeviceMapHost ); hipEventSynchronize(0);

       if(use_texture_lc==1){
         hipMalloc((void**)&vec_in_gpu_pointer,sizeof(hipDoubleComplex)*size);
         bind_x(vec_in_gpu_pointer,size);
       }else{
         hipHostAlloc((void**)&vec_in_gpu    , size*sizeof(hipDoubleComplex),        hipHostMallocMapped | hipHostMallocPortable );
         hipHostGetDevicePointer((void**) &vec_in_gpu_pointer ,vec_in_gpu , 0 );
       }

       hipHostAlloc((void**)&vec_tmp_gpu   , size*sizeof(hipDoubleComplex),        hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&vec_out_gpu   , size*sizeof(hipDoubleComplex),        hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&QUART_gpu     , size*sizeof(double),                 hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&diagsz_gpu    , size*sizeof(double),                 hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&offdiagsz_gpu , *offdiasize*sizeof(hipDoubleComplex), hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&noffsz_gpu    , size*sizeof(int)   ,                 hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&rankoffsz_gpu , *roff*sizeof(int)  ,                 hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&offdia_gpu    , size*sizeof(int)   ,                 hipHostMallocMapped | hipHostMallocPortable );
       hipEventSynchronize(0);

       hipHostGetDevicePointer((void**)   &vec_tmp_gpu_pointer ,  vec_tmp_gpu    , 0 );
       hipHostGetDevicePointer((void**)   &vec_out_gpu_pointer ,  vec_out_gpu    , 0 );
       hipHostGetDevicePointer((void**)   &QUART_gpu_pointer ,    QUART_gpu      , 0 );
       hipHostGetDevicePointer((void**)   &diagsz_gpu_pointer ,   diagsz_gpu     , 0 );
       hipHostGetDevicePointer((void**)   &offdiagsz_gpu_pointer, offdiagsz_gpu  , 0 );
       hipHostGetDevicePointer((void**)   &noffsz_gpu_pointer,    noffsz_gpu     , 0 );
       hipHostGetDevicePointer((void**)   &rankoffsz_gpu_pointer, rankoffsz_gpu  , 0 );
       hipHostGetDevicePointer((void**)   &offdia_gpu_pointer,    offdia_gpu     , 0 );
       hipEventSynchronize(0);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

       hipMemcpy(QUART_gpu,     QUART,       size*sizeof(double),        hipMemcpyHostToHost);
       hipMemcpy(diagsz_gpu,    diagsz,      size*sizeof(double),        hipMemcpyHostToHost);
       hipMemcpy(offdiagsz_gpu, offdiagsz,   *offdiasize*sizeof(hipDoubleComplex), hipMemcpyHostToHost);
       hipMemcpy(noffsz_gpu,    noffsz ,     size*sizeof(int),           hipMemcpyHostToHost);
       hipMemcpy(rankoffsz_gpu, rankoffsz,   *roff*sizeof(int),          hipMemcpyHostToHost);
       hipEventSynchronize(0);
       offdia_gpu[0]=0; for(int istate=1; istate<size; istate++) { offdia_gpu[istate]=offdia_gpu[istate-1]+noffsz[istate-1]; };
       hipEventSynchronize(0);

  if(verbose==1) printf( " initialized, now run actual Lanczos \n " );

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

 if(use_texture_lc==0){
  for(int i=0;i<size;i++) vec_in_gpu[i]=make_hipDoubleComplex(1.0,0.0); hipDeviceSynchronize();
 }else{
   for(int i=0;i<size;i++) vec_out_gpu[i]=make_hipDoubleComplex(1.0,0.0); hipDeviceSynchronize();
   hipMemcpy(vec_in_gpu_pointer,vec_out_gpu,size*sizeof(hipDoubleComplex),hipMemcpyHostToDevice);
 }

  for(int iter=0;iter<Niter_lanczos;iter++){
    if(verbose==1) printf( " Lanczos iteration %d / %d \n", iter,Niter_lanczos);
    one_step_lanczos_cuda_complex(blocksize,Niter_lanczos,size,iter,diag,subdiag,vec_tmp_gpu,vec_in_gpu,vec_out_gpu,vec_tmp_gpu_pointer,vec_in_gpu_pointer,vec_out_gpu_pointer,QUART_gpu_pointer,diagsz_gpu_pointer,noffsz_gpu_pointer,rankoffsz_gpu_pointer,offdiagsz_gpu_pointer,offdia_gpu_pointer,QUART_gpu,diagsz_gpu,noffsz_gpu,rankoffsz_gpu, offdiagsz_gpu, offdia_gpu);
  };

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

     hipEventSynchronize(0);
     hipHostFree(vec_tmp_gpu);
    if(use_texture_lc==0){
     hipHostFree(vec_in_gpu);
    }else{
     unbind_x();
     hipFree(vec_in_gpu_pointer);
    }
     hipHostFree(vec_out_gpu);
     hipHostFree(offdia_gpu);
     hipHostFree(QUART_gpu);
     hipHostFree(diagsz_gpu);
     hipHostFree(rankoffsz_gpu);
     hipHostFree(offdiagsz_gpu);
     hipHostFree(noffsz_gpu);
     hipEventSynchronize(0);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

extern "C" void lanczos_get_gs_cuda_complex_(int *pblocksize, int *Niter_lanczos_,int *offdiasize, int *roff, 
int *ntot, double *QUART, double *diagsz, int *noffsz, int *rankoffsz, hipDoubleComplex *offdiagsz, double *vecp, 
hipDoubleComplex *GS)
{

 //---------------------------------------------------------------------------------------//
     int verbose=0;
 //---------------------------------------------------------------------------------------//

     if(verbose==1) printf(" start Lanczos Real on GPU \n" );

     int Niter_lanczos=*Niter_lanczos_;
     double diag[Niter_lanczos], subdiag[Niter_lanczos];
     int size = *ntot; int blocksize =*pblocksize;

     hipDoubleComplex  *vec_in_gpu,*vec_out_gpu;
     double           *QUART_gpu,*diagsz_gpu;
     hipDoubleComplex  *offdiagsz_gpu,*vec_tmp_gpu;
     int              *noffsz_gpu,*rankoffsz_gpu;
     hipDoubleComplex  *vec_in_gpu_pointer,*vec_out_gpu_pointer,*vec_tmp_gpu_pointer;
     double           *QUART_gpu_pointer,*diagsz_gpu_pointer;
     hipDoubleComplex  *offdiagsz_gpu_pointer;
     int              *noffsz_gpu_pointer,*rankoffsz_gpu_pointer;
     int              *offdia_gpu, *offdia_gpu_pointer;
     hipDoubleComplex  *GS_gpu, *GS_gpu_pointer;

     if(verbose==1) printf(" \n GPU get Ground State, size of Lanczos vectors = %d \n ", size);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

       hipSetDevice(0); hipSetDeviceFlags( hipDeviceMapHost ); hipEventSynchronize(0);

       if(use_texture_lc==1){
        hipMalloc((void**)&vec_in_gpu_pointer,sizeof(hipDoubleComplex)*size);
        bind_x(vec_in_gpu_pointer,size);
      }else{
        hipHostAlloc((void**)&vec_in_gpu    , size*sizeof(hipDoubleComplex),        hipHostMallocMapped | hipHostMallocPortable );
        hipHostGetDevicePointer((void**)   &vec_in_gpu_pointer  ,  vec_in_gpu     , 0 );
      }

       hipHostAlloc((void**)&GS_gpu        , size*sizeof(hipDoubleComplex),        hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&vec_tmp_gpu   , size*sizeof(hipDoubleComplex),        hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&vec_out_gpu   , size*sizeof(hipDoubleComplex),        hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&QUART_gpu     , size*sizeof(double),                 hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&diagsz_gpu    , size*sizeof(double),                 hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&offdiagsz_gpu , *offdiasize*sizeof(hipDoubleComplex), hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&noffsz_gpu    , size*sizeof(int)   ,                 hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&rankoffsz_gpu , *roff*sizeof(int)  ,                 hipHostMallocMapped | hipHostMallocPortable );
       hipHostAlloc((void**)&offdia_gpu    , size*sizeof(int)   ,                 hipHostMallocMapped | hipHostMallocPortable );
       hipEventSynchronize(0);

       hipHostGetDevicePointer((void**)   &GS_gpu_pointer ,       GS_gpu         , 0 );
       hipHostGetDevicePointer((void**)   &vec_tmp_gpu_pointer ,  vec_tmp_gpu    , 0 );
       hipHostGetDevicePointer((void**)   &vec_out_gpu_pointer ,  vec_out_gpu    , 0 );
       hipHostGetDevicePointer((void**)   &QUART_gpu_pointer ,    QUART_gpu      , 0 );
       hipHostGetDevicePointer((void**)   &diagsz_gpu_pointer ,   diagsz_gpu     , 0 );
       hipHostGetDevicePointer((void**)   &offdiagsz_gpu_pointer, offdiagsz_gpu  , 0 );
       hipHostGetDevicePointer((void**)   &noffsz_gpu_pointer,    noffsz_gpu     , 0 );
       hipHostGetDevicePointer((void**)   &rankoffsz_gpu_pointer, rankoffsz_gpu  , 0 );
       hipHostGetDevicePointer((void**)   &offdia_gpu_pointer,    offdia_gpu     , 0 );
       hipEventSynchronize(0);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

       hipMemcpy(QUART_gpu,     QUART,       size*sizeof(double),        hipMemcpyHostToHost);
       hipMemcpy(diagsz_gpu,    diagsz,      size*sizeof(double),        hipMemcpyHostToHost);
       hipMemcpy(offdiagsz_gpu, offdiagsz,   *offdiasize*sizeof(hipDoubleComplex), hipMemcpyHostToHost);
       hipMemcpy(noffsz_gpu,    noffsz ,     size*sizeof(int),           hipMemcpyHostToHost);
       hipMemcpy(rankoffsz_gpu, rankoffsz,   *roff*sizeof(int),          hipMemcpyHostToHost);
       hipEventSynchronize(0);
       offdia_gpu[0]=0; for(int istate=1; istate<size; istate++) { offdia_gpu[istate]=offdia_gpu[istate-1]+noffsz[istate-1]; };
       hipEventSynchronize(0);

  if(verbose==1) printf( " initialized, now run actual Lanczos \n " );

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

  memset ((void **)GS_gpu, 0, size*sizeof(hipDoubleComplex)); hipDeviceSynchronize();  hipEventSynchronize(0);

 if(use_texture_lc==0){
  for(int i=0;i<size;i++) vec_in_gpu[i]=make_hipDoubleComplex(1.0,0.0); hipDeviceSynchronize();
 }else{
   for(int i=0;i<size;i++) vec_out_gpu[i]=make_hipDoubleComplex(1.0,0.0); hipDeviceSynchronize();
   hipMemcpy(vec_in_gpu_pointer,vec_out_gpu,size*sizeof(hipDoubleComplex),hipMemcpyHostToDevice);
 }

 if(verbose==1) printf(" start lanczos iterations \n ");

 double *normv_ker; double normv; double *normv_loc; hipMalloc((void**)&normv_ker,sizeof(double)); hipDoubleComplex coef; 

 if(use_texture_lc==0){
   coef = make_hipDoubleComplex(1.0/sqrt(hipCabs(cublasZdotu_no_device(size,vec_in_gpu,1,vec_in_gpu,1)))*vecp[0],0); hipDeviceSynchronize();  hipEventSynchronize(0);
   cublasZaxpy_no_device(size,coef,vec_in_gpu,1,GS_gpu,1); hipDeviceSynchronize(); hipEventSynchronize(0);hipDeviceSynchronize();
 }else{
   norm_vec_ker<<<1,512>>>(size,vec_in_gpu_pointer,normv_ker); hipEventSynchronize(0); hipDeviceSynchronize();
   normv_loc=&normv; hipMemcpy(normv_loc,normv_ker,sizeof(double),hipMemcpyDeviceToHost);
   hipMemcpy(vec_out_gpu,vec_in_gpu,size*sizeof(hipDoubleComplex),hipMemcpyDeviceToHost); 
   coef=make_hipDoubleComplex(vecp[0]/normv,0.0);
   cublasZaxpy_no_device(size,coef,vec_out_gpu,1,GS_gpu,1); hipDeviceSynchronize(); hipEventSynchronize(0);hipDeviceSynchronize();
 };

  if(verbose==1) printf( " first step ... \n ");

  one_step_lanczos_cuda_complex(blocksize, Niter_lanczos,size,0,diag,subdiag,vec_tmp_gpu,vec_in_gpu,vec_out_gpu,vec_tmp_gpu_pointer,vec_in_gpu_pointer,vec_out_gpu_pointer,QUART_gpu_pointer,diagsz_gpu_pointer,noffsz_gpu_pointer,rankoffsz_gpu_pointer,offdiagsz_gpu_pointer,offdia_gpu_pointer,QUART_gpu,diagsz_gpu,noffsz_gpu,rankoffsz_gpu, offdiagsz_gpu, offdia_gpu);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//
 
  for(int iter=1;iter<Niter_lanczos-1;iter++){

   if(verbose==1) printf( " iterations = %d \n " , iter);

   coef = make_hipDoubleComplex(1.0/sqrt(hipCabs(cublasZdotu_no_device(size,vec_out_gpu,1,vec_out_gpu,1))),0.0); 
   cublasZscal_no_device (size,coef, vec_out_gpu,1);  hipEventSynchronize(0);

   coef = make_hipDoubleComplex(vecp[iter],0);
   cublasZaxpy_no_device(size,coef,vec_out_gpu,1,GS_gpu,1); hipDeviceSynchronize(); hipEventSynchronize(0);

   one_step_lanczos_cuda_complex(blocksize,Niter_lanczos,size,iter,diag,subdiag,vec_tmp_gpu,vec_in_gpu,vec_out_gpu,vec_tmp_gpu_pointer,vec_in_gpu_pointer,vec_out_gpu_pointer,QUART_gpu_pointer,diagsz_gpu_pointer,noffsz_gpu_pointer,rankoffsz_gpu_pointer,offdiagsz_gpu_pointer,offdia_gpu_pointer,QUART_gpu,diagsz_gpu,noffsz_gpu,rankoffsz_gpu, offdiagsz_gpu, offdia_gpu);  hipEventSynchronize(0);hipDeviceSynchronize();
  };

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

   if(verbose==1) printf("done...\n");

   coef=make_hipDoubleComplex(1.0/sqrt(hipCabs(cublasZdotu_no_device(size,vec_out_gpu,1,vec_out_gpu,1))),0.0); 
   cublasZscal_no_device(size,coef,vec_out_gpu,1);hipEventSynchronize(0);

   coef=make_hipDoubleComplex(vecp[Niter_lanczos-1],0.0);
   cublasZaxpy_no_device(size,coef,vec_out_gpu,1,GS_gpu,1); hipDeviceSynchronize(); hipEventSynchronize(0); 

   coef=make_hipDoubleComplex(1.0/sqrt(hipCabs(cublasZdotu_no_device(size,GS_gpu,1,GS_gpu,1))),0.0);  hipDeviceSynchronize();  hipEventSynchronize(0);
   cublasZscal_no_device(size,coef,GS_gpu,1); hipEventSynchronize(0); hipDeviceSynchronize();

   hipMemcpy(GS,GS_gpu,size*sizeof(hipDoubleComplex),hipMemcpyHostToHost); hipEventSynchronize(0); hipDeviceSynchronize();

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

     hipEventSynchronize(0);
     hipHostFree(vec_tmp_gpu);
    if(use_texture_lc==0){
     hipHostFree(vec_in_gpu);
    }else{
     unbind_x();
     hipFree(vec_in_gpu_pointer);
    }
     hipHostFree(vec_out_gpu);
     hipHostFree(offdia_gpu);
     hipHostFree(QUART_gpu);
     hipHostFree(diagsz_gpu);
     hipHostFree(rankoffsz_gpu);
     hipHostFree(offdiagsz_gpu);
     hipHostFree(noffsz_gpu);
     hipEventSynchronize(0);

     hipFree(normv_ker); hipEventSynchronize(0); hipHostFree(GS_gpu);hipFree(GS_gpu_pointer); hipEventSynchronize(0);

 //---------------------------------------------------------------------------------------//
 //---------------------------------------------------------------------------------------//

}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
