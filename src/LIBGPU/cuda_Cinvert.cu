#include "hip/hip_runtime.h"

///********************************************************************
//*  File: cuda_invert.cu
//*
//*  Description:
//*     
//*     Mainfunction to compute an inverse Matrix from a positive definite 
//*   Matrix on the GPU. The Routine is using the Gaus Seidel Matrix invertion 
//*   algorithm. 
//*   
//*   1   2   1   |  1  0   0               1   0   0  |  -2.5   1.5   0.5
//*   2   3   1   |  0  1   0       =>      0   1   0  |   1.5  -0.5  -0.5 
//*   1   1   2   |  0  0   1               0   0   1  |   0.5  -0.5   0.5 
//*   Inputmatrix       E                       E          Inverse Matrix
//*
//*  Arguments: 
//*       - float *A            Input Matrix 1D, no data changes
//*   - float *invA         Output Matrix 1D, the inverse datamatrix  
//*   - int size            Matrix dimension in size, width = height = size
//*     
//*  Used custom kernels rutines:
//*       - GPUsetIdentity          
//*   - eliminateBlockL   
//*   - adjustColL         
//*   - eliminateColL      
//*   - eliminateRestL     
//*
//*   - eliminateBlockU    
//*   - adjustColU         
//*   - eliminateColU      
//*   - eliminateRestU     
//*
//*********************************************************************


#include <stdio.h>
#include <stdlib.h>
#include <cutil.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>

#define MAXSIZE 16

//************************************************
// Kernel GPUsetIdentity 
//************************************************

__global__ void GPUsetIdentity(int BLOCKSIZE,hipDoubleComplex* matrix,int width)
{

    int tx = threadIdx.x; int bx = blockIdx.x;

    int offset = bx * BLOCKSIZE + tx;
    
    matrix[offset * width + offset] = make_hipDoubleComplex(1.0,0.0);

}

//************************************************
// Kernel eliminateBlockL
//************************************************

__global__ void eliminateBlock(char S, int BLOCKSIZE, hipDoubleComplex *dInData, int size)
{
     int BLOCKSIZEMINUS1 = BLOCKSIZE-1;
     
    int tx = threadIdx.x; int ty = threadIdx.y;

    __shared__ hipDoubleComplex triangleBlock[MAXSIZE][MAXSIZE];

    if(S == 'L')
    {
         triangleBlock[ty][tx] = dInData[ ty * size + tx];
         
         __syncthreads ();

        //i equals the current row
    
        for (int i = 0; i < BLOCKSIZEMINUS1; i++)
        {
        
        // calculate the pivot element to get the current row i to zero
        
        
            hipDoubleComplex pivotEl = hipCdiv( triangleBlock[ty][i] , triangleBlock[i][i] );

            __syncthreads ();       // Each pivotEl have to be calculated and store in the registers

            if (ty > i)             // If all cols (ty) are below the current row (i)?
            {
                 if (tx > i)         // The element is right to the current row, subtract the element
                {
                     triangleBlock[ty][tx] = hipCsub(triangleBlock[ty][tx], hipCmul(pivotEl,triangleBlock[i][tx]));
                }
                
                if (tx == i)        // Store the pivot element in the current row
                {
                    triangleBlock[ty][tx] = pivotEl;
                }
                
            }
            
            __syncthreads ();       // Wait for each thread
          
          }
      
             dInData[ty * size + tx] = triangleBlock[ty][tx]; // Write the result back to memory
    }
    
    if(S == 'U')
    {

         triangleBlock[ty][tx] = dInData[ty * size + tx];
    
    __syncthreads ();

    //i equals the current row

    for (int i = BLOCKSIZEMINUS1; i > 0; i--)
    {
        // calculate the pivot element to get the current row i to zero

        hipDoubleComplex pivotEl = hipCdiv(triangleBlock[ty][i] , triangleBlock[i][i]);

        __syncthreads ();       // Each pivotEl have to be calculated and store in the registers

        if (ty < i)             // If all rows (ty) are above the current row (i)?
        {

            if (tx < i)         // The element is left to the current row, subtract the element
            {
                triangleBlock[ty][tx] = hipCsub(triangleBlock[ty][tx],hipCmul(pivotEl,triangleBlock[i][tx]));
            }

            if (tx == i)        // Store the pivot element in the current row
            {
                triangleBlock[ty][tx] = pivotEl;
            }

        }

        __syncthreads ();        // Wait for each thread

    }

         dInData[ty * size + tx] = triangleBlock[ty][tx];       //Write the result back to device memory
             
    }
    
}

//************************************************
// Kernel adjustRowL
//************************************************

                                                   
__global__ void adjustRowL(int BLOCKSIZE, hipDoubleComplex *dMatrixIn, hipDoubleComplex *dMatrixInDiag, hipDoubleComplex *dMatrixInv, int width, int diagEl)
{
    int BLOCKSIZEMINUS1 = BLOCKSIZE-1;
     
    int tx = threadIdx.x; int ty = threadIdx.y; int bx = blockIdx.x;

    __shared__ hipDoubleComplex pivotBlock[MAXSIZE][MAXSIZE];
    __shared__ hipDoubleComplex inBlock[MAXSIZE][MAXSIZE];
    __shared__ hipDoubleComplex invBlock[MAXSIZE][MAXSIZE];

    //* Adjust the rest blocks which are right from the prepared block of step 1 and adjust the inverse blocks
    
    if (bx * BLOCKSIZE > (diagEl + 1))
    {

        pivotBlock[ty][tx] = dMatrixInDiag[ty * width + tx];
        inBlock[ty][tx]    = dMatrixIn[ty * width + bx * BLOCKSIZE +tx];
        invBlock[ty][tx]   = dMatrixInv[ty * width + bx * BLOCKSIZE +tx];

        __syncthreads ();

        //i equals the current row where the pivot elements are stored

        for (int i = 0; i < BLOCKSIZEMINUS1; i++)
        {
            // if the cols are below  
            
            if (ty > i)
            {
                hipDoubleComplex pivot = pivotBlock[ty][i];
                
                //Subtract the row
                
                inBlock[ty][tx]  = hipCsub(inBlock[ty][tx], hipCmul(inBlock[i][tx],pivot));
                invBlock[ty][tx] = hipCsub(invBlock[ty][tx], hipCmul(invBlock[i][tx],pivot));
            }

            __syncthreads ();
        }
        
        // Store the results back in device memory

        dMatrixIn[ty * width + bx * BLOCKSIZE +tx] = inBlock[ty][tx];
        dMatrixInv[ty * width + bx * BLOCKSIZE +tx] = invBlock[ty][tx];
         
    }
    
    //* Adjust the last blocks from the indentity matrix which are left 
    
    else
    {
      pivotBlock[ty][tx] = dMatrixInDiag[ty * width + tx];
        invBlock[ty][tx] = dMatrixInv[ty * width + bx * BLOCKSIZE +tx];
         
        __syncthreads ();
        
        for (int i = 0; i < BLOCKSIZEMINUS1; i++) //last changed
        {
            if (ty > i)
            {
                hipDoubleComplex pivot = pivotBlock[ty][i];
                invBlock[ty][tx] = hipCsub(invBlock[ty][tx] ,hipCmul(invBlock[i][tx],pivot));
            }

            __syncthreads ();
        }

             dMatrixInv[ty * width + bx * BLOCKSIZE + tx] = invBlock[ty][tx];
                
    }
    
}

//************************************************
// Kernel adjustRowU
//************************************************


__global__ void adjustRowU(int BLOCKSIZE,hipDoubleComplex *dMatrixIn, hipDoubleComplex *dMatrixInv, int width,int diagEl)
{
    int BLOCKSIZEMINUS1 = BLOCKSIZE-1;
          
    int tx = threadIdx.x; int ty = threadIdx.y; int bx = blockIdx.x;

    __shared__ hipDoubleComplex pivotBlock[MAXSIZE][MAXSIZE];
    __shared__ hipDoubleComplex invBlock[MAXSIZE][MAXSIZE];

     pivotBlock[ty][tx] = dMatrixIn[ty * width + tx];
     invBlock[ty][tx] = dMatrixInv[ty * width + bx * BLOCKSIZE +tx];
    
    __syncthreads ();

    for (int i = BLOCKSIZEMINUS1; i > 0; i--)
    {
        if (ty < i)
        {
            hipDoubleComplex pivot = pivotBlock[ty][i];

            invBlock[ty][tx] = hipCsub(invBlock[ty][tx],hipCmul(invBlock[i][tx],pivot));
        }

        __syncthreads ();
    }

     dMatrixInv[ty * width + bx * BLOCKSIZE +tx] = invBlock[ty][tx];

}

//************************************************
// Kernel eliminateColL
//************************************************

__global__ void eliminateColL(int BLOCKSIZE, hipDoubleComplex *dMatrixIn, int size, int diagEl)
{
     
    int tx = threadIdx.x; int ty = threadIdx.y;

    // bx is used to adress the Blocks above the precalculated block from step 1
    
    int bx = blockIdx.x;    

    //only the blocks can enter which are above the precalculated block from step 1
    
    if (bx * BLOCKSIZE > (diagEl + 1))
    {
        int offset = diagEl * size;
        int blockOffset = bx * BLOCKSIZE *size;

        __shared__ hipDoubleComplex pivotBlock[MAXSIZE][MAXSIZE];
        __shared__ hipDoubleComplex inBlock[MAXSIZE][MAXSIZE];

        pivotBlock[ty][tx] = dMatrixIn[offset + ty * size + tx];   // The Block from step 1
           inBlock[ty][tx] = dMatrixIn[blockOffset + ty * size + tx]; // each Block which is above the pivotBlock

        __syncthreads ();
    
        //iterate through the block und calculate the pivot elements
    
        for (int i = 0; i < BLOCKSIZE; i++)
        {
            hipDoubleComplex pivotEl = hipCdiv(inBlock[ty][i],pivotBlock[i][i]);

            __syncthreads ();

            //adjust all values right to the current interation step
           
            if (tx > i)
            {
                //substract the row
                inBlock[ty][tx] = hipCsub(inBlock[ty][tx], hipCmul(pivotBlock[i][tx],pivotEl));
            }
           
            //store the pivot element in the col
           
            else
            {
                inBlock[ty][i] = pivotEl;
            }

            __syncthreads ();
        }

          dMatrixIn[blockOffset + ty * size + tx] = inBlock[ty][tx];
        
    }
}

//************************************************
// Kernel eliminateColU
//************************************************

__global__ void eliminateColU(int BLOCKSIZE, hipDoubleComplex *dMatrixIn, int size, int diagEl)
{
    int BLOCKSIZEMINUS1 = BLOCKSIZE-1;
          
    int tx = threadIdx.x; int ty = threadIdx.y; int bx = blockIdx.x;

    if (bx * BLOCKSIZE <diagEl)
    {
        int offset = diagEl * size;
        int blockOffset = bx * BLOCKSIZE *size;

        __shared__ hipDoubleComplex pivotBlock[MAXSIZE][MAXSIZE];
        __shared__ hipDoubleComplex inBlock[MAXSIZE][MAXSIZE];

       pivotBlock[ty][tx] = dMatrixIn[offset + ty * size + tx];
          inBlock[ty][tx] = dMatrixIn[blockOffset + ty * size + tx];
        
        __syncthreads ();

        for (int i = BLOCKSIZEMINUS1; i >= 0; i--)
        {
            hipDoubleComplex pivotEl = hipCdiv(inBlock[ty][i],pivotBlock[i][i]);

            __syncthreads ();

            if (tx < i)
            {
                inBlock[ty][tx] = hipCsub(inBlock[ty][tx],hipCmul(pivotBlock[i][tx],pivotEl));
            }
            else //* if (tx == i)
            {
                inBlock[ty][i] = pivotEl;
            }

            __syncthreads ();
        }

             dMatrixIn[blockOffset + ty * size + tx] = inBlock[ty][tx];
        
    }
}

//************************************************
// Kernel eliminateRestL
//************************************************

__global__ void eliminateRestL(int BLOCKSIZE, hipDoubleComplex *dMatrixIn, hipDoubleComplex *dMatrixInv, int size,int diagEl)
{
     
    int tx = threadIdx.x; int ty = threadIdx.y;
    int bx = blockIdx.x;  int by = blockIdx.y;

    __shared__ hipDoubleComplex pivEl[MAXSIZE][MAXSIZE];
    __shared__ hipDoubleComplex pivBlock[MAXSIZE][MAXSIZE];
    __shared__ hipDoubleComplex inBlock[MAXSIZE][MAXSIZE];

    //rest of the unadjusted Matrix which is right above the diagEl

    if (bx * BLOCKSIZE > (diagEl + 1) && by * BLOCKSIZE > (diagEl + 1))
    {

        int blockOffset = by * BLOCKSIZE * size + bx * BLOCKSIZE;
        int blockPivElOffset = by * BLOCKSIZE * size + diagEl;
        int blockPivOffset = diagEl * size + bx * BLOCKSIZE;

        inBlock[ty][tx] = dMatrixIn[blockOffset + ty * size + tx];
        pivEl[ty][tx] = dMatrixIn[blockPivElOffset + ty * size + tx];
        pivBlock[ty][tx] = dMatrixIn[blockPivOffset + ty * size + tx];
        
        __syncthreads ();

        //Subtract each row from the input Matrix =>dMatrixIn
        
        for (int i = 0; i < BLOCKSIZE; i++)
        {
            inBlock[ty][tx] = hipCsub(inBlock[ty][tx], hipCmul(pivEl[ty][i],pivBlock[i][tx]));
        }
        
        __syncthreads ();
        
        if( (blockOffset + ty*size + tx) < (size*size) )
        {
             dMatrixIn[blockOffset + ty * size + tx] = inBlock[ty][tx];
        }
        
        __syncthreads ();

         inBlock[ty][tx] = dMatrixInv[blockOffset + ty * size + tx];
        pivBlock[ty][tx] = dMatrixInv[blockPivOffset + ty * size + tx];
        
        __syncthreads ();

        //Subtract each row from the invers Matrix =>dMatrixInv
        
        for (int i = 0; i < BLOCKSIZE; i++)
        {
            inBlock[ty][tx] = hipCsub(inBlock[ty][tx], hipCmul(pivEl[ty][i],pivBlock[i][tx]));
        }

        __syncthreads ();
        
             dMatrixInv[blockOffset + ty * size + tx] = inBlock[ty][tx];
        
    }
    
    //Adjust the left Blocks from the invers matrix which are left from the diagEl
    
    else if (by * BLOCKSIZE > (diagEl + 1))
    {
        int blockOffset = by * BLOCKSIZE * size + bx * BLOCKSIZE;
        int blockPivElOffset = by * BLOCKSIZE * size + diagEl;
        int blockPivOffset = diagEl * size + bx * BLOCKSIZE;

         pivEl[ty][tx] = dMatrixIn[blockPivElOffset + ty * size + tx];
       inBlock[ty][tx] = dMatrixInv[blockOffset + ty * size + tx];
      pivBlock[ty][tx] = dMatrixInv[blockPivOffset + ty * size + tx];
        
        __syncthreads ();

        for (int i = 0; i < BLOCKSIZE; i++)
        {
            inBlock[ty][tx] = hipCsub(inBlock[ty][tx], hipCmul(pivEl[ty][i],pivBlock[i][tx]));
        }

        __syncthreads ();

       dMatrixInv[blockOffset + ty * size + tx] = inBlock[ty][tx];
        
    }
}

//************************************************
// Kernel eliminateRestU
//************************************************

__global__ void eliminateRestU(int BLOCKSIZE, hipDoubleComplex *dMatrixIn, hipDoubleComplex *dMatrixInv, int size,int diagEl)
{
    int BLOCKSIZEMINUS1 = BLOCKSIZE-1;
          
    int tx = threadIdx.x; int ty = threadIdx.y;
    int bx = blockIdx.x;  int by = blockIdx.y;

    __shared__ hipDoubleComplex pivEl[MAXSIZE][MAXSIZE];
    __shared__ hipDoubleComplex pivBlock[MAXSIZE][MAXSIZE];
    __shared__ hipDoubleComplex inBlock[MAXSIZE][MAXSIZE];

    //rest der unbearbeiteten Matrix bearbeiten

    if ((bx * BLOCKSIZE + 1) <diagEl && (by * BLOCKSIZE +1) <diagEl)     //linke seite von in; 0-pivblock
    {
        int blockOffset      = by * BLOCKSIZE * size + bx * BLOCKSIZE;
        int blockPivElOffset = by * BLOCKSIZE * size + diagEl;
        int blockPivOffset   = diagEl * size + bx * BLOCKSIZE;

         inBlock[ty][tx] = dMatrixIn[blockOffset + ty * size + tx];
           pivEl[ty][tx] = dMatrixIn[blockPivElOffset + ty * size + tx];
        pivBlock[ty][tx] = dMatrixIn[blockPivOffset + ty * size + tx];
        
        __syncthreads ();

        for (int i = BLOCKSIZEMINUS1; i >= 0; i--)
        {
            inBlock[ty][tx] = hipCsub(inBlock[ty][tx], hipCmul(pivEl[ty][i],pivBlock[i][tx]));
        }
        
        __syncthreads ();
        
        if( (blockOffset + ty*size + tx) < (size*size) )
        { 
             dMatrixIn[blockOffset + ty * size + tx] = inBlock[ty][tx];
        }
        
        __syncthreads ();


        inBlock[ty][tx] = dMatrixInv[blockOffset + ty * size + tx];
       pivBlock[ty][tx] = dMatrixInv[blockPivOffset + ty * size + tx];
        
        __syncthreads ();

        for (int i = BLOCKSIZEMINUS1; i >= 0; i--)
        {
            inBlock[ty][tx] = hipCsub(inBlock[ty][tx], hipCmul(pivEl[ty][i],pivBlock[i][tx]));
        }

        __syncthreads ();
        
        dMatrixInv[blockOffset + ty * size + tx] = inBlock[ty][tx];
        
    }
    
    else if (by * BLOCKSIZE < (diagEl))
    {
        int blockOffset = by * BLOCKSIZE *size + bx * BLOCKSIZE;
        int blockPivElOffset = by * BLOCKSIZE *size + diagEl;
        int blockPivOffset = diagEl * size + bx * BLOCKSIZE;

        pivEl[ty][tx] = dMatrixIn[blockPivElOffset + ty * size + tx];
      inBlock[ty][tx] = dMatrixInv[blockOffset + ty * size + tx];
     pivBlock[ty][tx] = dMatrixInv[blockPivOffset + ty * size + tx];
        
      __syncthreads ();



        for (int i = BLOCKSIZEMINUS1; i >= 0; i--)
        {
            inBlock[ty][tx] = hipCsub(inBlock[ty][tx], hipCmul(pivEl[ty][i],pivBlock[i][tx]));
        }

        __syncthreads ();

        dMatrixInv[blockOffset + ty * size + tx] = inBlock[ty][tx];
          
    }
}


//************************************************
// Kernel normalizeDiag
//************************************************

__global__ void normalizeDiag(int BLOCKSIZE,hipDoubleComplex *diagMatrix, hipDoubleComplex *invMatrix, int size,int row)
{
          
    int tx = threadIdx.x; int ty = threadIdx.y; int bx = blockIdx.x;

    int blockOffset = bx * BLOCKSIZE;

    __shared__ hipDoubleComplex diagEl[MAXSIZE];

    if (tx == ty )
    {
        diagEl[ty] = diagMatrix[row + ty * size + tx];
    }
    __syncthreads ();

    invMatrix[blockOffset + ty * size + tx] = hipCdiv(
    invMatrix[blockOffset + ty * size + tx],diagEl[ty]);
    
}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
// Wrapper Fortran / C
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************


extern "C" void cuda_complex_invert_(int *pBlockSize, hipDoubleComplex** A_, hipDoubleComplex** invA_, int *pndim)
{
    int i; int size = *pndim; int BlockSize = *pBlockSize;

    dim3 idyThreads (BlockSize);    
    dim3 idyBlocks ( size / BlockSize );
    dim3 nThreads (BlockSize, BlockSize);   
    dim3 nBlocks ( size / BlockSize );        
    dim3 nBlocksRest ( size / BlockSize, size / BlockSize);

    unsigned int mat_size=size*size*sizeof(hipDoubleComplex);

    int pinned = 1 ; if(size<3600) pinned=0;

//  printf(" copy memory host to gpu \n ");

    hipDoubleComplex *invA ; hipDoubleComplex *A  ; hipDoubleComplex *invAd ; hipDoubleComplex *Ad;


//---------------------------------------------------------------//
    if(pinned==0){
//    printf(" copy memory host to gpu  \n ");
      hipMalloc((void**) &invA, mat_size);
      hipMalloc((void**) &A   , mat_size);
      hipMemcpy(A   , A_   , mat_size, hipMemcpyHostToDevice);
      hipMemcpy(invA, invA_, mat_size, hipMemcpyHostToDevice);
    }else{
      hipSetDevice(0); hipSetDeviceFlags( hipDeviceMapHost );
      hipHostAlloc( (void**) &Ad,    mat_size, hipHostMallocMapped | hipHostMallocPortable );
      hipHostAlloc( (void**) &invAd, mat_size, hipHostMallocMapped | hipHostMallocPortable );
      hipHostGetDevicePointer((void**) &A   , Ad,    0 );
      hipHostGetDevicePointer((void**) &invA, invAd, 0 );
      hipMemcpy(Ad,    A_,    mat_size, hipMemcpyHostToHost);
      hipMemcpy(invAd, invA_, mat_size, hipMemcpyHostToHost);
      hipEventSynchronize(0);
    }
//---------------------------------------------------------------//



    GPUsetIdentity <<< idyBlocks, idyThreads >>> (BlockSize,invA, size);


    hipDeviceSynchronize ();

    //calculate the right diagonal Matrix (L)

    for (i = 0; i < size; i += BlockSize)
    {
        int offset = i * size + i;

        // *  step 1:
        // *  calculate the triangle matrix
        // *  store the pivot elements to left part of the triangel

        eliminateBlock<<< 1, nThreads >>> ('L',BlockSize,A + offset, size);
        
        hipDeviceSynchronize ();
    
        // * step 2:
        // *  calculate the rest of the rows with the pivot elements from step 1
        
        adjustRowL<<< nBlocks, nThreads >>> (BlockSize,A + i * size, A + offset, invA + i * size, size, i);
        
        hipDeviceSynchronize ();
    
        //* step 3:
        //* Fill the colls below the block with the pivot elements they are used
        //*    to get the colls to zero and multiply with the row
        
        eliminateColL<<< nBlocks, nThreads >>> (BlockSize,A + i, size, i);
        
        hipDeviceSynchronize ();

        //* step 4:
        //*  Adjust the rest of the Matrix with the calculated pivot Elements
        //*  El_new_0 -= (p0+p1+p2..+p15) * El_piv_0
        
        eliminateRestL<<< nBlocksRest, nThreads >>> (BlockSize,A, invA, size, i);
        
        hipDeviceSynchronize ();
    }
    


   //Set the left lower diagonalmatrix to zero (async?)

    for (i = 1; i < size; i++)
    {
      int offset = i * size;
      if(pinned==0){
         hipMemset ((void *) (A  + offset), 0, i*sizeof(hipDoubleComplex));}
      else{
         memset ((void *) (Ad  + offset), 0, i*sizeof(hipDoubleComplex));
      };
      hipEventSynchronize(0);
    }
    hipDeviceSynchronize ();




    //calculate the right diagonal Matrix (U)
    
    for (i = (size - BlockSize); i >= 0; i -= BlockSize)
    {
        int offset = i * size + i;

        //*  step 1:
        //*  calculate the triangle matrix
        //*  store the pivot elements to left part of the triangel
        
        eliminateBlock<<< 1, nThreads >>> ('U',BlockSize,A + offset, size);
        
        hipDeviceSynchronize ();

        //* step 2:
        //*  calculate the rest of the rows with the pivot elements from step 1
        
        adjustRowU<<< nBlocks, nThreads >>> (BlockSize,A + offset,invA + i*size, size, i);
        
        hipDeviceSynchronize ();

        //* step 3:
        //*  Fill the colls below the block with the pivot elements they are used
        //*      to get the colls to zero and multiply with the row
                
        eliminateColU<<< nBlocks, nThreads >>> (BlockSize,A + i, size, i);
        
        hipDeviceSynchronize ();

        //* step 4:
        //*  Adjust the rest of the Matrix with the calculated pivot Elements
        //*  El_new_0 -= (p0+p1+p2..+p15) * El_piv_0
        
        eliminateRestU<<< nBlocksRest, nThreads >>> (BlockSize,A, invA, size, i);
        
        hipDeviceSynchronize ();
    }
    
    
    //* force the diagonal entries to 1
    
    for (i = 0; i < size; i += BlockSize)
    {
        int rowOffset = i * size;
        normalizeDiag<<< nBlocks, nThreads >>> (BlockSize,A + rowOffset, invA + rowOffset, size, i);
        hipDeviceSynchronize ();
    }

//---------------------------------------------------------------//
  if(pinned==0){
     hipMemcpy(invA_,invA,mat_size,hipMemcpyDeviceToHost);
     hipFree(invA);
     hipFree(A);
   }else{
     hipMemcpy(invA_,invAd, mat_size,hipMemcpyHostToHost);
     hipHostFree(Ad);
     hipHostFree(invAd);
     hipEventSynchronize(0);
   };
//---------------------------------------------------------------//

//  printf(" SUCCESFULL DIAG. \n \n \n ");

    return;
}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

extern "C" void cuda_c_invert_(int *pBlockSize, double** A_, double** A_i, double** invA_, double** invA_i, int *pndim)
{

  int i; int nlines=*pndim; int ncolumns=*pndim; int ntot=nlines*ncolumns ;
  hipDoubleComplex **aa; hipDoubleComplex **inva;
  aa   = (hipDoubleComplex**) malloc(sizeof(hipDoubleComplex)*ntot);
  inva = (hipDoubleComplex**) malloc(sizeof(hipDoubleComplex)*ntot);

  for(i=0; i<ntot; i++)
   {
  *aa[i]=make_hipDoubleComplex(*A_[i],*A_i[i])
  ;};

  int k = nlines/2  ;
  cuda_complex_invert_(&k, aa, inva, &nlines);

  for(i=0; i<ntot; i++)
   {
  *invA_[i]  =hipCreal(*inva[i]);
  *invA_i[i] =hipCimag(*inva[i]);
  };

  free(inva);free(aa);

}

//********************************************
//********************************************
//********************************************
//********************************************
//********************************************
//********************************************

